#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif

#include "mex.h"
#include "matrix.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

// GPUmat
#include "GPUmat.hh"

// static paramaters
static int init = 0;
static GPUmat *gm;

#include "cudaKernels.h"

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  // At least 2 arguments expected
  // Input and result
  if ((nrhs!=3) && ((nlhs != 1) && (nrhs != 2)))
     mexErrMsgTxt("Wrong number of arguments: 1lhs + 2rhs or 3rhs acceptable");
  if (init == 0) {
    // Initialize function
    // mexLock();
    // load GPUmat
    gm = gmGetGPUmat();
    init = 1;
  }
  /* mex parameters are:
   0 Source data
   1 Destination data
   2 Coefficients [4]
  */

  // Sort out source/dest stuff
  GPUtype srcArray    = gm->gputype.getGPUtype(prhs[0]);
  double *opCoeffs;
  GPUtype dstArray;
  if(nrhs == 3) {
	dstArray    = gm->gputype.getGPUtype(prhs[1]);
	opCoeffs    = mxGetPr(prhs[2]);
  } else {
	//dstArray = create LHS
	// U = teh fu><X0r3dz, we no support diz
	opCoeffs    = mxGetPr(prhs[1]);
  }

  // Get some control variables sorted out
  const int *dims    = gm->gputype.getSize(srcArray);

  dim3 gridsize;
  gridsize.x = dims[0]*dims[2]/64;
  gridsize.y = dims[1]/8;
  gridsize.z = 1;

  dim3 blocksize; blocksize.x = blocksize.y = 10; blocksize.z = 1;

  int nx = dims[0];
  int ny = dims[1];
  int nz = dims[2]/8 - 1;

  SymmetricOperatorKernel<<<gridsize, blocksize>>>((double*)gm->gputype.getGPUptr(srcArray), (double*)gm->gputype.getGPUptr(dstArray), nx, ny, nz, opCoeffs[0], opCoeffs[1], opCoeffs[2], opCoeffs[3]);

}

