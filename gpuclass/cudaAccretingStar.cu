#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"
#include "mpi.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"

// File applies the effects of a star undergoing active accretion
// This is presumably done at the center of a disk
// Determines if any of the points we have access to out o the global grid currently have the
// star within range to accrete.
// If so, we examine all cells within a certain distance of the star in planes
// Effects to look out for:

// 1. Mass accretion - remove mass from the grid and add it to the star instead; bring the "blanked" cells o a halt
// 2. Angular momentum xfer - add the sum of angular momentum about the star of the removed mass to the star's L
// 3. Accretion luminosity - At least track the accretion luminosity

void __global__ cudaStarAccretes(double *rho, double *px, double *py, double *pz, double *E, int3 gridLL, double h, int nx, int ny, int nz, double *stateOut, int ncellsvert);
void __global__ cudaStarGravitation(double *rho, double *px, double *py, double *pz, double *E, int3 arraysize);

__constant__ __device__ double starState[14];
// Need to track star's
// position (3D), momentum (3D), angular momentum (3D), mass (1D), radius (1D), vaccum_rho grav_rho vac_E(3D) = 14 doubles
#define STAR_X      0
#define STAR_Y      1
#define STAR_Z      2
#define STAR_RADIUS 3
#define STAR_PX     4
#define STAR_PY     5
#define STAR_PZ     6
#define STAR_LX     7
#define STAR_LY     8
#define STAR_LZ     9
#define STAR_MASS   10
#define VACUUM_RHO  11
#define VACUUM_RHOG 12
#define VACUUM_E    13

__constant__ __device__ double gravParams[9];
#define GRAVP_GMDT 0
#define GRAVP_X0   1
#define GRAVP_Y0   2
#define GRAVP_Z0   3
#define GRAVP_H    4


#define ACCRETE_NX 8
#define ACCRETE_NY 8

#define GRAVITY_NX 16
#define GRAVITY_NY 16

// FIXME: NOTE: The below can be implemented but unless we need to again I see no reason to actually do that.

// Define: F = -beta * rho * grad(phi)
// rho_g = density for full effect of gravity 
// rho_c = minimum density to feel gravity at all
// beta = { rho_g < rho         : 1                                 }
//        { rho_c < rho < rho_g : [(rho-rho_c)/(rho_rho_g-rho_c)]^2 }
//        {         rho < rho_c : 0                                 }

// This provides a continuous (though not differentiable at rho = rho_g) way to surpress gravitation of the background fluid
// The original process of cutting gravity off below a critical density a few times the minimum
// density is believed to cause "blowups" at the inner edge of circular flow profiles due to being
// discontinuous. If even smoothness is insufficient and smooth differentiability is required,
// a more-times-continuous profile can be constructed, but let's not go there unless forced.

// Density below which we force gravity effects to zero
#define RHOMIN gravParams[5]
#define RHOGRAV gravParams[6]
// 1 / (rho_g - rho_c)
#define G1 gravParams[7]
// rho_c / (rho_g - rho_c)
#define G2 gravParams[8]

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
    // At least 2 arguments expected
    // Input and result
    if ((nrhs != 10) || (nlhs != 1)) mexErrMsgTxt("Wrong number of arguments: need newState = cudaAccretingStar(rho, px, py, pz, E, starState, lower left corner global index, grid size H, dt, topology_size);");

    cudaCheckError("entering cudaAccretingStar");

    double lowleft[3], upright[3];
    double *i0 = mxGetPr(prhs[6]);
    double H   = *mxGetPr(prhs[7]);
    double dtime = *mxGetPr(prhs[8]);
    double *topologySize = mxGetPr(prhs[9]);

    double *hostStarState = mxGetPr(prhs[5]);
    // Position transform the star into rank-local coordinates by subtracting off our offset coordinate scaled by h.
    double originalStarState[14];
    int j;
    for(j = 0; j < 3; j++) originalStarState[j] = hostStarState[j] - H*i0[3+j];
    for(j = 3; j < 14; j++) originalStarState[j] = hostStarState[j];
  
    hipMemcpyToSymbol(HIP_SYMBOL(starState), &originalStarState[0], 14*sizeof(double), 0, hipMemcpyHostToDevice);
    cudaCheckError("Copying star state to __constant__ memory");

    // Get source array info and create destination arrays
    ArrayMetadata amd;
    double **srcs = getGPUSourcePointers(prhs, &amd, 0, 4);

    // Check if any part of the stellar accretion region is on our grid.
    int mustAccrete = 1;
    for(j = 0; j < 3; j++) {
        lowleft[j] = (i0[j]-1+3*(topologySize[j] > 1) )*H; // Use topologySize to avoid accreting from halo region, which would get doublecounted
        upright[j] = (i0[j]-1+amd.dim[j]-3*(topologySize[j] > 1))*H;

        if(lowleft[j] > originalStarState[STAR_X+j] + originalStarState[STAR_RADIUS]) mustAccrete = 0; // If our left  > R_star from the star, no accretion
        if(upright[j] < originalStarState[STAR_X+j] - originalStarState[STAR_RADIUS]) mustAccrete = 0; // If our right > R_star from the star, no accretion
        }

    int bee;
    MPI_Comm_rank(MPI_COMM_WORLD, &bee);
    //printf("rank %i: LL=[%g %g %g] *=[%g %g %g] UR=[%g %g %g] mustAccrete=%i", bee, lowleft[0], lowleft[1], lowleft[2], originalStarState[STAR_X+0], originalStarState[STAR_X+1], originalStarState[STAR_X+2], upright[0], upright[1], upright[2], mustAccrete); fflush(stdout);

    // Each rank stores its final accumulated sum here
    double localFinalDelta[7];
    for(j = 0; j < 7; j++) localFinalDelta[j] = 0;
    int nparts = 0;
    double *hostDeltas;

    if(mustAccrete) {
//        printf("rank %i accreting...\n",bee);
        int starRadInCells = originalStarState[STAR_RADIUS] / H + 2;
        // Determine the target region
        dim3 acBlock, acGrid;
        acBlock.x = ACCRETE_NX; acBlock.y = ACCRETE_NY; acBlock.z = 1;
        acGrid.x = 2*starRadInCells/ACCRETE_NX; acGrid.x += (ACCRETE_NX*acGrid.x < 2*starRadInCells);
        acGrid.y = 2*starRadInCells/ACCRETE_NY; acGrid.y += (ACCRETE_NY*acGrid.y < 2*starRadInCells);
        acGrid.z = 1;
        double *stateOut;
        nparts = acGrid.x*ACCRETE_NX*acGrid.y*ACCRETE_NY;
        hostDeltas = (double *)malloc(sizeof(double)*nparts*8);

        hipError_t fail = hipMalloc((void **)&stateOut, sizeof(double)*nparts*8);

        // Makes sure that we don't accrete from the halo zone
        int3 LL;
        LL.x = (int)((originalStarState[0] - originalStarState[3])/H) - 2;
	LL.y = (int)((originalStarState[1] - originalStarState[3])/H) - 2;
	LL.z = (int)((originalStarState[2] - originalStarState[3])/H) - 2;
//printf("Orig  LL: %i %i %i\n", LL.x, LL.y, LL.z);
        // Force the block to not begin further left than the left part of our fluid domain
        if(LL.x < 3*(topologySize[0] > 1)) LL.x = 3*(topologySize[0] > 1);
        if(LL.y < 3*(topologySize[1] > 1)) LL.y = 3*(topologySize[1] > 1);
        if(LL.z < 3*(topologySize[2] > 1)) LL.z = 3*(topologySize[2] > 1);
        // Also force it to not begin further right
        // We presume that the domain & radius are compatible with the left and right edge conditions not being simultaneously met
        // As might be implied by the phrase "COMPACT object".
        if( (LL.x + acGrid.x*ACCRETE_NX) > (amd.dim[0]-3*(topologySize[0] > 1)) ) LL.x = amd.dim[0] - 3*(topologySize[0] > 1) - acGrid.x*ACCRETE_NX;
        if( (LL.y + acGrid.y*ACCRETE_NY) > (amd.dim[1]-3*(topologySize[1] > 1)) ) LL.y = amd.dim[1] - 3*(topologySize[1] > 1) - acGrid.y*ACCRETE_NY;
        int nvertical = 2*starRadInCells + 8;
        if( (LL.z + nvertical) > (amd.dim[2]-3*(topologySize[2] > 1)) ) LL.z = amd.dim[2] - 3*(topologySize[2] > 1) - nvertical;
        
//printf("check LL: %i %i %i\n", LL.x, LL.y, LL.z);
//printf("check gs: %i %i %i\n", acGrid.x, acGrid.y, acGrid.z);
//printf("check bs: %i %i %i\n", acBlock.x,acBlock.y,acBlock.z);

        // call accretion kernel: transfers bits of changed state to outputState
        cudaStarAccretes<<<acGrid, acBlock>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], LL, H, amd.dim[0], amd.dim[1], amd.dim[2], stateOut, 2*starRadInCells+8);

        hipDeviceSynchronize(); // Force accretion to finish.
        cudaCheckError("running cudaStarAccretes()");
        fail = hipMemcpy((void *)hostDeltas, (void *)stateOut, 8*sizeof(double)*nparts, hipMemcpyDeviceToHost);
        cudaCheckError("copying accretion results to host");
        hipDeviceSynchronize();
cudaCheckError("sync after copy start");
        hipFree(stateOut);
cudaCheckError("free stateOut after copy & sync");
        }
    

    // Produce a single accumulated delta for all ranks,
    // FIXME: We'll give a crap about absorbed angular momentum once we're actually in a position to /do/ something about it.
    if(mustAccrete) {
        int k;
        for(j = 0; j < nparts; j++) {
            for(k = 0; k < 5; k++) { localFinalDelta[k] += hostDeltas[8*j+k]; }
            // localFinalDelta[0 1 2 3 4] = absorbed [mass px py pz E] / dV        
        }

        free(hostDeltas);
    }

    // Add up all the changes
    double finalDelta[7];
    int mpi_error = MPI_Allreduce((void *)&localFinalDelta[0], (void *)&finalDelta[0], 5, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);

    // Produce the change we send back for the
    // [X Y Z R Px Py Pz Lx Ly Lz M rhoV, EV]
    mwSize outputdim[2]; outputdim[0] = 1; outputdim[1] =14;
    plhs[0] = mxCreateNumericArray(2, (const mwSize *)&outputdim, mxDOUBLE_CLASS, mxREAL);
    double *outputDelta = mxGetPr(plhs[0]);

    double dv = H*H*H;

    // First we need to calculate the output delta we'll hand back given a full timestep.
//    <calculate accretion rate here>
//    [fluid flux]   ^^^  [half accrete] [half star drift] [source grav.pot.] [.5 drift] [.5 accrete] [fluid flux]
// originalStarState
// Bear in mind, originalStarState is transformed re:position such that our grid's <0 0 0> index is the coordinate origin.

    // deltaX: evaluate with dt*[P + Pdot*dt/2]/[M + Mdot*dt/2]
    outputDelta[0] = dtime * (originalStarState[STAR_PX] + finalDelta[1]*dv)/(originalStarState[STAR_MASS] + finalDelta[0]*dv);
    outputDelta[1] = dtime * (originalStarState[STAR_PY] + finalDelta[2]*dv)/(originalStarState[STAR_MASS] + finalDelta[0]*dv);
    outputDelta[2] = dtime * (originalStarState[STAR_PZ] + finalDelta[3]*dv)/(originalStarState[STAR_MASS] + finalDelta[0]*dv);
    // delta in radius: 0 pending construction of a more sophisticated model
    outputDelta[3] = 0; 
    // delta in momentum
    outputDelta[4] = finalDelta[1]*dv;
    outputDelta[5] = finalDelta[2]*dv;
    outputDelta[6] = finalDelta[3]*dv;
    // delta in angular momentum: don't care
    outputDelta[7] = outputDelta[8] = outputDelta[9] = 0;
    // delta in mass
    outputDelta[10] = finalDelta[0]*dv;
    // delta in "vacuum" density/energy density
    outputDelta[11] = outputDelta[12] = outputDelta[13] = 0;
    // Now we need to calculate the position & mass at halftime so we can go ahead and calculate gravitation on the fluid
    // store parameters in constant memory: G*M*dt, [Xstar Ystar Zstar], H
    double gp[9];
    gp[0] = dtime*(originalStarState[STAR_MASS] + .5*outputDelta[STAR_MASS]);
    gp[1] = originalStarState[STAR_X] + .5*outputDelta[STAR_X];
    gp[2] = originalStarState[STAR_Y] + .5*outputDelta[STAR_Y];
    gp[3] = originalStarState[STAR_Z] + .5*outputDelta[STAR_Z];
    gp[4] = H;
    gp[5] = originalStarState[VACUUM_RHO];
    gp[6] = originalStarState[VACUUM_RHOG];
    gp[7] = 1.0/(originalStarState[VACUUM_RHOG] - originalStarState[VACUUM_RHO]);
    gp[8] = originalStarState[VACUUM_RHO] *gp[7];

//if(mustAccrete) {
//  int qq;
//  printf("copied to gravParams: ");
//  for(qq = 0; qq < 9; qq++) { printf("%lg ",gp[qq]); }
//printf("\n");
//  }
    cudaCheckError("memcpy to symbol before gravitate");
    hipMemcpyToSymbol(HIP_SYMBOL(gravParams), &gp[0], 9*sizeof(double), 0, hipMemcpyHostToDevice);
    cudaCheckError("point gravity symbol copy");
    hipDeviceSynchronize();

    dim3 gravBlock, gravGrid;

    int3 arraysize; arraysize.x = amd.dim[0]; arraysize.y = amd.dim[1]; arraysize.z = amd.dim[2];

    int *dim = &amd.dim[0];
    getLaunchForXYCoverage(dim, GRAVITY_NX, GRAVITY_NY, 0, &gravBlock, &gravGrid); 

    cudaStarGravitation<<<gravGrid, gravBlock>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], arraysize);
    cudaCheckError("Ran pointlike gravitation routine");

}


// This occupies a relatively small number of SMs, so we wish to run it concurrently with the point gravitation kernel which everybody does.


// Need to track star's
// position (3D), momentum (3D), angular momentum (3D), mass (1D), radius (1D), vaccum_rhoE(2D) = 13 doubles

// Store [X Y Z R Px Py Pz Lx Ly Lz M rhoV, EV] in full state vector:
//                   Need to read only [X Y Z R] to calc additions
// Calculate differential accumulation of the above which is
//                   Need output only [dP, dL, dM] = 7x1

// Launch with however many threads/blocks are appropriate to cover entire stellar accretion region.
// 
void __global__ cudaStarAccretes(double *rho, double *px, double *py, double *pz, double *E, int3 gridLL, double h, int nx, int ny, int nz, double *stateOut, int ncellsvert)
{
int myx = threadIdx.x + ACCRETE_NX * blockIdx.x + gridLL.x;
int myy = threadIdx.y + ACCRETE_NY * blockIdx.y + gridLL.y;
int myz = gridLL.z;
int z;

// Zero my contribution to delta-state
double dstate[7];
for(z = 0; z < 7; z++) dstate[z] = 0.0;

if((myx >= nx) || (myy >= ny)) return;
int globAddr = myx + nx*(myy + ny*myz);

// Load stellar state vector
double starX = starState[STAR_X];
double starY = starState[STAR_Y];
double starZ = starState[STAR_Z];
double starR = starState[STAR_RADIUS];
 
double accFactor = 1.0; // If we're at a face/edge/corner then multiple ranks will accrete so reduce appropriately.

if( (myx < 3) || (myx > (nx-4))) accFactor = .5;
if( (myy < 3) || (myy > (ny-4))) accFactor *= .5;
// FIXME: This needs to account for steppign through z. Unroll Z loops and add only half if at edge.
//if( (myz < 3) || (myz > (nz-4))) accFactor *= .5;

// We step up columns in the Z direction so the "axial" radius is fixed
double dXYsqr = (h*myx-starX)*(h*myx-starX) + (h*myy-starY)*(h*myy-starY);
double dz = h*myz - starZ;
double q;

for(z = 0; z < ncellsvert; z++) {
  // Calculate my grid position
//  if(dz > starR) break; // Quit once we're beyond the accretion sphere
  double rsqr = dXYsqr + dz*dz;

// Calculate how far it is from the given X of the star
  if(rsqr < starR*starR) {
    //If within, add stuff to local state vector:
    // We'll rescale by h^3 after on the cpu, once.

    // Move the mass to our dmass, set the density back to minimum
    q = rho[globAddr];
    dstate[0] += (q-starState[VACUUM_RHO]);
    rho[globAddr] = starState[VACUUM_RHO];
        
    // Add dv*mom to Pstar, write zero to mom
    q = px[globAddr]; dstate[1] += q; px[globAddr] = 0;
    q = py[globAddr]; dstate[2] += q; py[globAddr] = 0;
    q = pz[globAddr]; dstate[3] += q; pz[globAddr] = 0;

    // Move dv*(E - vaccuum_E) to star, write vacuum_E to ener
    q = E[globAddr]; dstate[4] += q; E[globAddr] = starState[VACUUM_E];

    }

  globAddr += nx*ny;
  dz += h;
  }

__syncthreads();

myx -= gridLL.x;
myy -= gridLL.y;
int i0 = (myx + ACCRETE_NX*gridDim.x*myy)*8;

for(z = 0; z < 7; z++) { stateOut[i0+z] = accFactor * dstate[z]; }

}


//access gravParams[] using:
//define GRAVP_GMDT 0
//define GRAVP_X0   1
//define GRAVP_Y0   2
//define GRAVP_Z0   3
//define GRAVP_H    4
//#define RHOMIN gravParams[5]
//#define RHOGRAV gravParams[6]
// 1 / (rho_g - rho_c)
//#define G1 gravParams[7]
// rho_c / (rho_g - rho_c)
//#define G2 gravParams[8]


void __global__ cudaStarGravitation(double *rho, double *px, double *py, double *pz, double *E, int3 arraysize)
{
int myx = threadIdx.x + GRAVITY_NX*blockIdx.x;
int myy = threadIdx.y + GRAVITY_NY*blockIdx.y;

int globAddr = myx + arraysize.x*myy;

if((myx >= arraysize.x) || (myy >= arraysize.y)) return;
double H = gravParams[GRAVP_H];

double dx = myx*H - gravParams[GRAVP_X0];
double dy = myy*H - gravParams[GRAVP_Y0];
double dz = -gravParams[GRAVP_Z0];
double rXYsqr = dx*dx + dy*dy; // This is constant.
double radius;

__shared__ double locRho[GRAVITY_NX][GRAVITY_NY];
__shared__ double locE  [GRAVITY_NX][GRAVITY_NY];
__shared__ double locMom[GRAVITY_NX][GRAVITY_NY];

double dQ;

//int Amax = arraysize.x*arraysize.y*arraysize.z;
int dAddr = arraysize.x * arraysize.y;
int z;
for(z = 0; z < arraysize.z; z++) {
//; globAddr < Amax; globAddr += arraysize.x*arraysize.y) {
  locRho[threadIdx.x][threadIdx.y] = rho[globAddr];
  locE  [threadIdx.x][threadIdx.y] = E[globAddr];

  if(locRho[threadIdx.x][threadIdx.y] > RHOGRAV) {
    radius = sqrt(rXYsqr + dz*dz);

    dQ = gravParams[GRAVP_GMDT] / (radius*radius*radius);
    // We have dQ = -G*M*dt*rhat / r^3
    // Then change in momentum = dP = F dt = rho d[x y z] dQ
    // And change in energy    = dE = F dot V dt = rho * V * d[x y z] dQ = P * d[x y z] * dQ;
    locMom[threadIdx.x][threadIdx.y] = px[globAddr];
    locE  [threadIdx.x][threadIdx.y] = -dQ*dx*locMom[threadIdx.x][threadIdx.y];
    px[globAddr] = locMom[threadIdx.x][threadIdx.y] - dQ*locRho[threadIdx.x][threadIdx.y]*dx;

    locMom[threadIdx.x][threadIdx.y] = py[globAddr];
    locE  [threadIdx.x][threadIdx.y] -= dQ*dy*locMom[threadIdx.x][threadIdx.y];
    py[globAddr] = locMom[threadIdx.x][threadIdx.y] - dQ*locRho[threadIdx.x][threadIdx.y]*dy;

    locMom[threadIdx.x][threadIdx.y] = pz[globAddr];
    locE  [threadIdx.x][threadIdx.y] -= dQ*dz*locMom[threadIdx.x][threadIdx.y];
    pz[globAddr] = locMom[threadIdx.x][threadIdx.y] - dQ*locRho[threadIdx.x][threadIdx.y]*dz;

    E[globAddr] += locE[threadIdx.x][threadIdx.y];
    }

    dz += H;
    globAddr += dAddr;
  }

}
