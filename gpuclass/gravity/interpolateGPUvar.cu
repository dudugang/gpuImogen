#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UoutputDimensionsIX
#include <stdint.h>
#include <unistd.h>
#endif

#include "mex.h"
#include "matrix.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaKernels.h"
#include "cudaCommon.h"

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  // At least 2 arguments expected
  // Input and result

ArrayMetadata srcmeta;
ArrayMetadata dstmeta;
double **srcArray;
double **dstArray;

int direct;
int scaleFactor;
int outputDimensions[3];
const int *inputDimensions;
int *launchdims;

if (nrhs == 3) {
  // Get GPU array pointers if both are provided
  srcArray = getGPUSourcePointers(prhs, &srcmeta, 0, 0);
  dstArray = getGPUSourcePointers(prhs, &dstmeta, 2, 2);
  } else if ((nlhs == 1) && (nrhs == 2)) {
  srcArray = getGPUSourcePointers(prhs, &srcmeta, 0, 0);
  } else mexErrMsgTxt("GPU interpolate error: either 3 RHS or 1LHS + 2RHS arguments required\n");

// Get scaling scaleFactoror
scaleFactor = (int)*mxGetPr(prhs[1]);
inputDimensions = srcmeta.dim;

if (scaleFactor < 0) {
  // If scaling down, divide output size outputDimensions by scaleFactor; We launch one thread per output cell
  scaleFactor = -scaleFactor;

  for(direct = 0; direct < 3; direct++) {
    outputDimensions[direct] = inputDimensions[direct] / scaleFactor;
    if(outputDimensions[direct]*scaleFactor < inputDimensions[direct]) outputDimensions[direct]++;
    }

  direct = -1;
  launchdims = outputDimensions;
  } else {
  // If scaling up, multiply output size outputDimensions by scaleFactoror; We launch one thread per input cell
  direct = 1;  
  outputDimensions[0] = inputDimensions[0] * scaleFactor;
  outputDimensions[1] = inputDimensions[1] * scaleFactor;
  outputDimensions[2] = inputDimensions[2] * scaleFactor;
  launchdims = (int *)inputDimensions;
  }

// Creating output array, it will match correct dimensions.
// If dest array is given, check for dimensional correctness.
if (nlhs == 1) {
  int64_t ref[5];
  ref[0] = 0; ref[1] = (outputDimensions[2] == 1 ? 2 : 3);
  ref[2] = outputDimensions[0];
  ref[3] = outputDimensions[1];
  ref[4] = outputDimensions[2];

  dstArray = makeGPUDestinationArrays(ref, plhs, 1);
  } else {
  const int *dstdims = dstmeta.dim;
  int d;
  for(d = 0; d < 3; d++) { if(dstdims[d] != outputDimensions[d]) mexErrMsgTxt("GPU interpolate error: destination array is wrong size.\n"); }
  }

dim3 gridsize;
gridsize.x = launchdims[0]/8;
gridsize.y = launchdims[1]/8;
gridsize.z = 1;

if(gridsize.x * 8 < launchdims[0]) gridsize.x++;
if(gridsize.y * 8 < launchdims[1]) gridsize.y++;

dim3 blocksize; blocksize.x = blocksize.y = 8;
blocksize.z = 1;

int nx = launchdims[0];
int ny = launchdims[1];
int nz = launchdims[2];
  if(nz == 0) nz = 1;

  if(direct > 0)
      upsampleKernel<<<gridsize, blocksize>>>(srcArray[0], dstArray[0], scaleFactor, nx, ny, nz);
  else
    downsampleKernel<<<gridsize, blocksize>>>(srcArray[0], dstArray[0], scaleFactor, nx, ny, nz);
  

}


