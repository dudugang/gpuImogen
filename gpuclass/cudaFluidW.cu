#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h" // This defines the getGPUSourcePointers and makeGPUDestinationArrays utility functions

/* THIS FUNCTION

This function calculates a single half-step of the conserved transport part of the fluid equations
(CFD or MHD) which is used as the predictor input to the matching TVD function.

*/

__global__ void cukern_Wstep_mhd_uniform  (double *P, double *Cfreeze, double lambdaqtr, int nx);
__global__ void cukern_Wstep_hydro_uniform(double *P, double *Cfreeze, double lambdaqtr, int nx);

#define BLOCKLEN 60
#define BLOCKLENP2 62
#define BLOCKLENP4 64

__constant__ __device__ double *inputPointers[8];
__constant__ __device__ double *outputPointers[5];
__constant__ __device__ double fluidQtys[5];
#define FLUID_GAMMA   fluidQtys[0]
#define FLUID_GM1     fluidQtys[1]
#define FLUID_GG1     fluidQtys[2]
#define FLUID_MINMASS fluidQtys[3]
#define FLUID_MINEINT fluidQtys[4]

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  // Input and result
  if ((nrhs!=13) || (nlhs != 5)) mexErrMsgTxt("Wrong number of arguments: need [5] = cudaWflux(rho, E, px, py, pz, bx, by, bz, Ptot, c_f, lambda, purehydro?, fluid gamma)\n");

  cudaCheckError("entering cudaFluidW");

  ArrayMetadata amd;
  double **srcs = getGPUSourcePointers(prhs, &amd, 0, 9);
  double **dest = makeGPUDestinationArrays((int64_t *)mxGetData(prhs[0]),  plhs, 5);

  // Establish launch dimensions & a few other parameters
  int fluxDirection = 1;
  double lambda     = *mxGetPr(prhs[10]);

  dim3 arraySize;
  arraySize.x = amd.dim[0];
  arraySize.y = amd.dim[1];
  arraySize.z = amd.dim[2];

  dim3 blocksize, gridsize;

  int nu;
  // This bit is actually redundant now since arrays are always rotated so the fluid step is finite-differenced in the X direction
  blocksize.x = BLOCKLEN+4; blocksize.y = blocksize.z = 1;
  switch(fluxDirection) {
    case 1: // X direction flux: u = x, v = y, w = z;
      gridsize.x = arraySize.y;
      gridsize.y = arraySize.z;
      nu = gridsize.x;
      break;
    case 2: // Y direction flux: u = y, v = x, w = z
      gridsize.x = arraySize.x;
      gridsize.y = arraySize.z;
      nu = gridsize.y;
      break;
    case 3: // Z direction flux: u = z, v = x, w = y;
      gridsize.x = arraySize.x;
      gridsize.y = arraySize.y;
      nu = gridsize.z;
      break;
    }
  double *thermo = mxGetPr(prhs[12]);
  double gamma = thermo[0];
  double rhomin= thermo[1];
  double gamHost[5];
  gamHost[0] = gamma;
  gamHost[1] = gamma-1.0;
  gamHost[2] = gamma*(gamma-1.0);
  gamHost[3] = rhomin;
// assert     cs > cs_min
//     g P / rho > g rho_min^(g-1)
// (g-1) e / rho > rho_min^(g-1)
//             e > rho rho_min^(g-1)/(g-1)
  gamHost[4] = powl(rhomin, gamma-1.0)/(gamma-1.0);
// Even for gamma=5/3, soundspeed is very weakly dependent on density (cube root)

  hipMemcpyToSymbol(HIP_SYMBOL(fluidQtys), &gamHost[0], 5*sizeof(double), 0, hipMemcpyHostToDevice);

// It appears this is only used in the null step. It was used in a previous W step but that kernel was irreperably broken.

// If the dimension has finite extent, performs actual step; If not, blits input arrays to output arrays
// NOTE: this situation should not occur, since the flux routine itself skips singleton dimensions for 1- and 2-d sims.

int hydroOnly;
hydroOnly = (int)*mxGetPr(prhs[11]);
  
if(hydroOnly == 1) {
  hipMemcpyToSymbol(HIP_SYMBOL(inputPointers),  srcs, 5*sizeof(double *), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(outputPointers), dest, 5*sizeof(double *), 0, hipMemcpyHostToDevice);
  cukern_Wstep_hydro_uniform<<<gridsize, blocksize>>>(srcs[8], srcs[9], .25*lambda, arraySize.x);
  } else {
  hipMemcpyToSymbol(HIP_SYMBOL(inputPointers),  srcs, 8*sizeof(double *), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(outputPointers), dest, 5*sizeof(double *), 0, hipMemcpyHostToDevice);
  cukern_Wstep_mhd_uniform<<<gridsize, blocksize>>>(srcs[8], srcs[9], lambda/4.0, arraySize.x);
}

hipError_t epicFail = hipGetLastError();
if(epicFail != hipSuccess) cudaLaunchError(epicFail, blocksize, gridsize, &amd, hydroOnly, "fluid W step");

}

__global__ void cukern_Wstep_mhd_uniform(double *P, double *Cfreeze, double lambdaqtr, int nx)
{
double C_f, velocity;
double q_i[5];
double b_i[3];
double w_i;
__shared__ double fluxLR[2][BLOCKLENP4];

/* Step 0 - obligatory annoying setup stuff (ASS) */
int I0 = nx*(blockIdx.x + gridDim.x * blockIdx.y);
int Xindex = (threadIdx.x-2);
int Xtrack = Xindex;
Xindex += nx*(threadIdx.x < 2);

int x; /* = Xindex % nx; */
int i;
bool doIflux = (threadIdx.x > 1) && (threadIdx.x < BLOCKLEN+2);

/* Step 1 - calculate W values */
C_f = Cfreeze[blockIdx.x + gridDim.x * blockIdx.y];

while(Xtrack < nx+2) {
    x = I0 + (Xindex % nx);

    q_i[0] = inputPointers[0][x];
    q_i[1] = inputPointers[1][x];       /* So we avoid multiple loops */
    q_i[2] = inputPointers[2][x];      /* over them inside the flux loop */
    q_i[3] = inputPointers[3][x];
    q_i[4] = inputPointers[4][x];
    b_i[0] = inputPointers[5][x];
    b_i[1] = inputPointers[6][x];
    b_i[2] = inputPointers[7][x];
    velocity = q_i[2] / q_i[0];

    /* rho, E, px, py, pz going down */
    /* Iterate over variables to flux */
    for(i = 0; i < 5; i++) {
        switch(i) {
            case 0: w_i = q_i[2]; break;
            case 1: w_i = (velocity * (q_i[1] + P[x]) - b_i[0]*(q_i[2]*b_i[0]+q_i[3]*b_i[1]+q_i[4]*b_i[2])/q_i[0] ) ; break;
            case 2: w_i = (velocity * q_i[2] + P[x] - b_i[0]*b_i[0]); break;
            case 3: w_i = (velocity * q_i[3]        - b_i[0]*b_i[1]); break;
            case 4: w_i = (velocity * q_i[4]        - b_i[0]*b_i[2]); break;
            }

        /* Step 2 - decouple to L/R flux */
        fluxLR[0][threadIdx.x] = (C_f*q_i[i] - w_i); /* Left  going flux */
        fluxLR[1][threadIdx.x] = (C_f*q_i[i] + w_i); /* Right going flux */
        // NOTE: a 0.5 is eliminated here. THis requires lambda to be rescaled by .5 in launch.
        /* Step 4 - Perform flux and write to output array */
        __syncthreads();

       if( doIflux && (Xindex < nx) ) {
            // NOTE: a .5 is missing here also, so lambda must ultimately be divided by 4.
            outputPointers[i][x] = q_i[i] - lambdaqtr * ( fluxLR[0][threadIdx.x] - fluxLR[0][threadIdx.x+1] + \
                                                          fluxLR[1][threadIdx.x] - fluxLR[1][threadIdx.x-1]  ); 

            }

        __syncthreads();
        }

    Xindex += BLOCKLEN;
    Xtrack += BLOCKLEN;
    __syncthreads();
    }

}

#define FLUXLa_OFFSET 0
#define FLUXLb_OFFSET (BLOCKLENP4)
#define FLUXRa_OFFSET (2*(BLOCKLENP4))
#define FLUXRb_OFFSET (3*(BLOCKLEN+4))
__global__ void cukern_Wstep_hydro_uniform(double *P, double *Cfreeze, double lambdaqtr, int nx)
{
double C_f, velocity;
double q_i[3];
double w_i;
double velocity_half;
__shared__ double fluxArray[4*(BLOCKLENP4)];
__shared__ double freezeSpeed[BLOCKLENP4];
/*
__shared__ double fluxL_a[BLOCKLENP4];
__shared__ double fluxL_b[BLOCKLENP4];
__shared__ double fluxR_a[BLOCKLENP4];
__shared__ double fluxR_b[BLOCKLENP4];*/

freezeSpeed[threadIdx.x] = 0;

/* Step 0 - obligatory annoying setup stuff (ASS) */
int I0 = nx*(blockIdx.x + gridDim.x * blockIdx.y);
int Xindex = (threadIdx.x-2);
int Xtrack = Xindex;
Xindex += nx*(threadIdx.x < 2);

int x; /* = Xindex % nx; */
bool doIflux = (threadIdx.x > 1) && (threadIdx.x < BLOCKLEN+2);

/* Step 1 - calculate W values */
C_f = Cfreeze[blockIdx.x + gridDim.x * blockIdx.y];
double locPsq;
double locE;

while(Xtrack < nx+2) {
    x = I0 + (Xindex % nx);

/*rho    q_i[0] = inputPointers[0][x];  Preload these out here 
E    q_i[1] = inputPointers[1][x];  So we avoid multiple loops 
px    q_i[2] = inputPointers[2][x];  over them inside the flux loop 
 py   q_i[3] = inputPointers[3][x];
  pz  q_i[4] = inputPointers[4][x];*/
    q_i[0] = inputPointers[0][x];
    q_i[1] = inputPointers[2][x];
    q_i[2] = inputPointers[1][x];
    locPsq   = P[x];

    velocity = q_i[1] / q_i[0];

    #define FLUXA_DECOUPLE(i) fluxArray[FLUXLa_OFFSET+threadIdx.x] = q_i[i]*C_f - w_i; fluxArray[FLUXRa_OFFSET+threadIdx.x] = q_i[i]*C_f + w_i;
    #define FLUXB_DECOUPLE(i) fluxArray[FLUXLb_OFFSET+threadIdx.x] = q_i[i]*C_f - w_i; fluxArray[FLUXRb_OFFSET+threadIdx.x] = q_i[i]*C_f + w_i;
/*    #define FLUXA_DECOUPLE(i) fluxL_a[threadIdx.x] = q_i[i]*C_f - w_i; fluxR_a[threadIdx.x] = q_i[i]*C_f + w_i;
    #define FLUXB_DECOUPLE(i) fluxL_b[threadIdx.x] = q_i[i]*C_f - w_i; fluxR_b[threadIdx.x] = q_i[i]*C_f + w_i;*/

    #define FLUXA_DELTA lambdaqtr*(fluxArray[FLUXLa_OFFSET+threadIdx.x] - fluxArray[FLUXLa_OFFSET+threadIdx.x+1] + fluxArray[FLUXRa_OFFSET+threadIdx.x] - fluxArray[FLUXRa_OFFSET+threadIdx.x-1])
    #define FLUXB_DELTA lambdaqtr*(fluxArray[FLUXLb_OFFSET+threadIdx.x] - fluxArray[FLUXLb_OFFSET+threadIdx.x+1] + fluxArray[FLUXRb_OFFSET+threadIdx.x] - fluxArray[FLUXRb_OFFSET+threadIdx.x-1])
/*    #define FLUXA_DELTA lambdaqtr*(fluxL_a[threadIdx.x] - fluxL_a[threadIdx.x+1] + fluxR_a[threadIdx.x] - fluxR_a[threadIdx.x-1])
    #define FLUXB_DELTA lambdaqtr*(fluxL_b[threadIdx.x] - fluxL_b[threadIdx.x+1] + fluxR_b[threadIdx.x] - fluxR_b[threadIdx.x-1])*/

    w_i = velocity*(q_i[2]+locPsq); /* E flux = v*(E+P) */
    FLUXA_DECOUPLE(2)
    w_i = (velocity*q_i[1] + locPsq); /* px flux = v*px + P */
    FLUXB_DECOUPLE(1)
    __syncthreads();
    if(doIflux && (Xindex < nx)) {
        locE = q_i[2] - FLUXA_DELTA; /* Calculate Ehalf */
        velocity_half = locPsq = q_i[1] - FLUXB_DELTA; /* Calculate Pxhalf */
        outputPointers[2][x] = locPsq; /* store pxhalf */
        }
    __syncthreads();

    locPsq *= locPsq; /* store p^2 in locPsq */

    q_i[0] = inputPointers[3][x];
    q_i[2] = inputPointers[4][x];
    w_i = velocity*q_i[0]; /* py flux = v*py */
    FLUXA_DECOUPLE(0)
    w_i = velocity*q_i[2]; /* pz flux = v pz */
    FLUXB_DECOUPLE(2)
    __syncthreads();
    if(doIflux && (Xindex < nx)) {
        q_i[0] -= FLUXA_DELTA;
        locPsq += q_i[0]*q_i[0];
        outputPointers[3][x] = q_i[0];
        q_i[2] -= FLUXB_DELTA;
        locPsq += q_i[2]*q_i[2]; /* Finished accumulating p^2 */
        outputPointers[4][x] = q_i[2];
        }
    __syncthreads();

    q_i[0] = inputPointers[0][x];
    w_i = q_i[1]; /* rho flux = px */
    FLUXA_DECOUPLE(0)
    __syncthreads();
    if(doIflux && (Xindex < nx)) {
        q_i[0] -= FLUXA_DELTA; /* Calculate rho_half */
//      outputPointers[0][x] = q_i[0];
        q_i[0] = (q_i[0] < FLUID_MINMASS) ? FLUID_MINMASS : q_i[0]; /* Enforce minimum mass density */
        outputPointers[0][x] = q_i[0];

        velocity_half /= q_i[0]; /* calculate velocity at the halfstep */


//      outputPointers[1][x] = locE; /* store total energy: We need to correct this for negativity shortly */
        locPsq = (locE - .5*(locPsq/q_i[0])); /* Calculate epsilon = E - T */
//      P[x] = FLUID_GM1*locPsq; /* Calculate P = (gamma-1) epsilon */

// For now we have to store the above before fixing them so the original freezeAndPtot runs unperturbed
// but assert the corrected P, C_f values below to see what we propose to do.
// it should match the freezeAndPtot very accurately.

// assert   cs^2 > cs^2(rho minimum)
//     g P / rho > g rho_min^(g-1) under polytropic EOS
//g(g-1) e / rho > g rho_min^(g-1)
//             e > rho rho_min^(g-1)/(g-1) = rho FLUID_MINEINT
        if(locPsq < q_i[0]*FLUID_MINEINT) {
          locE = locE - locPsq + q_i[0]*FLUID_MINEINT; // Assert minimum E = T + epsilon_min
          locPsq = q_i[0]*FLUID_MINEINT; // store minimum epsilon.
          } /* Assert minimum temperature */

        P[x] = FLUID_GM1*locPsq; /* Calculate P = (gamma-1) epsilon */
        outputPointers[1][x] = locE; /* store total energy: We need to correct this for negativity shortly */

        /* calculate local freezing speed */
        locPsq = abs(velocity_half) + sqrt(FLUID_GG1*locPsq/q_i[0]);
        if(locPsq > freezeSpeed[threadIdx.x]) freezeSpeed[threadIdx.x] = locPsq;
        }

    Xindex += BLOCKLEN;
    Xtrack += BLOCKLEN;
    __syncthreads();
    }

/* We have a block of 64 threads. Fold this shit in */

if(threadIdx.x > 32) return;

if(freezeSpeed[threadIdx.x+32] > freezeSpeed[threadIdx.x]) freezeSpeed[threadIdx.x] = freezeSpeed[threadIdx.x+32];
__syncthreads();
if(threadIdx.x > 16) return;

if(freezeSpeed[threadIdx.x+16] > freezeSpeed[threadIdx.x]) freezeSpeed[threadIdx.x] = freezeSpeed[threadIdx.x+16];
__syncthreads();
if(threadIdx.x > 8) return;

if(freezeSpeed[threadIdx.x+8] > freezeSpeed[threadIdx.x]) freezeSpeed[threadIdx.x] = freezeSpeed[threadIdx.x+8];
__syncthreads();
if(threadIdx.x > 4) return;

if(freezeSpeed[threadIdx.x+4] > freezeSpeed[threadIdx.x]) freezeSpeed[threadIdx.x] = freezeSpeed[threadIdx.x+4];
__syncthreads();
if(threadIdx.x > 2) return;

if(freezeSpeed[threadIdx.x+2] > freezeSpeed[threadIdx.x]) freezeSpeed[threadIdx.x] = freezeSpeed[threadIdx.x+2];
__syncthreads();
if(threadIdx.x > 1) return;
/*if(threadIdx.x > 0) return;
for(x = 0; x < BLOCKLENP4; x++) { if(freezeSpeed[x] > freezeSpeed[0]) freezeSpeed[0] = freezeSpeed[x]; }
Cfreeze[blockIdx.x + gridDim.x * blockIdx.y] = freezeSpeed[0];*/

Cfreeze[blockIdx.x + gridDim.x * blockIdx.y] = (freezeSpeed[1] > freezeSpeed[0]) ? freezeSpeed[1] : freezeSpeed[0];

}

