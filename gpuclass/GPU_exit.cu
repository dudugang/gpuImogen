#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  // wrapper for hipFree().
  if((nlhs != 0) || (nrhs != 0)) mexErrMsgTxt("Error: syntax is GPU_exit() to deactivate current CUDA device");

  hipDeviceReset();

  return;
}
