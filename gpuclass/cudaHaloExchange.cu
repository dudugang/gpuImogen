#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"
#include "mpi.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cudaCommon.h"
#include "parallel_halo_arrays.h"

/* THIS ROUTINE
   This routine interfaces with the parallel gateway halo routines
   */

/* X halo routines */
/* These are the suck; We have to grab 24-byte wide chunks en masse */
/* Fork ny by nz threads to do the job */
__global__ void cukern_HaloXToLinearL(double *mainarray, double *linarray, int nx);
__global__ void cukern_LinearToHaloXL(double *mainarray, double *linarray, int nx);
__global__ void cukern_HaloXToLinearR(double *mainarray, double *linarray, int nx);
__global__ void cukern_LinearToHaloXR(double *mainarray, double *linarray, int nx);

/* Y halo routines */
/* We grab an X-Z plane, making it easy to copy N linear strips of memory */
/* Fork off nz by 3 blocks to do the job */
__global__ void cukern_HaloYToLinearL(double *mainarray, double *linarray, int nx, int ny);
__global__ void cukern_LinearToHaloYL(double *mainarray, double *linarray, int nx, int ny);

__global__ void cukern_HaloYToLinearR(double *mainarray, double *linarray, int nx, int ny);
__global__ void cukern_LinearToHaloYR(double *mainarray, double *linarray, int nx, int ny);

/* Z halo routines */
/* The easiest; We make one copy of an Nx by Ny by 3 slab of memory */
/* No kernels necessary, we can simply memcpy our hearts out */

pParallelTopology topoStructureToC(const mxArray *prhs);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
/* Functional form:
    cudaHaloExchange(arraytag, [orientation 3x1], dimension_to_exchange, parallel topology information)

    1. get neighbors from halo library in dimension_to_exchange direction
    2. determine which memory direction that currently is
    3. If it's x or y, rip it to a linear array
    4. Aquire some host-pinned memory and dump to that
    5. pass that host pointer to halo_exchange
    6. wait for MPI to return control
*/
  if (nrhs!=5) mexErrMsgTxt("call form is cudaHaloExchange(arraytag, [3x1 orientation], dimension_to_xchg, topology, circularity\n");
  if(mxGetNumberOfElements(prhs[1]) != 3) mexErrMsgTxt("2nd argument must be a 3-element array\n");

  CHECK_CUDA_ERROR("entering cudaHaloExchange");

  int xchg = (int)*mxGetPr(prhs[2]) - 1;
  int orient[3];

  pParallelTopology parallelTopo = topoStructureToC(prhs[3]);

  if(parallelTopo->nproc[xchg] == 1) return;
  // Do not waste time if we can't possibly have any work to do

  ArrayMetadata amd;
  double **array = getGPUSourcePointers(prhs, &amd, 0,0);

  int ctr;
  for(ctr = 0; ctr < 3; ctr++) { orient[ctr] = (int)*(mxGetPr(prhs[1]) + ctr); }
//printf("orient: %i %i %i\n", orient[0], orient[1], orient[2]); fflush(stdout);

  int memDimension = orient[xchg]-1; // The actual in-memory direction we're gonna be exchanging
 
  // get # of transverse elements, *3 deep for halo
  int numToExchange = 3 * amd.numel / amd.dim[memDimension];
//printf("numel: %i, dim[dimension]: %i; #2xchg: %i\n", amd.numel, amd.dim[memDimension], numToExchange);

  hipError_t fail = hipGetLastError(); // Clear the error register
  double *pinnedMem[4];
  double *devPMptr[4];

  if(xchg+1 > parallelTopo->ndim) return; // The topology does not extend in this dimension
  if(parallelTopo->nproc[xchg] == 1) return; // Only 1 block in this direction.

  /* Be told if the left and right sides of the dimension are circular or not */
  double *interior = mxGetPr(prhs[4]);
  int leftCircular  = (int)interior[2*memDimension];
  int rightCircular = (int)interior[2*memDimension+1];

#ifdef NO_PINNEDMEM
  for(ctr = 0; ctr < 4; ctr++) {
    pinnedMem[ctr] = (double *)malloc(numToExchange * sizeof(double));
    fail = hipMalloc((double **)&devPMptr[ctr], numToExchange * sizeof(double));
    CHECK_CUDA_ERROR("cudaHaloExchange memory alloc w/o pinned memory");
    }
#else 
  for(ctr = 0; ctr < 4; ctr++) {
    fail = hipHostAlloc(&pinnedMem[ctr], numToExchange * sizeof(double), hipHostMallocDefault);
    CHECK_CUDA_ERROR("cudahaloExchange pinned memory alloc");
    fail = hipHostGetDevicePointer((void **)&devPMptr[ctr], (void *)pinnedMem[ctr], 0);
    }
#endif

  if(fail != hipSuccess) { dim3 f; CHECK_CUDA_LAUNCH_ERROR(f, f, &amd, ctr, "cudaHaloExchange.malloc"); }

  MPI_Comm commune = MPI_Comm_f2c(parallelTopo->comm);

  switch(memDimension) {
    case 0: { /* X halo arrangement */
      dim3 gridsize; gridsize.x = amd.dim[1]; gridsize.y = amd.dim[2]; gridsize.z = 1;
      dim3 blocksize; blocksize.x = 3; blocksize.y = 1; blocksize.z = 1; // This is horrible.

      cukern_HaloXToLinearL<<<gridsize, blocksize>>>(array[0], devPMptr[0], amd.dim[0]);
      CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, &amd, 0, "cudaHaloExchange.X_left_read");
      cukern_HaloXToLinearR<<<gridsize, blocksize>>>(array[0], devPMptr[1], amd.dim[0]);
      CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, &amd, 0, "cudaHaloExchange.X_right_read");

      #ifdef NO_PINNEDMEM
      hipMemcpy(pinnedMem[0], devPMptr[0], numToExchange*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(pinnedMem[1], devPMptr[1], numToExchange*sizeof(double), hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR("cudaHaloExchange X, NO_PINNEDMEM, D2H");
      #endif 
      hipDeviceSynchronize(); 
      parallel_exchange_dim_contig(parallelTopo, 0, pinnedMem[0], pinnedMem[1], pinnedMem[2], pinnedMem[3], numToExchange, MPI_DOUBLE);
MPI_Barrier(MPI_COMM_WORLD);
hipDeviceSynchronize();
      #ifdef NO_PINNEDMEM
      hipMemcpy(devPMptr[2], pinnedMem[2], numToExchange*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(devPMptr[3], pinnedMem[3], numToExchange*sizeof(double), hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR("cudaHaloExchange X, NO_PINNEDMEM, H2D");
      #endif

hipDeviceSynchronize();

      if(leftCircular) {
        cukern_LinearToHaloXL<<<gridsize, blocksize>>>(array[0], devPMptr[2], amd.dim[0]);
        CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, &amd, 0, "cudaHaloExchange.X_left_write");
        }
      if(rightCircular) {
        cukern_LinearToHaloXR<<<gridsize, blocksize>>>(array[0], devPMptr[3], amd.dim[0]);
        CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, &amd, 0, "cudaHaloExchange.X_right_write");
        }
      }; break;

    case 1: { /* Y halo arrangement */
      dim3 blocksize; blocksize.x = 256; blocksize.y = 1; blocksize.z = 1;
      dim3 gridsize; gridsize.x = amd.dim[0]/256; gridsize.y = amd.dim[2]; gridsize.z = 1;
      gridsize.x += gridsize.x*256 < amd.dim[0] ? 1 : 0;

      cukern_HaloYToLinearL<<<gridsize, blocksize>>>(array[0], devPMptr[0], amd.dim[0], amd.dim[1]);
      CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, &amd, 0, "cudaHaloExchange.Y_left_read");
      cukern_HaloYToLinearR<<<gridsize, blocksize>>>(array[0], devPMptr[1], amd.dim[0], amd.dim[1]);
      CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, &amd, 0, "cudaHaloExchange.Y_right_read");

      #ifdef NO_PINNEDMEM
      hipMemcpy(pinnedMem[0], devPMptr[0], numToExchange*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(pinnedMem[1], devPMptr[1], numToExchange*sizeof(double), hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR("cudaHaloExchange Y, NO_PINNEDMEM, D2H");
      #endif

      hipDeviceSynchronize();
      parallel_exchange_dim_contig(parallelTopo, 1, pinnedMem[0], pinnedMem[1], pinnedMem[2], pinnedMem[3], numToExchange, MPI_DOUBLE);

      #ifdef NO_PINNEDMEM
      hipMemcpy(devPMptr[2], pinnedMem[2], numToExchange*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(devPMptr[3], pinnedMem[3], numToExchange*sizeof(double), hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR("cudaHaloExchange Y, NO_PINNEDMEM, H2D");
      #endif

      if(leftCircular) {
        cukern_LinearToHaloYL<<<gridsize, blocksize>>>(array[0], devPMptr[2], amd.dim[0], amd.dim[1]);
        CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, &amd, 0, "cudaHaloExchange.Y_left_write");
        }
      if(rightCircular) {
        cukern_LinearToHaloYR<<<gridsize, blocksize>>>(array[0], devPMptr[3], amd.dim[0], amd.dim[1]);
        CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, &amd, 0, "cudaHaloExchange.Y_right_write");
        }
      }; break;

    case 2: { /* Z halo arrangement */
      dim3 gridsize, blocksize;
      hipMemcpy(pinnedMem[0], array[0], numToExchange*sizeof(double), hipMemcpyDeviceToHost);
      CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, &amd, 0, "cudaHaloExchange.Z_left_readmemcpy");
      hipMemcpy(pinnedMem[1], &array[0][amd.numel - numToExchange], numToExchange*sizeof(double), hipMemcpyDeviceToHost);
      CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, &amd, 0, "cudaHaloExchange.Z_right_readmemcpy");

      #ifdef NO_PINNEDMEM
      hipMemcpy(pinnedMem[0], devPMptr[0], numToExchange*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(pinnedMem[1], devPMptr[1], numToExchange*sizeof(double), hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR("cudaHaloExchange Z, NO_PINNEDMEM, D2H");
      #endif

      hipDeviceSynchronize();
      parallel_exchange_dim_contig(parallelTopo, 2, pinnedMem[0], pinnedMem[1], pinnedMem[2], pinnedMem[3], numToExchange, MPI_DOUBLE);

      #ifdef NO_PINNEDMEM
      hipMemcpy(devPMptr[2], pinnedMem[2], numToExchange*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(devPMptr[3], pinnedMem[3], numToExchange*sizeof(double), hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR("cudaHaloExchange Z, NO_PINNEDMEM, H2D");
      #endif

      if(leftCircular) {
        hipMemcpy(array[0], pinnedMem[2], numToExchange*sizeof(double), hipMemcpyHostToDevice);
        CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, &amd, 0, "cudaHaloExchange.Z_left_writememcpy");
        }
      if(rightCircular) {
        hipMemcpy(&array[0][amd.numel - numToExchange], pinnedMem[3], numToExchange*sizeof(double), hipMemcpyDeviceToHost);
        CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, &amd, 0, "cudaHaloExchange.Z_right_writememcpy");
        }
      }; break;
    }

#ifdef NO_PINNEDMEM
  for(ctr = 0; ctr < 4; ctr++) {
    free(pinnedMem[ctr]);
    hipFree(devPMptr[ctr]);
    }
#else
  for(ctr = 0; ctr < 4; ctr++) hipHostFree(pinnedMem[ctr]);
#endif

hipError_t epicFail = hipDeviceSynchronize();
CHECK_CUDA_ERROR("cudaHaloExchange: post exchange device sync");
/*if(epicFail != hipSuccess) CHECK_CUDA_LAUNCH_ERROR(epicFail, 256, 128, &amd, memDimension, "halo exchange");*/

free(parallelTopo);
}

/* X halo routines */
/* These are the suck; We have to grab 24-byte wide chunks en masse */
/* Fork ny by nz threads to do the job */
/* We trust the optimizing compiler to clean up the first two lines of sanity-maintainence for us */

/* Copies mainarray(3:5,:,:) to linarray, indexed from 0 */
__global__ void cukern_HaloXToLinearL(double *mainarray, double *linarray, int nx)
{
int myx = threadIdx.x; int myy = blockIdx.x; int myz = blockIdx.y;
int ny = gridDim.x;

int addr = (myx + 3) + (myy + myz*ny)*nx;
int linAddr = myx + 3*(myy + ny*myz);

linarray[linAddr] = mainarray[addr];
}

/* Copies mainarray( (nx-6):(nx-4), :,:) to linarray, indexed from 0 */
__global__ void cukern_HaloXToLinearR(double *mainarray, double *linarray, int nx)
{
int myx = threadIdx.x; int myy = blockIdx.x; int myz = blockIdx.y;
int ny = gridDim.x; 

int addr = (nx - 6 + myx) + (myy + myz*ny)*nx;
int linAddr = myx + 3*(myy + ny * myz);

linarray[linAddr] = mainarray[addr];
}

/* Copies linarray to mainarray(0:2,:,:) */
__global__ void cukern_LinearToHaloXL(double *mainarray, double *linarray, int nx)
{
int myx = threadIdx.x; int myy = blockIdx.x; int myz = blockIdx.y;
int ny = gridDim.x;

int addr = (myx) + (myy + myz*ny)*nx;
int linAddr = myx + 3*(myy + ny * myz);

mainarray[addr] = linarray[linAddr];
}

/* Copies linarray to mainarray((nx-3):(nx-1),:,:) */
__global__ void cukern_LinearToHaloXR(double *mainarray, double *linarray, int nx)
{
int myx = threadIdx.x; int myy = blockIdx.x; int myz = blockIdx.y;
int ny = gridDim.x;


int addr = (nx - 3 + myx) + (myy + myz*ny)*nx;
int linAddr = myx + 3*(myy + ny * myz);

mainarray[addr] = linarray[linAddr];
}

/* Y halo routines */
/* We grab an X-Z plane, making it easy to copy N linear strips of memory */
/* Fork off nz by 3 blocks to do the job */

/* Fork enough threads to cover the X direction , and ny blocks in the y dir  */
__global__ void cukern_HaloYToLinearL(double *mainarray, double *linarray, int nx, int ny)
{
int myx = threadIdx.x + blockDim.x * blockIdx.x;
int myz = blockIdx.y;
int nz = gridDim.y;

if(myx >= nx) return;

int addr = myx + nx*ny*myz + 3*nx;
int linAddr = myx + nx*myz;

int ctz;
for(ctz = 0; ctz < 3; ctz++) {
  linarray[linAddr] = mainarray[addr];
  addr += nx;
  linAddr += nx*nz;
  }
}

__global__ void cukern_LinearToHaloYL(double *mainarray, double *linarray, int nx, int ny)
{
int myx = threadIdx.x + blockDim.x * blockIdx.x;
int myz = blockIdx.y;
int nz = gridDim.y;

if(myx >= nx) return; 

int addr = myx + nx*ny*myz;
int linAddr = myx + nx*myz;

int ctz;
for(ctz = 0; ctz < 3; ctz++) {
  mainarray[addr] = linarray[linAddr];
  addr += nx;
  linAddr += nx*nz;
  }

}

__global__ void cukern_HaloYToLinearR(double *mainarray, double *linarray, int nx, int ny)
{
int myx = threadIdx.x + blockDim.x * blockIdx.x;
int myz = blockIdx.y;
int nz = gridDim.y;

if(myx >= nx) return;

int addr = myx + nx*ny*myz + (ny-6)*nx;
int linAddr = myx + nx*myz;

int ctz;
for(ctz = 0; ctz < 3; ctz++) {
  linarray[linAddr] = mainarray[addr];
  addr += nx;
  linAddr += nx*nz;
  }


}

__global__ void cukern_LinearToHaloYR(double *mainarray, double *linarray, int nx, int ny)
{
int myx = threadIdx.x + blockDim.x * blockIdx.x;
int myz = blockIdx.y;
int nz = gridDim.y;

if(myx >= nx) return;

int addr = myx + nx*ny*myz + (ny-3)*nx;
int linAddr = myx + nx*myz;

int ctz;
for(ctz = 0; ctz < 3; ctz++) {
  mainarray[addr] = linarray[linAddr];
  addr += nx;
  linAddr += nx*nz;
  }

}

pParallelTopology topoStructureToC(const mxArray *prhs)
{
mxArray *a;

pParallelTopology pt = (pParallelTopology)malloc(sizeof(ParallelTopology));

a = mxGetFieldByNumber(prhs,0,0);
pt->ndim = (int)*mxGetPr(a);
a = mxGetFieldByNumber(prhs,0,1);
pt->comm = (int)*mxGetPr(a);

int *val;
int i;

val = (int *)mxGetData(mxGetFieldByNumber(prhs,0,2));
for(i = 0; i < pt->ndim; i++) pt->coord[i] = val[i];

val = (int *)mxGetData(mxGetFieldByNumber(prhs,0,3));
for(i = 0; i < pt->ndim; i++) pt->neighbor_left[i] = val[i];

val = (int *)mxGetData(mxGetFieldByNumber(prhs,0,4));
for(i = 0; i < pt->ndim; i++) pt->neighbor_right[i] = val[i];

val = (int *)mxGetData(mxGetFieldByNumber(prhs,0,5));
for(i = 0; i < pt->ndim; i++) pt->nproc[i] = val[i];

return pt;

}
