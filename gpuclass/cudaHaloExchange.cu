#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"
#include "mpi.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

// PGW
#include "parallel_halo_arrays.h"

// My stuff
#include "cudaCommon.h"
#include "compiled_common.h"
#include "cudaHaloExchange.h"

/* THIS ROUTINE
   This routine interfaces with the parallel gateway halo routines
   The N MGArrays *ed to by phi swap ghost cells as described by topo
   circularity[
 */

#ifdef STANDALONE_MEX_FUNCTION
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	/* Functional form:
    cudaHaloExchange(arraytag, dimension_to_exchange, parallel topology information, circularity)

    1. get neighbors from halo library in dimension_to_exchange direction
    2. determine which memory direction that currently is
    3. If it's x or y, rip it to a linear array
    4. Aquire some host-pinned memory and dump to that
    5. pass that host pointer to halo_exchange
    6. wait for MPI to return control
	 */
	if (nrhs!=4) mexErrMsgTxt("call form is cudaHaloExchange(arraytag, dimension_to_xchg, topology, circularity).\n");

	CHECK_CUDA_ERROR("entering cudaHaloExchange");
	int xchg = (int)*mxGetPr(prhs[1]) - 1;

	pParallelTopology parallelTopo = topoStructureToC(prhs[2]);

	if(parallelTopo->nproc[xchg] == 1) return;
	// Do not waste time if we can't possibly have any work to do

	MGArray phi;
	int worked = MGA_accessMatlabArrays(prhs, 0, 0, &phi);

	exchange_MPI_Halos(&phi, 1, parallelTopo, xchg);
}
#endif

int exchange_MPI_Halos(MGArray *phi, int nArrays, pParallelTopology topo, int xchgDir)
{
	CHECK_CUDA_ERROR("entering exchange_MPI_Halos");
	xchgDir -= 1; // Convert 1-2-3 index into 0-1-2 memory index

	// Avoid wasting time...
	if(xchgDir+1 > topo->ndim) return ERROR_NULL_OPERATION;
	if(topo->nproc[xchgDir] == 1) return ERROR_NULL_OPERATION;

	int memDir;

	int i;
	for(i = 0; i < nArrays; i++) {
		memDir = phi->currentPermutation[xchgDir]; // The actual in-memory direction we're gonna be exchanging

		double *ptrHalo;

		/* Be told if the left and right sides of the dimension are circular or not */
		int leftCircular, rightCircular;
		switch(xchgDir) {
		case 0:
			leftCircular = (phi->circularBoundaryBits & MGA_BOUNDARY_XMINUS) ? 1 : 0;
			rightCircular = (phi->circularBoundaryBits & MGA_BOUNDARY_XPLUS) ? 1 : 0;
			break;
		case 1:
			leftCircular = (phi->circularBoundaryBits & MGA_BOUNDARY_YMINUS) ? 1 : 0;
			rightCircular = (phi->circularBoundaryBits & MGA_BOUNDARY_YPLUS) ? 1 : 0;
			break;

		case 2:
			leftCircular = (phi->circularBoundaryBits & MGA_BOUNDARY_YMINUS) ? 1 : 0;
			rightCircular = (phi->circularBoundaryBits & MGA_BOUNDARY_YPLUS) ? 1 : 0;
			break;
		}

		// Find the size of the swap buffer
		int numPerHalo = MGA_wholeFaceHaloNumel(phi, memDir, 3);

		hipError_t fail = hipHostMalloc((void **)&ptrHalo, 4*numPerHalo*sizeof(double));
		CHECK_CUDA_ERROR("hipHostAlloc");

		MPI_Comm commune = MPI_Comm_f2c(topo->comm);

		double *ptmp = ptrHalo;
		// Fetch left face
		if(leftCircular)
			MGA_wholeFaceToLinear(phi, memDir, 0, 0, 3, &ptmp);

		ptmp = ptrHalo + numPerHalo;
		// Fetch right face
		if(rightCircular)
			MGA_wholeFaceToLinear(phi, memDir, 1, 0, 3, &ptmp);

		// synchronize to make sure host sees what was uploaded
		int j;
		for(j = 0; j < phi->nGPUs; j++) {
			hipSetDevice(phi->deviceID[j]);
			hipDeviceSynchronize();
		}

		parallel_exchange_dim_contig(topo, xchgDir, (void*)ptrHalo,
				(void*)(ptrHalo + numPerHalo),\
				(void*)(ptrHalo+2*numPerHalo),\
				(void*)(ptrHalo+3*numPerHalo), numPerHalo, MPI_DOUBLE);
		MPI_Barrier(MPI_COMM_WORLD);

		// write left face
		ptmp = ptrHalo + 2*numPerHalo;
		if(leftCircular)
			MGA_wholeFaceToLinear(phi, memDir, 0, 1, 3, &ptmp);

		ptmp = ptrHalo + 3*numPerHalo;
		// Fetch right face
		if(rightCircular)
			MGA_wholeFaceToLinear(phi, memDir, 1, 1, 3, &ptmp);

		hipHostFree((void **)ptrHalo);

		// Move to the next array to exchange
		phi++;
	}

	return SUCCESSFUL;

}



