#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"
#include "mpi.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

// PGW
#include "parallel_halo_arrays.h"

// My stuff
#include "cudaCommon.h"
#include "cudaHaloExchange.h"

/* THIS ROUTINE
   This routine interfaces with the parallel gateway halo routines
   The N MGArrays *ed to by phi swap ghost cells as described by topo
   circularity[...

 */

#ifdef STANDALONE_MEX_FUNCTION
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	/* Functional form:
    cudaHaloExchange(arraytag, dimension_to_exchange, parallel topology information, circularity)

    1. get neighbors from halo library in dimension_to_exchange direction
    2. determine which memory direction that currently is
    3. If it's x or y, rip it to a linear array
    4. Aquire some host-pinned memory and dump to that
    5. pass that host pointer to halo_exchange
    6. wait for MPI to return control
	 */
	if (nrhs!=4) mexErrMsgTxt("call form is cudaHaloExchange(arraytag, dimension_to_xchg, topology, circularity).\n");

	CHECK_CUDA_ERROR("entering cudaHaloExchange");
	int xchg = (int)*mxGetPr(prhs[1]);

	pParallelTopology parallelTopo = topoStructureToC(prhs[2]);

	if(parallelTopo->nproc[xchg] == 1) return;
	// Do not waste time if we can't possibly have any work to do

	MGArray phi;
	int worked = MGA_accessMatlabArrays(prhs, 0, 0, &phi);
	if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) { DROP_MEX_ERROR("Failed to access GPU array."); }

	if(CHECK_IMOGEN_ERROR(exchange_MPI_Halos(&phi, 1, parallelTopo, xchg)) != SUCCESSFUL) {
		DROP_MEX_ERROR("Failed to perform MPI halo exchange!");
	}
}
#endif

int exchange_MPI_Halos(MGArray *phi, int nArrays, pParallelTopology topo, int xchgDir)
{
	int returnCode = CHECK_CUDA_ERROR("entering exchange_MPI_Halos");
	if(returnCode != SUCCESSFUL) { return returnCode; }

	xchgDir -= 1; // Convert 1-2-3 index into 0-1-2 memory index

	// Avoid wasting time...
	if(xchgDir+1 > topo->ndim) return SUCCESSFUL;
	if(topo->nproc[xchgDir] == 1) return SUCCESSFUL;

	int memDir;

	int i;
	for(i = 0; i < nArrays; i++) {
		/* Be told if the left and right sides of the dimension are circular or not */
		int leftCircular, rightCircular;
		switch(xchgDir) {
		case 0:
			leftCircular = (phi->circularBoundaryBits & MGA_BOUNDARY_XMINUS) ? 1 : 0;
			rightCircular = (phi->circularBoundaryBits & MGA_BOUNDARY_XPLUS) ? 1 : 0;
			break;
		case 1:
			leftCircular = (phi->circularBoundaryBits & MGA_BOUNDARY_YMINUS) ? 1 : 0;
			rightCircular = (phi->circularBoundaryBits & MGA_BOUNDARY_YPLUS) ? 1 : 0;
			break;

		case 2:
			leftCircular = (phi->circularBoundaryBits & MGA_BOUNDARY_YMINUS) ? 1 : 0;
			rightCircular = (phi->circularBoundaryBits & MGA_BOUNDARY_YPLUS) ? 1 : 0;
			break;
		default:
			int mpirank;
			MPI_Comm_rank(MPI_COMM_WORLD, &mpirank);
			printf("============= FAULT IN COMPILED CODE: RANK %i\nValid exchange directions are 1/2/3\nI was called with %i\n=================\n", xchgDir + 1);
			return ERROR_INVALID_ARGS;
		}

		memDir = phi->currentPermutation[xchgDir]; // The actual in-memory direction we're gonna be exchanging

		double *ptrHalo;

		// Find the size of the swap buffer
		int numPerHalo = MGA_wholeFaceHaloNumel(phi, memDir, 3);

		hipHostMalloc((void **)&ptrHalo, 4*numPerHalo*sizeof(double));
		returnCode = CHECK_CUDA_ERROR("hipHostAlloc");
		if(returnCode != SUCCESSFUL) return returnCode;

		MPI_Comm commune = MPI_Comm_f2c(topo->comm);

		double *ptmp = ptrHalo;
		// Fetch left face
		if(leftCircular)
			returnCode = MGA_wholeFaceToLinear(phi, memDir, 0, 0, 3, &ptmp);
		if(returnCode != SUCCESSFUL) return CHECK_IMOGEN_ERROR(returnCode);


		ptmp = ptrHalo + numPerHalo;
		// Fetch right face
		if(rightCircular)
			returnCode = MGA_wholeFaceToLinear(phi, memDir, 1, 0, 3, &ptmp);
		if(returnCode != SUCCESSFUL) return CHECK_IMOGEN_ERROR(returnCode);

		// synchronize to make sure host sees what was uploaded
		int j;
		for(j = 0; j < phi->nGPUs; j++) {
			hipSetDevice(phi->deviceID[j]);
			hipDeviceSynchronize();
		}

		parallel_exchange_dim_contig(topo, xchgDir, (void*)ptrHalo,
				(void*)(ptrHalo + numPerHalo),\
				(void*)(ptrHalo+2*numPerHalo),\
				(void*)(ptrHalo+3*numPerHalo), numPerHalo, MPI_DOUBLE);
		MPI_Barrier(MPI_COMM_WORLD);

		// write left face
		ptmp = ptrHalo + 2*numPerHalo;
		if(leftCircular)
			returnCode = MGA_wholeFaceToLinear(phi, memDir, 0, 1, 3, &ptmp);
		if(returnCode != SUCCESSFUL) return CHECK_IMOGEN_ERROR(returnCode);

		ptmp = ptrHalo + 3*numPerHalo;
		// Fetch right face
		if(rightCircular)
			returnCode = MGA_wholeFaceToLinear(phi, memDir, 1, 1, 3, &ptmp);
		if(returnCode != SUCCESSFUL) return CHECK_IMOGEN_ERROR(returnCode);

		hipHostFree((void **)ptrHalo);

		// Move to the next array to exchange
		phi++;
	}

	return SUCCESSFUL;

}



