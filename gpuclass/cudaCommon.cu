#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"


/* Given the RHS, an array to return array size, and the set of array indexes to take *s from */
double **getGPUSourcePointers(const mxArray *prhs[], ArrayMetadata *metaReturn, int fromarg, int toarg)
{

  double **gpuPointers = (double **)malloc((1+toarg-fromarg) * sizeof(double *));
  int iter;

  mxClassID dtype;

  dtype = mxGetClassID(prhs[fromarg]);
  if(dtype != mxINT64_CLASS) mexErrMsgTxt("cudaCommon: fatal, tried to get gpu src pointer from something not a gpu tag.");

  int64_t *dims = (int64_t *)mxGetData(prhs[fromarg]);
  for(iter = 0; iter < 3; iter++) { metaReturn->dim[iter] = (int)dims[2+iter]; } // copy metadata out of first gpu*
  metaReturn->numel = metaReturn->dim[0]*metaReturn->dim[1]*metaReturn->dim[2];
  metaReturn->ndims = dims[1];

  for(iter = fromarg; iter <= toarg; iter++) {
     dtype = mxGetClassID(prhs[iter]);
    if(dtype != mxINT64_CLASS) {
      printf("For argument %i\n",iter);
      mexErrMsgTxt("cudaCommon: fatal, tried to get gpu src pointer from something not a gpu tag.");
      }

    dims = (int64_t *)mxGetData(prhs[iter]);
    gpuPointers[iter-fromarg] = (double *)dims[0];
  }

return gpuPointers;
}

/* Creates destination array that the kernels write to; Returns the GPU memory pointer, and assigns the LHS it's passed */
double **makeGPUDestinationArrays(int64_t *reference, mxArray *retArray[], int howmany)
{

double **rvals = (double **)malloc(howmany*sizeof(double *));
int i;
mwSize dims[2]; dims[0] = 5; dims[1] = 1;

int64_t *rv; size_t numel;

numel = reference[2]*reference[3]*reference[4];

for(i = 0; i < howmany; i++) {
  retArray[i] = mxCreateNumericArray(2, dims, mxINT64_CLASS, mxREAL);
  rv = (int64_t *)mxGetData(retArray[i]);

  hipError_t fail = hipMalloc((void **)&rv[0], numel*sizeof(double));
  if(fail != hipSuccess) {
    printf("On array %i/%i: %s\n", i+1, howmany, hipGetErrorString(fail));
    cudaCheckError("In makeGPUDestinationArrays: malloc failed and I am sad.");
    }

  int q; for(q = 1; q < 5; q++) rv[q] = reference[q];
  rvals[i] = (double *)rv[0];
  }

//size_t tot,fre;

//hipMemGetInfo(&fre, &tot);
//printf("Now free: %u\n", fre);

return rvals;

}

void cudaLaunchError(hipError_t E, dim3 blockdim, dim3 griddim, ArrayMetadata *a, int i, char *srcname)
{
if(E == hipSuccess) return;

printf("Severe CUDA failure in %s: %s -> %s\n", srcname, errorName(E), hipGetErrorString(E));
printf("Array info: dims=<%i %i %i>, numel=%i. I was passed the integer %i.\n", a->dim[0], a->dim[1], a->dim[2], a->numel, i);
printf("Block and grid dimensions: <%i %i %i>, <%i %i %i>\n", blockdim.x, blockdim.y, blockdim.z, griddim.x, griddim.y, griddim.z);
mexErrMsgTxt("Forcing program halt due to CUDA error");

}

void cudaCheckError(char *where)
{
hipError_t epicFail = hipGetLastError();
if(epicFail == hipSuccess) return;

printf("Encountered error at %s: %s -> %s\n", where, errorName(epicFail), hipGetErrorString(epicFail));
mexErrMsgTxt("Forcing program halt due to pre-existing CUDA error");
}

void printdim3(char *name, dim3 dim)
{
printf("dim3 %s is [%i %i %i]\n", name, dim.x, dim.y, dim.z);
}

void printgputag(char *name, int64_t *tag)
{
printf("gputag %s is [*=%lu dims=%lu size=(%lu %lu %lu)]\n", name, tag[0], tag[1], tag[2], tag[3], tag[4]);
}

const char *errorName(hipError_t E)
{
/* Written the stupid way because nvcc is a flaming retarded shitcock that claims these are all "case inaccessible" if it's done with a switch.

Fuck you, dumb lying braindead cockbite. */

  if(E == hipSuccess) { static const char err[]="hipSuccess"; return err; }
  if(E == hipErrorMissingConfiguration) { static const char err[]="hipErrorMissingConfiguration"; return err;  }
  if(E == hipErrorOutOfMemory) { static const char err[]="hipErrorOutOfMemory"; return err; }
  if(E == hipErrorNotInitialized) { static const char err[]="hipErrorNotInitialized"; return err; }
  if(E == hipErrorLaunchFailure) { static const char err[]="hipErrorLaunchFailure"; return err; }
  if(E == hipErrorPriorLaunchFailure) { static const char err[]="cudaerrorPriorLaunchFailure"; return err; }
  if(E == hipErrorLaunchTimeOut) { static const char err[]="hipErrorLaunchTimeOut"; return err; }
  if(E == hipErrorLaunchOutOfResources) { static const char err[]="hipErrorLaunchOutOfResources"; return err; }
  if(E == hipErrorInvalidDeviceFunction) { static const char err[]="hipErrorInvalidDeviceFunction"; return err; }
  if(E == hipErrorInvalidConfiguration) { static const char err[]="cudaErrorInvalidDeviceConfiguration"; return err; }
  if(E == hipErrorInvalidDevice) { static const char err[]="hipErrorInvalidDevice"; return err; }
  if(E == hipErrorInvalidValue) { static const char err[]="hipErrorInvalidValue"; return err; }
  if(E == hipErrorInvalidPitchValue) { static const char err[]="hipErrorInvalidPitchValue"; return err; }
  if(E == hipErrorInvalidSymbol) { static const char err[]="hipErrorInvalidSymbol"; return err; }
  if(E == hipErrorMapFailed) { static const char err[]="hipErrorMapFailed"; return err; }
  if(E == hipErrorUnmapFailed) { static const char err[]="hipErrorUnmapFailed"; return err; }
  if(E == cudaErrorInvalidHostPointer) { static const char err[]="cudaErrorInvalidHostPointer"; return err; }
  if(E == hipErrorInvalidDevicePointer) { static const char err[]="cudaerrorInvalidDevicePointer"; return err; }
  if(E == hipErrorInvalidTexture) { static const char err[]="hipErrorInvalidTexture"; return err; }
  if(E == cudaErrorInvalidTextureBinding) { static const char err[]="cudaErrorInvalidTextureBinding"; return err; }
  if(E == hipErrorInvalidChannelDescriptor) { static const char err[]="hipErrorInvalidChannelDescriptor"; return err; }
  if(E == hipErrorInvalidMemcpyDirection) { static const char err[]="hipErrorInvalidMemcpyDirection"; return err; }
  if(E == cudaErrorAddressOfConstant) { static const char err[]="cudaErrorAddressOfConstant"; return err; }
  if(E == cudaErrorTextureFetchFailed) { static const char err[]="cudaErrorTextureFetchFailed"; return err; }
  if(E == cudaErrorTextureNotBound) { static const char err[]="cudaErrorTextureNotBound"; return err; }
  if(E == cudaErrorSynchronizationError) { static const char err[]="cudaErrorSynchronizationError"; return err; }
  if(E == cudaErrorInvalidFilterSetting) { static const char err[]="cudaErrorInvalidFilterSetting"; return err; }
  if(E == cudaErrorInvalidNormSetting) { static const char err[]="cudaErrorInvalidNormSetting"; return err; }
  if(E == cudaErrorMixedDeviceExecution) { static const char err[]="cudaErrorMixedDeviceExecution"; return err; }
  if(E == hipErrorDeinitialized) { static const char err[]="hipErrorDeinitialized"; return err; }
  if(E == hipErrorUnknown) { static const char err[]="hipErrorUnknown"; return err; }
  if(E == cudaErrorNotYetImplemented) { static const char err[]="cudaErrorNotYetImplemented"; return err; }
  if(E == cudaErrorMemoryValueTooLarge) { static const char err[]="cudaErrorMemoryValueTooLarge"; return err; }
  if(E == hipErrorInvalidHandle) { static const char err[]="cudaErrorInvalidResourcehandle"; return err; }
  if(E == hipErrorNotReady) { static const char err[]="hipErrorNotReady"; return err; }
  if(E == hipErrorInsufficientDriver) { static const char err[]="hipErrorInsufficientDriver"; return err; }
  if(E == hipErrorSetOnActiveProcess) { static const char err[]="hipErrorSetOnActiveProcess"; return err; }
  if(E == cudaErrorInvalidSurface) { static const char err[]="cudaErrorInvalidSurface"; return err; }
  if(E == hipErrorNoDevice) { static const char err[]="hipErrorNoDevice"; return err; }
  if(E == hipErrorECCNotCorrectable) { static const char err[]="hipErrorECCNotCorrectable"; return err; }
  if(E == hipErrorSharedObjectSymbolNotFound) { static const char err[]="cudaErroSharedObjectSymbolNotFound"; return err; }
  if(E == hipErrorSharedObjectInitFailed) { static const char err[]="cudaErroSharedObjectInitFailed"; return err; }
  if(E == hipErrorUnsupportedLimit) { static const char err[]="hipErrorUnsupportedLimit"; return err; }
  if(E == cudaErrorDuplicateVariableName) { static const char err[]="cudaErrorDuplicateVariableName"; return err; }
  if(E == cudaErrorDuplicateTextureName) { static const char err[]="cudaErrorDuplicateTextureName"; return err; }
  if(E == cudaErrorDuplicateSurfaceName) { static const char err[]="cudaErrorDuplicateSurfaceName"; return err; }
  if(E == cudaErrorDevicesUnavailable) { static const char err[]="cudaErrorDevicesUnavailable"; return err; }
  if(E == hipErrorInvalidImage) { static const char err[]="hipErrorInvalidImage"; return err; }
  if(E == hipErrorNoBinaryForGpu) { static const char err[]="hipErrorNoBinaryForGpu"; return err; }
  if(E == cudaErrorIncompatibleDriverContext) { static const char err[]="cudaErrorIncompatibleDriverContext"; return err; }
  if(E == hipErrorPeerAccessAlreadyEnabled) { static const char err[]="hipErrorPeerAccessAlreadyEnabled"; return err; }
  if(E == hipErrorPeerAccessNotEnabled) { static const char err[]="hipErrorPeerAccessNotEnabled"; return err; }
  if(E == hipErrorContextAlreadyInUse) { static const char err[]="hipErrorContextAlreadyInUse"; return err; }
  if(E == hipErrorProfilerDisabled) { static const char err[]="hipErrorProfilerDisabled"; return err; }
  if(E == hipErrorProfilerNotInitialized) { static const char err[]="CudaErrorProfilerNotInitialized"; return err; }
  if(E == hipErrorProfilerAlreadyStarted) { static const char err[]="hipErrorProfilerAlreadyStarted"; return err; }
  if(E == hipErrorProfilerAlreadyStopped) { static const char err[]="hipErrorProfilerAlreadyStopped"; return err; }
//  if(E == hipErrorAssert) { static const char err[]="hipErrorAssert"; return err; }
//  if(E == cudaErrorTooManyPeers) { static const char err[]="cudaErrorTooManyPeers"; return err; }
//  if(E == hipErrorHostMemoryAlreadyRegistered) { static const char err[]="hipErrorHostMemoryAlreadyRegistered"; return err; }
//  if(E == hipErrorHostMemoryNotRegistered) { static const char err[]="hipErrorHostMemoryNotRegistered"; return err; }
//  if(E == hipErrorOperatingSystem) { static const char err[]="cudaErrorOperatingsystem"; return err; }
  if(E == cudaErrorStartupFailure ) { static const char err[]="cudaErrorStartupFailure "; return err; }

return NULL;
}
