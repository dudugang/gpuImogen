#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"


/* Given the RHS, an array to return array size, and the set of array indexes to take *s from */
double **getGPUSourcePointers(const mxArray *prhs[], ArrayMetadata *metaReturn, int fromarg, int toarg)
{

  double **gpuPointers = (double **)malloc((1+toarg-fromarg) * sizeof(double *));
  int iter;

  mxClassID dtype;

  dtype = mxGetClassID(prhs[fromarg]);
  if(dtype != mxINT64_CLASS) mexErrMsgTxt("cudaCommon: fatal, tried to get gpu src pointer from something not a gpu tag.");

  int64_t *dims = (int64_t *)mxGetData(prhs[fromarg]);
  for(iter = 0; iter < 3; iter++) { metaReturn->dim[iter] = (int)dims[2+iter]; } // copy metadata out of first gpu*
  metaReturn->numel = metaReturn->dim[0]*metaReturn->dim[1]*metaReturn->dim[2];
  metaReturn->ndims = dims[1];

  for(iter = fromarg; iter <= toarg; iter++) {
     dtype = mxGetClassID(prhs[iter]);
    if(dtype != mxINT64_CLASS) {
      printf("For argument %i\n",iter);
      mexErrMsgTxt("cudaCommon: fatal, tried to get gpu src pointer from something not a gpu tag.");
      }

    dims = (int64_t *)mxGetData(prhs[iter]);
    gpuPointers[iter-fromarg] = (double *)dims[0];
  }

return gpuPointers;
}

/* Creates destination array that the kernels write to; Returns the GPU memory pointer, and assigns the LHS it's passed */
double **makeGPUDestinationArrays(int64_t *reference, mxArray *retArray[], int howmany)
{

double **rvals = (double **)malloc(howmany*sizeof(double *));
int i;
mwSize dims[2]; dims[0] = 5; dims[1] = 1;

int64_t *rv; size_t numel;

numel = reference[2]*reference[3]*reference[4];

for(i = 0; i < howmany; i++) {
  retArray[i] = mxCreateNumericArray(2, dims, mxINT64_CLASS, mxREAL);
  rv = (int64_t *)mxGetData(retArray[i]);

  hipError_t fail = hipMalloc((void **)&rv[0], numel*sizeof(double));
  if(fail != hipSuccess) {
    printf("On array %i/%i: %s\n", i+1, howmany, hipGetErrorString(fail));
    cudaCheckError("In makeGPUDestinationArrays: malloc failed and I am sad.");
    }

  int q; for(q = 1; q < 5; q++) rv[q] = reference[q];
  rvals[i] = (double *)rv[0];
  }

//size_t tot,fre;

//hipMemGetInfo(&fre, &tot);
//printf("Now free: %u\n", fre);

return rvals;

}

void getLaunchForXYCoverage(int *dims, int blkX, int blkY, int nhalo, dim3 *blockdim, dim3 *griddim)
{

blockdim->x = blkX;
blockdim->y = blkY;
blockdim->z = 1;

griddim->x = dims[0] / (blkX-2*nhalo); griddim->x += (griddim->x * (blkX-2*nhalo) < dims[0]);
griddim->y = dims[1] / (blkY-2*nhalo); griddim->y += (griddim->y * (blkY-2*nhalo) < dims[1]);
griddim->z = 1;
}

void cudaLaunchError(hipError_t E, dim3 blockdim, dim3 griddim, ArrayMetadata *a, int i, char *srcname)
{
if(E == hipSuccess) return;

printf("Severe CUDA failure in %s: %s -> %s\n", srcname, errorName(E), hipGetErrorString(E));
printf("Array info: dims=<%i %i %i>, numel=%i. I was passed the integer %i.\n", a->dim[0], a->dim[1], a->dim[2], a->numel, i);
printf("Block and grid dimensions: <%i %i %i>, <%i %i %i>\n", blockdim.x, blockdim.y, blockdim.z, griddim.x, griddim.y, griddim.z);
mexErrMsgTxt("Forcing program halt due to CUDA error");

}

void cudaCheckError(char *where)
{
hipError_t epicFail = hipGetLastError();
if(epicFail == hipSuccess) return;

printf("Encountered error at %s: %s -> %s\n", where, errorName(epicFail), hipGetErrorString(epicFail));
mexErrMsgTxt("Forcing program halt due to CUDA error");
}

void printdim3(char *name, dim3 dim)
{
printf("dim3 %s is [%i %i %i]\n", name, dim.x, dim.y, dim.z);
}

void printgputag(char *name, int64_t *tag)
{
printf("gputag %s is [*=%lu dims=%lu size=(%lu %lu %lu)]\n", name, tag[0], tag[1], tag[2], tag[3], tag[4]);
}

#define NOM(x) if(E == x) { static const char err[]=#x; return err; }

const char *errorName(hipError_t E)
{
/* Written the stupid way because nvcc is idiotically claims these are all "case inaccessible" if it's done with a switch.

WRONG, asshole! */
// OM...
NOM(hipSuccess)
NOM(hipErrorMissingConfiguration)
NOM(hipErrorOutOfMemory)
NOM(hipErrorNotInitialized)
NOM(hipErrorLaunchFailure)
NOM(hipErrorPriorLaunchFailure)
NOM(hipErrorLaunchTimeOut)
NOM(hipErrorLaunchOutOfResources)
NOM(hipErrorInvalidDeviceFunction)
NOM(hipErrorInvalidConfiguration)
NOM(hipErrorInvalidDevice)
NOM(hipErrorInvalidValue)
NOM(hipErrorInvalidPitchValue)
NOM(hipErrorInvalidSymbol)
NOM(hipErrorMapFailed)
NOM(hipErrorUnmapFailed)
NOM(cudaErrorInvalidHostPointer)
NOM(hipErrorInvalidDevicePointer)
NOM(hipErrorInvalidTexture)
NOM(cudaErrorInvalidTextureBinding)
NOM(hipErrorInvalidChannelDescriptor)
NOM(hipErrorInvalidMemcpyDirection)
NOM(cudaErrorAddressOfConstant)
NOM(cudaErrorTextureFetchFailed)
NOM(cudaErrorTextureNotBound)
NOM(cudaErrorSynchronizationError)
NOM(cudaErrorInvalidFilterSetting)
NOM(cudaErrorInvalidNormSetting)
NOM(cudaErrorMixedDeviceExecution)
NOM(hipErrorDeinitialized)
NOM(hipErrorUnknown)
NOM(cudaErrorNotYetImplemented)
NOM(cudaErrorMemoryValueTooLarge)
NOM(hipErrorInvalidHandle)
NOM(hipErrorNotReady)
NOM(hipErrorInsufficientDriver)
NOM(hipErrorSetOnActiveProcess)
NOM(cudaErrorInvalidSurface)
NOM(hipErrorNoDevice)
NOM(hipErrorECCNotCorrectable)
NOM(hipErrorSharedObjectSymbolNotFound)
NOM(hipErrorSharedObjectInitFailed)
NOM(hipErrorUnsupportedLimit)
NOM(cudaErrorDuplicateVariableName)
NOM(cudaErrorDuplicateTextureName)
NOM(cudaErrorDuplicateSurfaceName)
NOM(cudaErrorDevicesUnavailable)
NOM(hipErrorInvalidImage)
NOM(hipErrorNoBinaryForGpu)
NOM(cudaErrorIncompatibleDriverContext)
NOM(hipErrorPeerAccessAlreadyEnabled)
NOM(hipErrorPeerAccessNotEnabled)
NOM(hipErrorContextAlreadyInUse)
NOM(hipErrorProfilerDisabled)
NOM(hipErrorProfilerNotInitialized)
NOM(hipErrorProfilerAlreadyStarted)
NOM(hipErrorProfilerAlreadyStopped)
/*hipErrorAssert
cudaErrorTooManyPeers
hipErrorHostMemoryAlreadyRegistered
hipErrorHostMemoryNotRegistered
hipErrorOperatingSystem*/
NOM(cudaErrorStartupFailure)
// ... NOM, ASSHOLE!
return NULL;
}
