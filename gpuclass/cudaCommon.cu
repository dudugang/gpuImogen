#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"


/* Given the RHS, an array to return array size, and the set of array indexes to take *s from */
double **getGPUSourcePointers(const mxArray *prhs[], ArrayMetadata *metaReturn, int fromarg, int toarg)
{

  double **gpuPointers = (double **)malloc((1+toarg-fromarg) * sizeof(double *));
  int iter;

  mxClassID dtype;

  dtype = mxGetClassID(prhs[fromarg]);
  if(dtype != mxINT64_CLASS) mexErrMsgTxt("cudaCommon: fatal, tried to get gpu src pointer from something not a gpu tag.");

  int64_t *dims = (int64_t *)mxGetData(prhs[fromarg]);
  for(iter = 0; iter < 3; iter++) { metaReturn->dim[iter] = (int)dims[2+iter]; } // copy metadata out of first gpu*
  metaReturn->numel = metaReturn->dim[0]*metaReturn->dim[1]*metaReturn->dim[2];
  metaReturn->ndims = dims[1];

  for(iter = fromarg; iter <= toarg; iter++) {
     dtype = mxGetClassID(prhs[iter]);
    if(dtype != mxINT64_CLASS) {
      printf("For argument %i\n",iter);
      mexErrMsgTxt("cudaCommon: fatal, tried to get gpu src pointer from something not a gpu tag.");
      }

    dims = (int64_t *)mxGetData(prhs[iter]);
    gpuPointers[iter-fromarg] = (double *)dims[0];
  }

return gpuPointers;
}

/* Creates destination array that the kernels write to; Returns the GPU memory pointer, and assigns the LHS it's passed */
double **makeGPUDestinationArrays(int64_t *reference, mxArray *retArray[], int howmany)
{

double **rvals = (double **)malloc(howmany*sizeof(double *));
int i;
mwSize dims[2]; dims[0] = 5; dims[1] = 1;

int64_t *rv; size_t numel;

numel = reference[2]*reference[3]*reference[4];

for(i = 0; i < howmany; i++) {
  retArray[i] = mxCreateNumericArray(2, dims, mxINT64_CLASS, mxREAL);
  rv = (int64_t *)mxGetData(retArray[i]);

  hipError_t fail = hipMalloc((void **)&rv[0], numel*sizeof(double));
  if(fail != hipSuccess) {
    printf("%s\n", hipGetErrorString(fail));
    mexErrMsgTxt("makeGPUDestinationArrays: I haz an hipMalloc fail. And a sad.");
    }

  int q; for(q = 1; q < 5; q++) rv[q] = reference[q];
  rvals[i] = (double *)rv[0];
  }

return rvals;

}

