#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"
#include "cudaStatics.h"

/* THIS FUNCTION:
   cudaStatics is used in the imposition of several kinds of boundary conditions
   upon arrays. Given a list of indices I, coefficients C and values V, it
   writes out

   phi[I] = (1-C)*phi[I] + C[i]*V[i],
   causing phi[I] to fade to V[i] at an exponential rate.

   It is also able to set mirror boundary conditions (FIXME: Not fully tested!)
 */

/* FIXME: rewrite this crap with template<>s */
/* X DIRECTION SYMMETRIC/ANTISYMMETRIC BC KERNELS FOR MIRROR BCS */
/* Assume a block size of [3 A B] */
__global__ void cukern_xminusSymmetrize(double *phi, int nx, int ny, int nz);
__global__ void cukern_xminusAntisymmetrize(double *phi, int nx, int ny, int nz);
__global__ void cukern_xplusSymmetrize(double *phi, int nx, int ny, int nz);
__global__ void cukern_xplusAntisymmetrize(double *phi, int nx, int ny, int nz);
/* Y DIRECTION SYMMETRIC/ANTISYMMETRIC BC KERNELS */
/* assume a block size of [N 1 M] */
__global__ void cukern_yminusSymmetrize(double *phi, int nx, int ny, int nz);
__global__ void cukern_yminusAntisymmetrize(double *phi, int nx, int ny, int nz);
__global__ void cukern_yplusSymmetrize(double *phi, int nx, int ny, int nz);
__global__ void cukern_yplusAntisymmetrize(double *phi, int nx, int ny, int nz);
/* Z DIRECTION SYMMETRIC/ANTISYMMETRIC BC KERNELS */
/* Assume launch with size [U V 1] */
__global__ void cukern_zminusSymmetrize(double *Phi, int nx, int ny, int nz);
__global__ void cukern_zminusAntisymmetrize(double *Phi, int nx, int ny, int nz);
__global__ void cukern_zplusSymmetrize(double *Phi, int nx, int ny, int nz);
__global__ void cukern_zplusAntisymmetrize(double *Phi, int nx, int ny, int nz);

/* X direction extrapolated boundary conditions */
/* Launch size [3 A B] */
__global__ void cukern_extrapolateLinearBdyXMinus(double *phi, int nx, int ny, int nz);
__global__ void cukern_extrapolateLinearBdyXPlus(double *phi, int nx, int ny, int nz);

__global__ void cukern_extrapolateConstBdyXMinus(double *phi, int nx, int ny, int nz);
__global__ void cukern_extrapolateConstBdyXPlus(double *phi, int nx, int ny, int nz);
__global__ void cukern_extrapolateConstBdyYMinus(double *phi, int nx, int ny, int nz);
__global__ void cukern_extrapolateConstBdyYPlus(double *phi, int nx, int ny, int nz);
__global__ void cukern_extrapolateConstBdyZMinus(double *phi, int nx, int ny, int nz);
__global__ void cukern_extrapolateConstBdyZPlus(double *phi, int nx, int ny, int nz);

__global__ void cukern_applySpecial_fade(double *phi, double *statics, int nSpecials, int blkOffset);

int setBoundarySAS(MGArray *phi, int side, int direction, int sas);

#ifdef STANDALONE_MEX_FUNCTION

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	if( (nlhs != 0) || (nrhs != 3)) { mexErrMsgTxt("cudaStatics operator is cudaStatics(ImogenArray, blockdim, direction)"); }

	CHECK_CUDA_ERROR("entering cudaStatics");

	setBoundaryConditions(NULL, prhs[0], (int)*mxGetPr(prhs[2]));
}
#endif

/* FIXME: This is terrible.
 * FIXME: MGArray needs to provision carrying its own boundary condition metadata around somehow.
 */
int setBoundaryConditions(MGArray *array, const mxArray *matlabhandle, int direction)
{
	CHECK_CUDA_ERROR("entering setBoundaryConditions");

	MGArray phi, statics;
	int worked;
	if(array == NULL) {
		worked = MGA_accessMatlabArrays((const mxArray **)&matlabhandle, 0, 0, &phi);
		BAIL_ON_FAIL(worked)
	} else {
		worked = (array->matlabClassHandle == matlabhandle) ? SUCCESSFUL : ERROR_CRASH;
		if(worked != SUCCESSFUL) {
			PRINT_FAULT_HEADER;
			printf("setBoundaryConditions permits both the MGArray and its Matlab handle to be passed because the MGA may have been internally modified without the handle having been, but the MGArray must name that Matlab handle as its originator.\n");
			PRINT_FAULT_FOOTER;
			BAIL_ON_FAIL(worked);
		}
		phi = array[0];
	}

	/* Grabs the whole boundaryData struct from the ImogenArray class */
	mxArray *boundaryData = mxGetProperty(matlabhandle, 0, "boundaryData");
	if(boundaryData == NULL) {
		printf("FATAL: field 'boundaryData' D.N.E. in class. Not a class? Not an ImogenArray/FluidArray?\n");
		return ERROR_INVALID_ARGS;
	}

	/* The statics describe "solid" structures which we force the grid to have */
	mxArray *gpuStatics = mxGetField(boundaryData, 0, "staticsData");
	if(gpuStatics == NULL) {
		printf("FATAL: field 'staticsData' D.N.E. in boundaryData struct. Statics not compiled?\n");
		return ERROR_INVALID_ARGS;
	}
	worked = MGA_accessMatlabArrays((const mxArray **)(&gpuStatics), 0, 0, &statics);
	BAIL_ON_FAIL(worked)

	int *perm = &phi.currentPermutation[0];
	int offsetidx = 2*(perm[0]-1) + 1*(perm[1] > perm[2]);

	/* The offset array describes the index offsets for the data in the gpuStatics array */
	mxArray *offsets    = mxGetField(boundaryData, 0, "compOffset");
	if(offsets == NULL) {
		printf("FATAL: field 'compOffset' D.N.E. in boundaryData. Not an ImogenArray? Statics not compiled?\n");
		return ERROR_INVALID_ARGS;
	}
	double *offsetcount = mxGetPr(offsets);
	long int staticsOffset = (long int)offsetcount[2*offsetidx];
	int staticsNumel  = (int)offsetcount[2*offsetidx+1];

	/* Parameter describes what block size to launch with... */
	int blockdim = 8;

	dim3 griddim; griddim.x = staticsNumel / blockdim + 1;
	if(griddim.x > 32768) {
		griddim.x = 32768;
		griddim.y = staticsNumel/(blockdim*griddim.x) + 1;
	}

	/* Every call results in applying specials */
	if(staticsNumel > 0) {
		PAR_WARN(phi);
		cukern_applySpecial_fade<<<griddim, blockdim>>>(phi.devicePtr[0], statics.devicePtr[0] + staticsOffset, staticsNumel, statics.dim[0]);
		worked = CHECK_CUDA_LAUNCH_ERROR(blockdim, griddim, &phi, 0, "cuda statics application");
		if(worked != SUCCESSFUL) return worked;
	}

	/* Indicates which part of a 3-vector this array is (0 = scalar, 123=XYZ) */
	mxArray *comp = mxGetProperty(matlabhandle, 0, "component");
	int vectorComponent;
	if(comp != NULL) {
		vectorComponent = (int)(*mxGetPr(comp));
	} else {
		printf("Failed to fetch 'component' field of class: Not an ImogenArray? Bailing.\n");
		return ERROR_INVALID_ARGS;
	}

	/* BEGIN DETERMINATION OF ANALYTIC BOUNDARY CONDITIONS */
	int numDirections = 1;

	mxArray *bcModes = mxGetField(boundaryData, 0, "bcModes");
	if(bcModes == NULL) {
		printf("FATAL: bcModes structure not present. Not an ImogenArray? Not initialized?\n");
		return ERROR_INVALID_ARGS;
	}

	int j;
	for(j = 0; j < numDirections; j++) {
		if(direction == 0) continue; /* Skips edge BCs if desired. */
		int memoryDirection = perm[direction-1];

		/* So this is kinda brain-damaged, but the boundary condition modes are stored in the form
       { 'type minus x', 'type minus y', 'type minus z';
	 'type plus  x', 'type plus y',  'type plus z'};
       Yes, strings in a cell array. */
		/* Okay, that's not kinda, it's straight-up stupid. */

		mxArray *bcstr; char *bs;

		int d; for(d = 0; d < 2; d++) {
			bcstr = mxGetCell(bcModes, 2*(direction-1) + d);
			bs = (char *)malloc(sizeof(char) * (mxGetNumberOfElements(bcstr)+1));
			mxGetString(bcstr, bs, mxGetNumberOfElements(bcstr)+1);

			// Sets a mirror BC: scalar, vector_perp f(b+x) = f(b-x), vector normal f(b+x) = -f(b-x)
			if(strcmp(bs, "mirror") == 0)
				worked = setBoundarySAS(&phi, d, memoryDirection, vectorComponent == direction);

			// Extrapolates f(b+x) = f(b)
			if(strcmp(bs, "const") == 0) {
				worked = setBoundarySAS(&phi, d, memoryDirection, 2);
			}

			// Extrapolates f(b+x) = f(b) + x f'(b)
			// WARNING: This is unconditionally unstable unless normal flow rate is supersonic
			if(strcmp(bs, "linear") == 0) {
				worked = setBoundarySAS(&phi, d, memoryDirection, 3);
			}

			if(strcmp(bs, "wall") == 0) {
				printf("Wall BC is not implemented!\n");
				return ERROR_INVALID_ARGS;
			} 

		}
		if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) return worked;
	}

	return SUCCESSFUL;
}

/* Sets the given array+AMD's boundary in the following manner:
   side      -> 0 = negative edge  1 = positive edge
   direction -> 1 = X	      2 = Y               3 = Z*
   sas       -> 0 = symmetrize      1 => antisymmetrize
	     -> 2 = extrap constant 3-> extrap linear

 *: As passed, assuming ImogenArray's indexPermute has been handled for us.
 */

void callBCKernel(dim3 griddim, dim3 blockdim, double *x, int nx, int ny, int nz, int ktable)
{
	switch(ktable) {
	case 0: cukern_xminusSymmetrize<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 1: cukern_xminusAntisymmetrize<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 2: cukern_extrapolateConstBdyXMinus<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 3: cukern_extrapolateLinearBdyXMinus<<<griddim, blockdim>>>(x, nx, ny, nz); break;

	case 4: cukern_xplusSymmetrize<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 5: cukern_xplusAntisymmetrize<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 6: cukern_extrapolateConstBdyXPlus<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 7: cukern_extrapolateLinearBdyXPlus<<<griddim, blockdim>>>(x, nx, ny, nz); break;

	case 8: cukern_yminusSymmetrize<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 9: cukern_yminusAntisymmetrize<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 10: cukern_extrapolateConstBdyYMinus<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 11: mexErrMsgTxt("Fatal: This boundary condition (y-minus, linear) has not been implemented yet."); break;

	case 12: cukern_yplusSymmetrize<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 13: cukern_yplusAntisymmetrize<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 14: cukern_extrapolateConstBdyYPlus<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 15: mexErrMsgTxt("Fatal: This boundary condition (y-plus, linear) has not been implemented yet."); break;

	case 16: cukern_zminusSymmetrize<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 17: cukern_zminusAntisymmetrize<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 18: cukern_extrapolateConstBdyZMinus<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 19: mexErrMsgTxt("Fatal: This boundary condition (z-minus linear) has not been implemented yet."); break;

	case 20: cukern_zplusSymmetrize<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 21: cukern_zplusAntisymmetrize<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 22: cukern_extrapolateConstBdyZPlus<<<griddim, blockdim>>>(x, nx, ny, nz); break;
	case 23: mexErrMsgTxt("Fatal: This boundary condition (z-plus linear) has not been implemented yet."); break;
	}

}
void *getBCKernel(int X)
{
	void *PLACEHOLDER = NULL;

	void *kerntable[24] = {(void *)&cukern_xminusSymmetrize, \
			(void *)&cukern_xminusAntisymmetrize, \
			(void *)&cukern_extrapolateConstBdyXMinus, \
			(void *)&cukern_extrapolateLinearBdyXMinus, \

			(void *)&cukern_xplusSymmetrize, \
			(void *)&cukern_xplusAntisymmetrize,
			(void *)&cukern_extrapolateConstBdyXPlus, \
			(void *)&cukern_extrapolateLinearBdyXPlus, \

			(void *)&cukern_yminusSymmetrize, \
			(void *)&cukern_yminusAntisymmetrize, \
			PLACEHOLDER, \
			PLACEHOLDER, \

			(void *)&cukern_yplusSymmetrize, \
			(void *)&cukern_yplusAntisymmetrize,
			PLACEHOLDER, \
			PLACEHOLDER, \

			(void *)&cukern_zminusSymmetrize, \
			(void *)&cukern_zminusAntisymmetrize, \
			PLACEHOLDER, \
			PLACEHOLDER, \

			(void *)&cukern_zplusSymmetrize, \
			(void *)&cukern_zplusAntisymmetrize, \
			PLACEHOLDER, \
			PLACEHOLDER };

	return kerntable[X];
}

int setBoundarySAS(MGArray *phi, int side, int direction, int sas)
{
	dim3 blockdim, griddim;
	void (* bckernel)(double *, int, int, int);
	int i, sub[6];

	int returnCode;

	switch(direction) {
	case 1: { blockdim.x = 3; blockdim.y = 16; blockdim.z = 8; } break;
	case 2: { blockdim.x = 16; blockdim.y = 1; blockdim.z = 16; } break;
	case 3: { blockdim.x = 16; blockdim.y = 16; blockdim.z = 1; } break;
	}

	// This is the easy case; We just have to apply a left-side condition to the leftmost partition and a
	// right-side condition to the rightmost partition and we're done
	if(direction == phi->partitionDir) {
		switch(direction) {
		case 1: {
			griddim.x = phi->dim[1] / blockdim.y; griddim.x += (griddim.x*blockdim.y < phi->dim[1]);
			griddim.y = phi->dim[2] / blockdim.z; griddim.y += (griddim.y*blockdim.z < phi->dim[2]);
		} break;
		case 2: {
			griddim.x = phi->dim[0] / blockdim.x; griddim.x += (griddim.x*blockdim.x < phi->dim[0]);
			griddim.y = phi->dim[2] / blockdim.z; griddim.y += (griddim.y*blockdim.z < phi->dim[2]);
		} break;
		case 3: {
			griddim.x = phi->dim[0] / blockdim.x; griddim.x += (griddim.x*blockdim.x < phi->dim[0]);
			griddim.y = phi->dim[1] / blockdim.y; griddim.y += (griddim.y*blockdim.y < phi->dim[1]);
		} break;
		}
		i = (side == 0) ? 0 : (phi->nGPUs - 1);
		hipSetDevice(phi->deviceID[i]);
		returnCode = CHECK_CUDA_ERROR("hipSetDevice()");
		if(returnCode != SUCCESSFUL) return returnCode;

		//bckernel = (void (*)(double *, int, int, int))getBCKernel(sas + 4*side + 8*(direction-1));
		//if((void *)bckernel == NULL) mexErrMsgTxt("Fatal: This boundary condition has not been implemented yet.");

		//bckernel<<<griddim, blockdim>>>(phi->devicePtr[i], phi->dim[0], phi->dim[1], phi->dim[2]);
		calcPartitionExtent(phi, i, sub);

		callBCKernel(griddim, blockdim, phi->devicePtr[i], sub[3], sub[4], sub[5], sas + 4*side + 8*(direction-1));
		returnCode = CHECK_CUDA_LAUNCH_ERROR(blockdim, griddim, phi, sas + 2*side + 4*direction, "In setBoundarySAS; integer -> cukern table index");
		if(returnCode != SUCCESSFUL) return returnCode;
	} else {
		// If the BC isn't on a face that's aimed in the partitioned direction,
		// we have to loop and apply it to all partitions.
		for(i = 0; i < phi->nGPUs; i++) {
			calcPartitionExtent(phi, i, sub);
			// Set the launch size based on partition extent
			switch(direction) {
			case 1: {
				griddim.x = sub[4] / blockdim.y; griddim.x += (griddim.x*blockdim.y < sub[4]);
				griddim.y = sub[5] / blockdim.z; griddim.y += (griddim.y*blockdim.z < sub[5]);
			} break;
			case 2: {
				griddim.x = sub[3] / blockdim.x; griddim.x += (griddim.x*blockdim.x < sub[3]);
				griddim.y = sub[5] / blockdim.z; griddim.y += (griddim.y*blockdim.z < sub[5]);
			} break;
			case 3: {
				griddim.x = sub[3] / blockdim.x; griddim.x += (griddim.x*blockdim.x < sub[3]);
				griddim.y = sub[4] / blockdim.y; griddim.y += (griddim.y*blockdim.y < sub[4]);
			} break;
			}
			hipSetDevice(phi->deviceID[i]);
			returnCode = CHECK_CUDA_ERROR("hipSetDevice()");
			if(returnCode != SUCCESSFUL) return returnCode;

			//bckernel = (void (*)(double *, int, int, int))getBCKernel(sas + 4*side + 8*(direction-1));
			//if((void *)bckernel == NULL)

			callBCKernel(griddim, blockdim, phi->devicePtr[i], sub[3], sub[4], sub[5], sas + 4*side + 8*(direction-1));

			//bckernel<<<griddim, blockdim>>>(phi->devicePtr[i], sub[3], sub[4], sub[5]);
			returnCode = CHECK_CUDA_LAUNCH_ERROR(blockdim, griddim, phi, sas + 4*side + 8*(direction-1), "In setBoundarySAS; integer -> cukern table index");
			if(returnCode != SUCCESSFUL) return returnCode;
		}

	}

	return SUCCESSFUL;
}



__global__ void cukern_applySpecial_fade(double *phi, double *statics, int nSpecials, int blkOffset)
{
	int myAddr = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x*blockIdx.y);
	if(myAddr >= nSpecials) return;
	statics += myAddr;

	long int xaddr = (long int)statics[0];
	double f0      =	   statics[blkOffset];
	double c       =	   statics[blkOffset*2];

	//	if(c >= 0) {
	// Fade condition: Exponentially pulls cell towards c with rate constant f0;
	phi[xaddr] = f0*c + (1.0-c)*phi[xaddr];
	//	} else {
	// Wall condition: Any transfer between the marked cells is reversed
	// Assumptions: 2nd cell (xprimeaddr) must be in a stationary, no-flux region
	//		long int xprimeaddr = (long int) statics[myAddr + blkOffset*3];
	//		phi[xaddr] += (phi[xprimeaddr]-f0);
	//		phi[xprimaddr] = f0;
	//	}

}


/* X DIRECTION SYMMETRIC/ANTISYMMETRIC BC KERNELS FOR MIRROR BCS */
/* Assume a block size of [3 A B] with grid dimensions [M N 1] s.t. AM >= ny, BN >= nz*/
/* Define the preamble common to all of these kernels: */
#define XSASKERN_PREAMBLE \
		int stridey = nx; int stridez = nx*ny; \
		int yidx = threadIdx.y + blockIdx.x*blockDim.y; \
		int zidx = threadIdx.z + blockIdx.y*blockDim.z; \
		if(yidx >= ny) return; if(zidx >= nz) return; \
		phi += stridey*yidx + stridez*zidx;

/* We establish symmetry or antisymmetry such that we have 
 * [... A B C D  C  B  A|-> BOUNDARY
 * [... A B C D -C -B -A|-> BOUNDARY 
 * i.e. symmetry is about the 4th cell from the boundary */


// X direction kernels just use 3 threads in order to acheive slightly less terrible
// memory access patterns
__global__ void cukern_xminusSymmetrize(double *phi, int nx, int ny, int nz)
{
	XSASKERN_PREAMBLE
	phi[2-threadIdx.x] = phi[4+threadIdx.x];
}

__global__ void cukern_xminusAntisymmetrize(double *phi, int nx, int ny, int nz)
{
	XSASKERN_PREAMBLE
	phi[2-threadIdx.x] = -phi[4+threadIdx.x];
}

__global__ void cukern_xplusSymmetrize(double *phi, int nx, int ny, int nz)
{
	XSASKERN_PREAMBLE
	phi[nx-3+threadIdx.x] = phi[nx-5-threadIdx.x];
}

__global__ void cukern_xplusAntisymmetrize(double *phi, int nx, int ny, int nz)
{
	XSASKERN_PREAMBLE
	phi[nx-3+threadIdx.x] = -phi[nx-5-threadIdx.x];
}


/* These are called when a BC is set to 'const' or 'linear' */
__global__ void cukern_extrapolateConstBdyXMinus(double *phi, int nx, int ny, int nz)
{
	XSASKERN_PREAMBLE
	phi[threadIdx.x] = phi[3];
}

__global__ void cukern_extrapolateConstBdyXPlus(double *phi, int nx, int ny, int nz)
{
	XSASKERN_PREAMBLE
	phi[nx-3+threadIdx.x] = phi[nx-4];
}

__global__ void cukern_extrapolateLinearBdyXMinus(double *phi, int nx, int ny, int nz)
{
	__shared__ double f[3];
	XSASKERN_PREAMBLE
	f[threadIdx.x] = phi[threadIdx.x+3];
	__syncthreads();
	phi[threadIdx.x] = phi[3] + (3-threadIdx.x)*(f[0]-f[1]);
}

__global__ void cukern_extrapolateLinearBdyXPlus(double *phi, int nx, int ny, int nz)
{
	__shared__ double f[3];
	XSASKERN_PREAMBLE
	phi += nx - 5;
	f[threadIdx.x] = phi[threadIdx.x];
	__syncthreads();
	phi[threadIdx.x+2] = f[1] + (threadIdx.x+1)*(f[1]-f[0]);
}


/* Y DIRECTION SYMMETRIC/ANTISYMMETRIC BC KERNELS */
/* assume a block size of [A 1 B] with grid dimensions [M N 1] s.t. AM >= nx, BN >=nz */
#define YSASKERN_PREAMBLE \
		int stridez = nx*ny; \
		int xidx = threadIdx.x + blockIdx.x*blockDim.x; \
		int zidx = threadIdx.z + blockIdx.y*blockDim.y; \
		if(xidx >= nx) return; if(zidx >= nz) return;   \
		phi += xidx + stridez*zidx; \
		int q;

__global__ void cukern_yminusSymmetrize(double *phi, int nx, int ny, int nz)
{
	YSASKERN_PREAMBLE
	for(q = 0; q < 3; q++) { phi[nx*q] = phi[nx*(6-q)]; }
}

__global__ void cukern_yminusAntisymmetrize(double *phi, int nx, int ny, int nz)
{
	YSASKERN_PREAMBLE
	for(q = 0; q < 3; q++) { phi[nx*q] = -phi[nx*(6-q)]; }
}

__global__ void cukern_yplusSymmetrize(double *phi, int nx, int ny, int nz)
{
	YSASKERN_PREAMBLE
	for(q = 0; q < 3; q++) { phi[nx*(ny-1-q)] = phi[nx*(ny-7+q)]; }
}

__global__ void cukern_yplusAntisymmetrize(double *phi, int nx, int ny, int nz)
{
	YSASKERN_PREAMBLE
	for(q = 0; q < 3; q++) { phi[nx*(ny-1-q)] = -phi[nx*(ny-7+q)]; }
}

__global__ void cukern_extrapolateConstBdyYMinus(double *phi, int nx, int ny, int nz)
{
	YSASKERN_PREAMBLE
	double f = phi[3*nx];
	for(q = 0; q < 3; q++) { phi[q*nx] = f; }
}

__global__ void cukern_extrapolateConstBdyYPlus(double *phi, int nx, int ny, int nz)
{
	YSASKERN_PREAMBLE
	double f = phi[(ny-4)*nx];
	for(q = 0; q < 3; q++) { phi[(ny-3+q)*nx] = f; }
}

/* Z DIRECTION SYMMETRIC/ANTISYMMETRIC BC KERNELS */
/* Assume launch with size [A B 1] and grid of size [M N 1] s.t. AM >= nx, BN >= ny*/
#define ZSASKERN_PREAMBLE \
		int xidx = threadIdx.x + blockIdx.x * blockDim.x; \
		int yidx = threadIdx.y + blockIdx.y * blockDim.y; \
		if(xidx >= nx) return; if(yidx >= ny) return; \
		phi += xidx + nx*yidx; \
		int stride = nx*ny;

__global__ void cukern_zminusSymmetrize(double *phi, int nx, int ny, int nz)
{
	ZSASKERN_PREAMBLE

	int q;
	for(q = 0; q < 3; q++) // nvcc will unroll it
		phi[q*stride] = phi[(6-q)*stride];
}

__global__ void cukern_zminusAntisymmetrize(double *phi, int nx, int ny, int nz)
{
	ZSASKERN_PREAMBLE
	int q;
	for(q = 0; q < 3; q++) 
		phi[q*stride] = -phi[(6-q)*stride];
}

__global__ void cukern_zplusSymmetrize(double *phi, int nx, int ny, int nz)
{
	ZSASKERN_PREAMBLE
	int q;
	for(q = 0; q < 3; q++)
		phi[stride*(nz-1-q)] = phi[stride*(nz-7+q)];
}

__global__ void cukern_zplusAntisymmetrize(double *phi, int nx, int ny, int nz)
{
	ZSASKERN_PREAMBLE
	int q;
	for(q = 0; q < 3; q++)
		phi[stride*(nz-1-q)] = -phi[stride*(nz-7+q)];

}

__global__ void cukern_extrapolateConstBdyZMinus(double *phi, int nx, int ny, int nz)
{
	ZSASKERN_PREAMBLE
	int q;
	for(q = 0; q < 3; q++)
		phi[stride*q] = phi[stride*4];
}

__global__ void cukern_extrapolateConstBdyZPlus(double *phi, int nx, int ny, int nz)
{

	ZSASKERN_PREAMBLE
	int q;
	for(q = 0; q < 3; q++)
		phi[stride*(nz-1-q)] = phi[stride*(nz-4)];

}
