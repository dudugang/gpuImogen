#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"

/* X DIRECTION SYMMETRIC/ANTISYMMETRIC BC KERNELS FOR MIRROR BCS */
/* Assume a block size of [3 A B] */
__global__ void cukern_xminusSymmetrize(double *phi, int nx, int ny, int nz);
__global__ void cukern_xminusAntisymmetrize(double *phi, int nx, int ny, int nz);
__global__ void cukern_xplusSymmetrize(double *phi, int nx, int ny, int nz);
__global__ void cukern_xplusAntisymmetrize(double *phi, int nx, int ny, int nz);
/* Y DIRECTION SYMMETRIC/ANTISYMMETRIC BC KERNELS */
/* assume a block size of [N 1 M] */
__global__ void cukern_yminusSymmetrize(double *phi, int nx, int ny, int nz);
__global__ void cukern_yminusAntisymmetrize(double *phi, int nx, int ny, int nz);
__global__ void cukern_yplusSymmetrize(double *phi, int nx, int ny, int nz);
__global__ void cukern_yplusAntisymmetrize(double *phi, int nx, int ny, int nz);
/* Z DIRECTION SYMMETRIC/ANTISYMMETRIC BC KERNELS */
/* Assume launch with size [U V 1] */
__global__ void cukern_zminusSymmetrize(double *Phi, int nx, int ny, int nz);
__global__ void cukern_zminusAntisymmetrize(double *Phi, int nx, int ny, int nz);
__global__ void cukern_zplusSymmetrize(double *Phi, int nx, int ny, int nz);
__global__ void cukern_zplusAntisymmetrize(double *Phi, int nx, int ny, int nz);

__global__ void cukern_applySpecial_fade(double *phi, double *statics, int nSpecials, int blkOffset);

void setBoundarySAS(double *gpuarray, ArrayMetadata *amd, int side, int direction, int sas);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  if( (nlhs != 0) || (nrhs != 3)) { mexErrMsgTxt("cudaStatics operator is cudaStatics(ImogenArray, blockdim, direction)"); }

  CHECK_CUDA_ERROR("entering cudaStatics");

  ArrayMetadata ama, amf;

  /* This will force an error exit if invalid */
  double **array = getGPUSourcePointers(prhs, &ama, 0, 0);

  /* Grabs the whole boundaryData struct from the ImogenArray class */
  mxArray *boundaryData = mxGetProperty(prhs[0], 0, "boundaryData");
  if(boundaryData == NULL) mexErrMsgTxt("FATAL: field 'boundaryData' D.N.E. in class. Not a class? Not an ImogenArray?\n");

  /* The statics describe "solid" structures which we force the grid to have */
  mxArray *gpuStatics = mxGetField(boundaryData, 0, "staticsData");
  if(gpuStatics == NULL) mexErrMsgTxt("FATAL: field 'staticsData' D.N.E. in boundaryData struct. Statics weren't compiled?\n");
  double **statics = getGPUSourcePointers((const mxArray **)(&gpuStatics), &amf, 0, 0);

  /* The indexPermute property tells us how the array's indices are currently oriented. */
  mxArray *permArray =  mxGetProperty(prhs[0], 0, "indexPermute");
  if(permArray == NULL) mexErrMsgTxt("FATAL: field 'indexPermute' D.N.E. in class. Not an ImogenArray?\n");
  double *perm = mxGetPr(permArray);
  int offsetidx = 2*(perm[0]-1) + 1*(perm[1] > perm[2]);

  /* The offset array describes the index offsets for the data in the gpuStatics array */
  mxArray *offsets    = mxGetField(boundaryData, 0, "compOffset");
  if(offsets == NULL) mexErrMsgTxt("FATAL: field 'compOffset' D.N.E. in boundaryData. Not an ImogenArray? Statics not compiled?\n");
  double *offsetcount = mxGetPr(offsets);
  long int staticsOffset = (long int)offsetcount[2*offsetidx];
  int staticsNumel  = (int)offsetcount[2*offsetidx+1];

  /* Parameter describes what block size to launch with... */
  int blockdim = (int)*mxGetPr(prhs[1]);

  dim3 griddim; griddim.x = staticsNumel / blockdim + 1;
  if(griddim.x > 32768) {
    griddim.x = 32768;
    griddim.y = staticsNumel/(blockdim*griddim.x) + 1;
    }

  /* Every call results in applying specials */
  cukern_applySpecial_fade<<<griddim, blockdim>>>(array[0], statics[0] + staticsOffset, staticsNumel, amf.dim[0]);

  CHECK_CUDA_LAUNCH_ERROR(blockdim, griddim, &ama, 0, "cuda statics application");

  /* Indicates which part of a 3-vector this array is (0 = scalar, 123=XYZ) */
  int vectorComponent = (int)(*mxGetPr(mxGetProperty(prhs[0], 0, "component")) );
  

  /* BEGIN DETERMINATION OF ANALYTIC BOUNDARY CONDITIONS */
  int numDirections = mxGetNumberOfElements(prhs[2]);
  if(numDirections > 3) {
    mexErrMsgTxt("More than 3 directions specified to apply boundary conditions to. We only have 3...?\n");
    }
  double *directionToSet = mxGetPr(prhs[2]);

  mxArray *bcModes = mxGetField(boundaryData, 0, "bcModes");
  if(bcModes == NULL) mexErrMsgTxt("FATAL: bcModes structure not present. defective class detected.\n");

  int j;
  for(j = 0; j < numDirections; j++) {
    if((int)directionToSet[j] == 0) continue; /* Skips edge BCs if desired. */
    int trueDirect = (int)perm[(int)directionToSet[j]-1];

    /* So this is kinda dain-bramaged, but the boundary condition modes are stored in the form
       { 'type minus x', 'type minus y', 'type minus z';
         'type plus  x', 'type plus y',  'type plus z'};
       Yes, strings in a cell array. But hey, you can totally read that off by eye if you're
       in Matlab debug mode and the desire to print it out strikes you. */

    mxArray *bcstr; char *bs;
    
    int d; for(d = 0; d < 2; d++) {
      bcstr = mxGetCell(bcModes, 2*(trueDirect-1) + d);
      bs = (char *)malloc(sizeof(char) * (mxGetNumberOfElements(bcstr)+1));
      mxGetString(bcstr, bs, mxGetNumberOfElements(bcstr)+1);

      if(strcmp(bs, "mirror") == 0)
        setBoundarySAS(array[0], &ama, d, (int)directionToSet[j], vectorComponent == trueDirect);
       
      if(strcmp(bs, "const") == 0) {
//      ...
      }
      if(strcmp(bs, "linear") == 0) {
//      ...
      }
      
    }
  }

  free(array);
  free(statics);

}

/* Sets the given array+AMD's boundary in the following manner:
   side      -> 0 = negative edge  1 = positive edge
   direction -> 1 = X              2 = Y               3 = Z*
   sas       -> 0 = symmetric      1 => antisymmetric

   *: As passed, assuming ImogenArray's indexPermute has been handled for us.
   */

void setBoundarySAS(double *gpuarray, ArrayMetadata *amd, int side, int direction, int sas)
{
dim3 blockdim, griddim;

void *kerntable[12] = {(void *)&cukern_xminusSymmetrize, \
                       (void *)&cukern_xminusAntisymmetrize, \
		       (void *)&cukern_xplusSymmetrize, \
		       (void *)&cukern_xplusAntisymmetrize,
                       (void *)&cukern_yminusSymmetrize, \
                       (void *)&cukern_yminusAntisymmetrize, \
                       (void *)&cukern_yplusSymmetrize, \
                       (void *)&cukern_yplusAntisymmetrize,
                       (void *)&cukern_zminusSymmetrize, \
                       (void *)&cukern_zminusAntisymmetrize, \
                       (void *)&cukern_zplusSymmetrize, \
                       (void *)&cukern_zplusAntisymmetrize };

void (* bckernel)(double *, int, int, int) = (void (*)(double *, int, int, int))kerntable[sas + 2*side + 4*(direction-1)];

switch(direction) {
  case 1: {
    blockdim.x = 3;
    blockdim.y = 16;
    blockdim.z = 16;
    griddim.x = amd->dim[1] / blockdim.y; griddim.x += (griddim.x*blockdim.y < amd->dim[1]);
    griddim.y = amd->dim[2] / blockdim.z; griddim.y += (griddim.y*blockdim.z < amd->dim[2]);
    }; break;
  case 2: {
    blockdim.x = 16;
    blockdim.y = 1;
    blockdim.z = 16;
    griddim.x = amd->dim[0] / blockdim.x; griddim.x += (griddim.x*blockdim.x < amd->dim[0]);
    griddim.y = amd->dim[2] / blockdim.z; griddim.y += (griddim.y*blockdim.z < amd->dim[2]);
    } break;
  case 3: {
    blockdim.x = 16;
    blockdim.y = 16;
    blockdim.z = 1;
    griddim.x = amd->dim[0] / blockdim.x; griddim.x += (griddim.x*blockdim.x < amd->dim[0]);
    griddim.y = amd->dim[1] / blockdim.y; griddim.y += (griddim.y*blockdim.y < amd->dim[1]);
    } break;
  }

bckernel<<<griddim, blockdim>>>(gpuarray, amd->dim[0], amd->dim[1], amd->dim[2]);

CHECK_CUDA_LAUNCH_ERROR(blockdim, griddim, amd, sas + 2*side + 4*direction, "In setBoundarySAS; integer -> cukern table index");

return;
}

__global__ void cukern_applySpecial_fade(double *phi, double *statics, int nSpecials, int blkOffset)
{
int myAddr = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x*blockIdx.y);
if(myAddr >= nSpecials) return;

long int xaddr = (long int)statics[myAddr];
double f0      =           statics[myAddr + blkOffset];
double c       =           statics[myAddr + blkOffset*2];

phi[xaddr] = f0*c + (1.0-c)*phi[xaddr];

}

/* X DIRECTION SYMMETRIC/ANTISYMMETRIC BC KERNELS FOR MIRROR BCS */
/* Assume a block size of [3 A B] with grid dimensions [M N 1] s.t. AM >= ny, BN >= nz*/
/* Define the preamble common to all of these kernels: */
#define XSASKERN_PREAMBLE \
int stridey = nx; int stridez = nx*ny; \
int yidx = threadIdx.y + blockIdx.x*blockDim.y; \
int zidx = threadIdx.z + blockIdx.y*blockDim.z; \
if(yidx >= ny) return; if(zidx >= nz) return;


__global__ void cukern_xminusSymmetrize(double *phi, int nx, int ny, int nz)
{
XSASKERN_PREAMBLE

phi += stridey*yidx + stridez*zidx;
phi[2-threadIdx.x] = phi[4+threadIdx.x];
}

__global__ void cukern_xminusAntisymmetrize(double *phi, int nx, int ny, int nz)
{
XSASKERN_PREAMBLE

phi += stridey*yidx + stridez*zidx;
phi[2-threadIdx.x] = -phi[4+threadIdx.x];
}

__global__ void cukern_xplusSymmetrize(double *phi, int nx, int ny, int nz)
{
XSASKERN_PREAMBLE

phi += stridey*yidx + stridez*zidx + nx - 7;
phi[4+threadIdx.x] = phi[2-threadIdx.x];
}

__global__ void cukern_xplusAntisymmetrize(double *phi, int nx, int ny, int nz)
{
XSASKERN_PREAMBLE

phi += stridey*yidx + stridez*zidx + nx - 7;
phi[4+threadIdx.x] = -phi[2-threadIdx.x];
}

/* Y DIRECTION SYMMETRIC/ANTISYMMETRIC BC KERNELS */
/* assume a block size of [A 1 B] with grid dimensions [M N 1] s.t. AM >= nx, BN >=nz */
#define YSASKERN_PREAMBLE \
int xidx = threadIdx.x + blockIdx.x*blockDim.x; \
int zidx = threadIdx.z + blockIdx.y*blockDim.y; \
if(xidx >= nx) return; if(zidx >= nz) return;   \
phi += nx*ny*zidx; 

__global__ void cukern_yminusSymmetrize(double *phi, int nx, int ny, int nz)
{
YSASKERN_PREAMBLE
int q;
for(q = 0; q < 3; q++) { phi[xidx+nx*q] = phi[xidx+nx*(6-q)]; }
}

__global__ void cukern_yminusAntisymmetrize(double *phi, int nx, int ny, int nz)
{
YSASKERN_PREAMBLE
int q;
for(q = 0; q < 3; q++) { phi[xidx+nx*q] = -phi[xidx+nx*(6-q)]; }
}

__global__ void cukern_yplusSymmetrize(double *phi, int nx, int ny, int nz)
{
YSASKERN_PREAMBLE
int q;
for(q = 0; q < 3; q++) { phi[xidx-nx*q] = phi[xidx+nx*(q-6)]; }
}

__global__ void cukern_yplusAntisymmetrize(double *phi, int nx, int ny, int nz)
{
YSASKERN_PREAMBLE
int q;
for(q = 0; q < 3; q++) { phi[xidx-nx*q] = -phi[xidx+nx*(q-6)]; }
}

/* Z DIRECTION SYMMETRIC/ANTISYMMETRIC BC KERNELS */
/* Assume launch with size [A B 1] and grid of size [M N 1] s.t. AM >= nx, BN >= ny*/
#define ZSASKERN_PREAMBLE \
int xidx = threadIdx.x + blockIdx.x * blockDim.x; \
int yidx = threadIdx.y + blockIdx.y * blockDim.y; \
if(xidx >= nx) return; if(yidx >= ny) return; \
phi += xidx + nx*yidx;

__global__ void cukern_zminusSymmetrize(double *phi, int nx, int ny, int nz)
{
ZSASKERN_PREAMBLE

double p[3];
int stride = nx*ny;

p[0] = phi[4*stride];
p[1] = phi[5*stride];
p[2] = phi[6*stride];

phi[  0     ] = p[2];
phi[  stride] = p[1];
phi[2*stride] = p[0];
}

__global__ void cukern_zminusAntisymmetrize(double *phi, int nx, int ny, int nz)
{
ZSASKERN_PREAMBLE

double p[3];
int stride = nx*ny;

p[0] = phi[4*stride];
p[1] = phi[5*stride];
p[2] = phi[6*stride];

phi[  0     ] = -p[2];
phi[  stride] = -p[1];
phi[2*stride] = -p[0];
}

__global__ void cukern_zplusSymmetrize(double *phi, int nx, int ny, int nz)
{
ZSASKERN_PREAMBLE

double p[3];
int stride = nx*ny;

p[0] = phi[0];
p[1] = phi[stride];
p[2] = phi[2*stride];

phi[4*stride] = p[2];
phi[5*stride] = p[1];
phi[6*stride] = p[0];
}

__global__ void cukern_zplusAntisymmetrize(double *phi, int nx, int ny, int nz)
{
ZSASKERN_PREAMBLE

double p[3];
int stride = nx*ny;

p[0] = phi[0];
p[1] = phi[stride];
p[2] = phi[2*stride];

phi[4*stride] = -p[2];
phi[5*stride] = -p[1];
phi[6*stride] = -p[0];

}


