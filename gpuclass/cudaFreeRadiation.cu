#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cudaCommon.h"

/* THIS FUNCTION
   cudaFreeRadiation performs a purely local update to energy density of the form

   E = E - dt * beta rho^(2-theta) Pgas^(theta),

   i.e. a sink on the energy term of

   Lambda = beta rho^(2-theta) Pgas^(theta)

   where E is the total energy density, dt the time to pass, beta the radiation strength scale
   factor, rho the mass density, Pgas the thermal pressure, and theta parameterizes the
   radiation (nonrelativistic bremsstrahlung is theta = 0.5)

   It implements a temperature floor (Lambda = 0 for T < T_critical) and checks for negative
   energy density both before (safety) and after (time accuracy truncation) the physics.
*/

__global__ void cukern_FreeHydroRadiationRate(double *rho, double *px, double *py, double *pz, double *E, double *radrate, int numel);
__global__ void cukern_FreeMHDRadiationRate(double *rho, double *px, double *py, double *pz, double *E, double *bx, double *by, double *bz, double *radrate, int numel);

__global__ void cukern_FreeHydroRadiation(double *rho, double *px, double *py, double *pz, double *E, int numel);
__global__ void cukern_FreeMHDRadiation(double *rho, double *px, double *py, double *pz, double *E, double *bx, double *by, double *bz, int numel);

__constant__ __device__ double radparam[5];
#define GAMMA_M1 radparam[0]
#define STRENGTH radparam[1]
#define EXPONENT radparam[2]
#define TWO_MEXPONENT radparam[3]
#define TFLOOR radparam[4]


#define BLOCKDIM 256
#define GRIDDIM 64

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  if ((nrhs != 9) || (nlhs > 1))
     mexErrMsgTxt("Wrong number of arguments. Expected forms: rate = cudaFreeRadiation(rho, px, py, pz, E, bx, by, bz, [gamma theta beta*dt Tmin isPureHydro]) or cudaFreeRadiation(rho, px, py, pz, E, bx, by , bz, [gamma theta beta*dt Tmin isPureHydro]\n");

  double gam       = (mxGetPr(prhs[8]))[0];
  double exponent  = (mxGetPr(prhs[8]))[1];
  double strength  = (mxGetPr(prhs[8]))[2];
  double minTemp   = (mxGetPr(prhs[8]))[3];
  int isHydro= ((int)(mxGetPr(prhs[8]))[4]) != 0;

  ArrayMetadata amd;
  double **arrays = getGPUSourcePointers(prhs, &amd, 0, 4);
  CHECK_CUDA_ERROR("Entering cudaFreeRadiation");

  double **dest = NULL;
  if(nlhs == 1) {
    dest = makeGPUDestinationArrays(&amd, plhs, 1);
    }

  double hostRP[5];
  hostRP[0] = gam-1.0;
  hostRP[1] = strength;
  hostRP[2] = exponent;
  hostRP[3] = 2.0 - exponent;
  hostRP[4] = minTemp;
  hipMemcpyToSymbol(HIP_SYMBOL(radparam), hostRP, 5*sizeof(double), 0, hipMemcpyHostToDevice);

  switch(isHydro + 2*nlhs) {
    case 0: {
      double **B = getGPUSourcePointers(prhs, &amd, 5, 7);
      cukern_FreeMHDRadiation<<<GRIDDIM, BLOCKDIM>>>(arrays[0], arrays[1], arrays[2], arrays[3], arrays[4], B[0], B[1], B[2], amd.numel);
      free(B);
      break; }
    case 1: {
      cukern_FreeHydroRadiation<<<GRIDDIM, BLOCKDIM>>>(arrays[0], arrays[1], arrays[2], arrays[3], arrays[4], amd.numel);
      break; }
    case 2: {
      double **B = getGPUSourcePointers(prhs, &amd, 5, 7);
      cukern_FreeMHDRadiationRate<<<GRIDDIM, BLOCKDIM>>>(arrays[0], arrays[1], arrays[2], arrays[3], arrays[4], B[0], B[1], B[2], dest[0], amd.numel);
      free(B);
      break; }
    case 3: {
      cukern_FreeHydroRadiationRate<<<GRIDDIM, BLOCKDIM>>>(arrays[0], arrays[1], arrays[2], arrays[3], arrays[4], dest[0], amd.numel);
      break; }
    }

free(arrays);

CHECK_CUDA_LAUNCH_ERROR(BLOCKDIM, GRIDDIM, &amd, 666, "cudaFreeGasRadiation");

}


#define PSQUARED px[x]*px[x]+py[x]*py[x]+pz[x]*pz[x]
#define BSQUARED bx[x]*bx[x]+by[x]*by[x]+bz[x]*bz[x]

__global__ void cukern_FreeHydroRadiation(double *rho, double *px, double *py, double *pz, double *E, int numel)
{
int x = threadIdx.x + BLOCKDIM*blockIdx.x;

double P; double dE; double den;

while(x < numel) {
  den = rho[x];
  P = GAMMA_M1*(E[x] - (PSQUARED)/(2*den)); // gas pressure
  dE = STRENGTH*pow(rho[x], TWO_MEXPONENT)*pow(P, EXPONENT); // amount to be lost
  if(P - (GAMMA_M1*dE) < den*TFLOOR) { E[x] -= (P-den*TFLOOR)/GAMMA_M1; } else { E[x] -= dE; }

  x += BLOCKDIM*GRIDDIM;
  }

}

__global__ void cukern_FreeMHDRadiation(double *rho, double *px, double *py, double *pz, double *E, double *bx, double *by, double *bz, int numel)
{
int x = threadIdx.x + BLOCKDIM*blockIdx.x;

double P, dE, den;

while(x < numel) {
  den = rho[x];
  P = GAMMA_M1*(E[x] - (  (PSQUARED)/den + (BSQUARED))/2.0); // gas pressure
  dE = STRENGTH*pow(rho[x], TWO_MEXPONENT)*pow(P, EXPONENT);
  if(P - (GAMMA_M1 * dE) < den*TFLOOR) { E[x] -= (P-den*TFLOOR)/GAMMA_M1; } else { E[x] -= dE; }

  x += BLOCKDIM*GRIDDIM;
  }

}

__global__ void cukern_FreeHydroRadiationRate(double *rho, double *px, double *py, double *pz, double *E, double *radrate, int numel)
{
int x = threadIdx.x + BLOCKDIM*blockIdx.x;

double P;
while(x < numel) {
  P = GAMMA_M1*(E[x] - (PSQUARED)/(2*rho[x])); // gas pressure
  radrate[x] = pow(rho[x], TWO_MEXPONENT)*pow(P, EXPONENT);

  x += BLOCKDIM*GRIDDIM;
  }

}

__global__ void cukern_FreeMHDRadiationRate(double *rho, double *px, double *py, double *pz, double *E, double *bx, double *by, double *bz, double *radrate, int numel)
{
int x = threadIdx.x + BLOCKDIM*blockIdx.x;

double P;
while(x < numel) {
  P = GAMMA_M1*(E[x] - (  (PSQUARED)/rho[x] + (BSQUARED))/2.0); // gas pressure
  radrate[x] = pow(rho[x], TWO_MEXPONENT)*pow(P, EXPONENT);

  x += BLOCKDIM*GRIDDIM;
  }

}

