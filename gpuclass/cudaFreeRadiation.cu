#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cudaCommon.h"

/* THIS FUNCTION
   cudaFreeRadiation solves the operator equation
   d/dt E = -beta rho^2 T^theta


   Lambda = beta rho^(2-theta) Pgas^(theta)

   where E is the total energy density, dt the time to pass, beta the radiation strength scale
   factor, rho the mass density, Pgas the thermal pressure, and theta parameterizes the
   radiation (nonrelativistic bremsstrahlung is theta = 0.5)

   It implements a temperature floor (Lambda = 0 for T < T_critical) and checks for negative
   energy density both before (safety) and after (time accuracy truncation) the physics.
*/

__global__ void cukern_FreeHydroRadiationRate(double *rho, double *px, double *py, double *pz, double *E, double *radrate, int numel);
__global__ void cukern_FreeMHDRadiationRate(double *rho, double *px, double *py, double *pz, double *E, double *bx, double *by, double *bz, double *radrate, int numel);

template <unsigned int keyvalueOfTheta>
__global__ void cukern_FreeHydroRadiation(double *rho, double *px, double *py, double *pz, double *E, int numel);
__global__ void cukern_FreeMHDRadiation(double *rho, double *px, double *py, double *pz, double *E, double *bx, double *by, double *bz, int numel);

__constant__ __device__ double radparam[5];
#define GAMMA_M1 radparam[0]
#define STRENGTH radparam[1]
#define EXPONENT radparam[2]
#define TWO_MEXPONENT radparam[3]
#define TFLOOR radparam[4]

#define BLOCKDIM 256
#define GRIDDIM 64

// These and the freeHydroRadiation templating are because of the different
// integral outcomes when theta is exactly zero (P-independent) or one (logarithm outcome)
#define KEYVALUE_ZERO 0
#define KEYVALUE_ONE 1
#define KEYVALUE_NOT 2

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	if ((nrhs != 9) || (nlhs > 1))
		mexErrMsgTxt("Wrong number of arguments. Expected forms: rate = cudaFreeRadiation(rho, px, py, pz, E, bx, by, bz, [gamma theta beta*dt Tmin isPureHydro]) or cudaFreeRadiation(rho, px, py, pz, E, bx, by , bz, [gamma theta beta*dt Tmin isPureHydro]\n");

	double *inParams = mxGetPr(prhs[8]);

	double gam      = inParams[0];
	double exponent = inParams[1];
	double strength = inParams[2];
	double minTemp  = inParams[3];
	int isHydro     = (int)inParams[4] != 0;

	MGArray f[8];

	if( isHydro == false ) {
		accessMGArrays(prhs, 0, 7, &f[0]);
	} else {
		accessMGArrays(prhs, 0, 4, &f[0]);
	}

	MGArray *dest;
	if(nlhs == 1) {
		dest = createMGArrays(plhs, 1, &f[0]);
	}

	double hostRP[5];
	hostRP[0] = gam-1.0;
	hostRP[1] = strength;
	hostRP[2] = exponent;
	hostRP[3] = 2.0 - exponent;
	hostRP[4] = minTemp;

	int j, k;
	for(j = 0; j < f->nGPUs; j++) {
		hipSetDevice(f->deviceID[j]);
		hipMemcpyToSymbol(HIP_SYMBOL(radparam), hostRP, 5*sizeof(double), 0, hipMemcpyHostToDevice);
	}

	int sub[6];
	for(j = 0; j < f[0].nGPUs; j++) {
		calcPartitionExtent(&f[0], j, sub);
		hipSetDevice(f[0].deviceID[j]);

		double *ptrs[8];
		for(k = 0; k < 8; k++) { ptrs[k] = f[k].devicePtr[j]; }
		// Save some readability below...

		switch(isHydro + 2*nlhs) {
		case 0: {
			cukern_FreeMHDRadiation<<<GRIDDIM, BLOCKDIM>>>(ptrs[0], ptrs[1], ptrs[2], ptrs[3], ptrs[4], ptrs[5], ptrs[6], ptrs[7], f[0].partNumel[j]);
			break; }
		case 1: {
			cukern_FreeHydroRadiation<KEYVALUE_NOT><<<GRIDDIM, BLOCKDIM>>>(ptrs[0], ptrs[1], ptrs[2], ptrs[3], ptrs[4], f[0].partNumel[j]);
			break; }
		case 2: {
			cukern_FreeMHDRadiationRate<<<GRIDDIM, BLOCKDIM>>>(ptrs[0], ptrs[1], ptrs[2], ptrs[3], ptrs[4], ptrs[5], ptrs[6], ptrs[7], dest->devicePtr[j], f[0].partNumel[j]);
			break; }
		case 3: {
			cukern_FreeHydroRadiationRate<<<GRIDDIM, BLOCKDIM>>>(ptrs[0], ptrs[1], ptrs[2], ptrs[3], ptrs[4], dest->devicePtr[j], f[0].partNumel[j]);
			break; }
		}
	}

	if(nlhs == 1) free(dest);

	//CHECK_CUDA_LAUNCH_ERROR(BLOCKDIM, GRIDDIM, &amd, 666, "cudaFreeGasRadiation");

}

/* NOTE: This uses an explicit algorithm to perform radiation,
 * i.e. E[t+dt] = E[t] - Lambda[t] dt with radiation rate Lambda
 * This is conditionally stable with a CFL set by Lambda dt < E
 * 
 * Normally E / Lambda >> [dx / (c+max(Vx)) ], i.e. cooling time much
 * longer than advection time, but an implicit algorithm would be
 * wise as it is unconditionally stable. */
#define PSQUARED px[x]*px[x]+py[x]*py[x]+pz[x]*pz[x]
#define BSQUARED bx[x]*bx[x]+by[x]*by[x]+bz[x]*bz[x]
template <unsigned int keyvalueOfTheta>
__global__ void cukern_FreeHydroRadiation(double *rho, double *px, double *py, double *pz, double *E, int numel)
{
	int x = threadIdx.x + BLOCKDIM*blockIdx.x;

	int i;
	double P, Pf, beta, den;

	while(x < numel) {
		den = rho[x];
		P = GAMMA_M1*(E[x] - (PSQUARED)/(2*den)); // gas pressure

		// Do nothing if temperature too low
		if(P > den*TFLOOR) { 
		    switch(keyvalueOfTheta) {
			case KEYVALUE_ZERO: // Special case - analytic: dE = -strength*rho^2 dt
			    Pf = P - GAMMA_M1*STRENGTH*den*den; break;
			case KEYVALUE_ONE: // Special case - analytic: dE = -strength rho P dt
			    Pf = exp(log(P) - GAMMA_M1*STRENGTH*den); break;
			case KEYVALUE_NOT: // General case - dE/dt = -strength rho^(2-theta) P^theta
			    beta = .5*STRENGTH*pow(den, TWO_MEXPONENT)*GAMMA_M1;
			    // Explicit prediction
			    Pf = P - 2*beta*pow(P, EXPONENT);
			    // Some newton-raphson to finish it off
			    for(i = 0; i < 4; i++) {
				Pf -= (Pf - P + beta*(pow(Pf,EXPONENT) + pow(P, EXPONENT)))/(1+beta*EXPONENT*pow(Pf,EXPONENT-1.0));
			    }
		    }
		Pf = (Pf > den*TFLOOR) ? Pf : den*TFLOOR;

		E[x] += (Pf-P)/GAMMA_M1;
		
		}
		// Cell completely cooled during this timestep
//		if(P > den*TFLOOR) {
//			if(P - (GAMMA_M1*dE) < den*TFLOOR) { E[x] -= (P-den*TFLOOR)/GAMMA_M1; } else { E[x] -= dE; } }

		x += BLOCKDIM*GRIDDIM;
	}

}

// STRENGTH = beta*dt
__global__ void cukern_FreeMHDRadiation(double *rho, double *px, double *py, double *pz, double *E, double *bx, double *by, double *bz, int numel)
{
	int x = threadIdx.x + BLOCKDIM*blockIdx.x;

	double P, dE, den;

	while(x < numel) {
		den = rho[x];
		P = GAMMA_M1*(E[x] - (  (PSQUARED)/den + (BSQUARED))/2.0); // gas pressure
		dE = STRENGTH*pow(den, TWO_MEXPONENT)*pow(P, EXPONENT);
		if(P > den*TFLOOR) {
			if(P - (GAMMA_M1 * dE) < den*TFLOOR) { E[x] -= (P-den*TFLOOR)/GAMMA_M1; } else { E[x] -= dE; } }

		x += BLOCKDIM*GRIDDIM;
	}

}

/* These functions return the instantaneous rate, strictly the first derivative e_t */
__global__ void cukern_FreeHydroRadiationRate(double *rho, double *px, double *py, double *pz, double *E, double *radrate, int numel)
{
	int x = threadIdx.x + BLOCKDIM*blockIdx.x;

	double P;
	while(x < numel) {
		P = GAMMA_M1*(E[x] - (PSQUARED)/(2*rho[x])); // gas pressure
		radrate[x] = pow(rho[x], TWO_MEXPONENT)*pow(P, EXPONENT);

		x += BLOCKDIM*GRIDDIM;
	}

}

__global__ void cukern_FreeMHDRadiationRate(double *rho, double *px, double *py, double *pz, double *E, double *bx, double *by, double *bz, double *radrate, int numel)
{
	int x = threadIdx.x + BLOCKDIM*blockIdx.x;

	double P;
	while(x < numel) {
		P = GAMMA_M1*(E[x] - (  (PSQUARED)/rho[x] + (BSQUARED))/2.0); // gas pressure
		radrate[x] = pow(rho[x], TWO_MEXPONENT)*pow(P, EXPONENT);

		x += BLOCKDIM*GRIDDIM;
	}

}

