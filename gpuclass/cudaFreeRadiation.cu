#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cudaCommon.h"

__global__ void cukern_FreeHydroRadiationRate(double *rho, double *px, double *py, double *pz, double *E, double *radrate, int numel);
__global__ void cukern_FreeMHDRadiationRate(double *rho, double *px, double *py, double *pz, double *E, double *bx, double *by, double *bz, double *radrate, int numel);

__global__ void cukern_FreeHydroRadiation(double *rho, double *px, double *py, double *pz, double *E, int numel);
__global__ void cukern_FreeMHDRadiation(double *rho, double *px, double *py, double *pz, double *E, double *bx, double *by, double *bz, int numel);

__constant__ __device__ double radparam[5];
#define GAMMA_M1 radparam[0]
#define STRENGTH radparam[1]
#define EXPONENT radparam[2]
#define TWO_MEXPONENT radparam[3]
#define PFLOOR radparam[4]


#define BLOCKDIM 256
#define GRIDDIM 64

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  if ((nrhs != 9) || (nlhs > 1))
     mexErrMsgTxt("Wrong number of arguments. Expected forms: rate = cudaFreeRadiation(rho, px, py, pz, E, bx, by, bz, [gamma theta 0 isPureHydro]) or cudaFreeRadiation(rho, px, py, pz, E, bx, by , bz, [gamma theta beta*dt isPureHydro]\n");

  double gam       = (mxGetPr(prhs[8]))[0];
  double exponent  = (mxGetPr(prhs[8]))[1];
  double strength  = (mxGetPr(prhs[8]))[2];
  int isHydro   = ((int)(mxGetPr(prhs[8]))[3]) != 0;

  ArrayMetadata amd;
  double **arrays = getGPUSourcePointers(prhs, &amd, 0, 4);
  cudaCheckError("Entering cudaFreeRadiation");

  double **dest = NULL;
  if(nlhs == 1) {
    dest = makeGPUDestinationArrays(&amd, plhs, 1);
    }

  double hostRP[5];
  hostRP[0] = gam-1.0;
  hostRP[1] = strength;
  hostRP[2] = exponent;
  hostRP[3] = 2.0 - exponent;
  hostRP[4] = 1.0; // pressure floor = 1.0
  hipMemcpyToSymbol(HIP_SYMBOL(radparam), hostRP, 5*sizeof(double), 0, hipMemcpyHostToDevice);

  switch(isHydro + 2*nlhs) {
    case 0: {
      double **B = getGPUSourcePointers(prhs, &amd, 5, 7);
      cukern_FreeMHDRadiation<<<GRIDDIM, BLOCKDIM>>>(arrays[0], arrays[1], arrays[2], arrays[3], arrays[4], B[0], B[1], B[2], amd.numel);
      break; }
    case 1: {
      cukern_FreeHydroRadiation<<<GRIDDIM, BLOCKDIM>>>(arrays[0], arrays[1], arrays[2], arrays[3], arrays[4], amd.numel);
      break; }
    case 2: {
      double **B = getGPUSourcePointers(prhs, &amd, 5, 7);
      cukern_FreeMHDRadiationRate<<<GRIDDIM, BLOCKDIM>>>(arrays[0], arrays[1], arrays[2], arrays[3], arrays[4], B[0], B[1], B[2], dest[0], amd.numel);
      break; }
    case 3: {
      cukern_FreeHydroRadiationRate<<<GRIDDIM, BLOCKDIM>>>(arrays[0], arrays[1], arrays[2], arrays[3], arrays[4], dest[0], amd.numel);
      break; }
    }

hipError_t epicFail = hipGetLastError();
if(epicFail != hipSuccess) cudaLaunchError(epicFail, BLOCKDIM, GRIDDIM, &amd, 666, "cudaFreeGasRadiation");

}

__global__ void cukern_FreeHydroRadiation(double *rho, double *px, double *py, double *pz, double *E, int numel)
{
int x = threadIdx.x + BLOCKDIM*blockIdx.x;

double P; double dE;

while(x < numel) {
  P = GAMMA_M1*(E[x] - (px[x]*px[x]+py[x]*py[x]+pz[x]*pz[x])/(2*rho[x])); // gas pressure
  dE = STRENGTH*pow(rho[x], TWO_MEXPONENT)*pow(P, EXPONENT);
  if(P - (GAMMA_M1*dE) < PFLOOR) { E[x] -= (P-PFLOOR)/GAMMA_M1; } else { E[x] -= dE; }

  x += BLOCKDIM*GRIDDIM;
  }

}

__global__ void cukern_FreeMHDRadiation(double *rho, double *px, double *py, double *pz, double *E, double *bx, double *by, double *bz, int numel)
{
int x = threadIdx.x + BLOCKDIM*blockIdx.x;

double P;
double dE;
while(x < numel) {
  P = GAMMA_M1*(E[x] - (  (px[x]*px[x]+py[x]*py[x]+pz[x]*pz[x])/rho[x] +\
                           (bx[x]*bx[x]+by[x]*by[x]+bz[x]*bz[x]))/2.0); // gas pressure
  dE = STRENGTH*pow(rho[x], TWO_MEXPONENT)*pow(P, EXPONENT);
  if(P - (GAMMA_M1 * dE) < PFLOOR) { E[x] -= (P-PFLOOR)/GAMMA_M1; } else { E[x] -= dE; }

  x += BLOCKDIM*GRIDDIM;
  }

}

__global__ void cukern_FreeHydroRadiationRate(double *rho, double *px, double *py, double *pz, double *E, double *radrate, int numel)
{
int x = threadIdx.x + BLOCKDIM*blockIdx.x;

double P;
while(x < numel) {
  P = GAMMA_M1*(E[x] - (px[x]*px[x]+py[x]*py[x]+pz[x]*pz[x])/(2*rho[x])); // gas pressure
  radrate[x] = pow(rho[x], TWO_MEXPONENT)*pow(P, EXPONENT);

  x += BLOCKDIM*GRIDDIM;
  }

}

__global__ void cukern_FreeMHDRadiationRate(double *rho, double *px, double *py, double *pz, double *E, double *bx, double *by, double *bz, double *radrate, int numel)
{
int x = threadIdx.x + BLOCKDIM*blockIdx.x;

double P;
while(x < numel) {
  P = GAMMA_M1*(E[x] - (  (px[x]*px[x]+py[x]*py[x]+pz[x]*pz[x])/rho[x] +\
                           (bx[x]*bx[x]+by[x]+by[x]+bz[x]+bz[x]))/2.0); // gas pressure
  radrate[x] = pow(rho[x], TWO_MEXPONENT)*pow(P, EXPONENT);

  x += BLOCKDIM*GRIDDIM;
  }

}


//            gasPressure = pressure(ENUM.PRESSURE_GAS, run, mass, mom, ener, mag);
//            result      = obj.strength*mass.array.^(2 - obj.exponent) .* gasPressure.^obj.exponent;

