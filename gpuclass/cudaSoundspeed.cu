#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"

/* THIS FUNCTION:

   When not passed a magnetic field, calculates the local adiabatic sound speed of the fluid at 
   all points,

   c_s^2 = gamma*P/rho

   where c_s is the adiabatic sound speed, gamma is the adiabatic index (1 <= gamma <= 5/3), P
   is the thermal pressure (gamma-1)*(Etotal - rho v^2/2) and rho is the matter density.

   When passed a magnetic field, calculates the maximal fast Alfven velocity,
   C_fast^2 = C_s^2 + C_a^2,

   where C_s is the thermal sound speed above (Except subtracting magnetic energy density from
   the total energy as well) and C_a is the Alfven speed,

   C_a^2 = (B^2)/rho.
*/

__global__ void cukern_Soundspeed_mhd(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *dout, int n);
__global__ void cukern_Soundspeed_hd(double *rho, double *E, double *px, double *py, double *pz, double *dout, int n);

#define BLOCKDIM 256

__constant__ double pressParams[6];
#define MHD_CS_B pressParams[0]
#define GG1 pressParams[1]

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  // Determine appropriate number of arguments for RHS
  if( (nlhs != 1) || ( (nrhs != 9) && (nrhs != 6) ))
    mexErrMsgTxt("calling form for cudaSoundspeed is c_s = cudaSoundspeed(mass, ener, momx, momy, momz, [bx, by, bz,] gamma);");

  CHECK_CUDA_ERROR("entering cudaSoundspeed");

  dim3 blocksize; blocksize.x = BLOCKDIM; blocksize.y = blocksize.z = 1;
  ArrayMetadata amd;
  dim3 gridsize;

  // Select the appropriate kernel to invoke
    int pureHydro = (nrhs == 6);

    double gam; double **srcs;

    if(pureHydro == 1) {
      gam = *mxGetPr(prhs[5]);
      srcs = getGPUSourcePointers(prhs, &amd, 0, 4);
      } else {
      gam = *mxGetPr(prhs[8]);
      srcs = getGPUSourcePointers(prhs, &amd, 0, 7);
      }

    gridsize.x = BLOCKDIM;
    gridsize.y = gridsize.z =1;
    double **destPtr = makeGPUDestinationArrays(&amd, plhs, 1);
    double gg1 = gam*(gam-1);

    double hostP[6];
    hostP[0] = ALFVEN_CSQ_FACTOR - .5*gg1;
    hostP[1] = gg1;
    
    hipMemcpyToSymbol(HIP_SYMBOL(pressParams), &hostP[0], 6*sizeof(double), 0, hipMemcpyHostToDevice);

    if(pureHydro == 1) {
      cukern_Soundspeed_hd<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], destPtr[0], amd.numel);
      } else {
      cukern_Soundspeed_mhd<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], srcs[5], srcs[6], srcs[7], destPtr[0], amd.numel);
      }

    CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, &amd, nrhs, "cuda sound speed");

    free(destPtr);
}

// THIS KERNEL CALCULATES SOUNDSPEED IN THE MHD CASE, TAKEN AS THE FAST MA SPEED
// We increase the Alfven contribution to stabilize the code
__global__ void cukern_Soundspeed_mhd(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *dout, int n)
{

int x = threadIdx.x + blockIdx.x * BLOCKDIM;
int dx = blockDim.x * gridDim.x;
double csq, T, Bsq;
double invrho;

while(x < n) {
    invrho = 1.0 / rho[x];
    T = .5*(px[x]*px[x] + py[x]*py[x] + pz[x]*pz[x])*invrho;
    Bsq = bx[x]*bx[x] + by[x]*by[x] + bz[x]*bz[x];

    // MHD_CS_B is (alfven constant A) - .5(gamma)(gamma-1), where A is physically 1
    // but may be increased beyond 1 to stabilize simulations where low-beta conditions occur
    csq = (GG1*(E[x] - T) + MHD_CS_B * Bsq ) * invrho ;
    if(csq < 0.0) csq = 0.0;
    dout[x] = sqrt(csq);
    x += dx;
    }

}

// THIS KERNEL CALCULATES SOUNDSPEED IN THE HYDRODYNAMIC CASE
__global__ void cukern_Soundspeed_hd(double *rho, double *E, double *px, double *py, double *pz, double *dout, int n)
{
int x = threadIdx.x + blockIdx.x * BLOCKDIM;
int dx = blockDim.x * gridDim.x;
double csq, rhoinv;

while(x < n) {
	rhoinv = 1/rho[x];
    csq = GG1*(E[x] - .5*(px[x]*px[x] + py[x]*py[x] + pz[x]*pz[x])*rhoinv)*rhoinv;
    // Imogen's energy flux is unfortunately not positivity preserving
    if(csq < 0.0) csq = 0.0;
    dout[x] = sqrt(csq);
    x += dx;
    }

}


