#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"

__global__ void cukern_Soundspeed_mhd(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *dout, double gam, int n);
__global__ void cukern_Soundspeed_hd(double *rho, double *E, double *px, double *py, double *pz, double *dout, double gam, int n);

#define BLOCKDIM 256

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  // Determine appropriate number of arguments for RHS
  if( (nlhs != 1) || ( (nrhs != 9) && (nrhs != 6) ))
    mexErrMsgTxt("calling form for cudaSoundspeed is c_s = cudaSoundspeed(mass, ener, momx, momy, momz, bx, by, bz, gamma);");

  cudaCheckError("entering cudaSoundspeed");

  dim3 blocksize; blocksize.x = BLOCKDIM; blocksize.y = blocksize.z = 1;
  ArrayMetadata amd;
  dim3 gridsize;

  // Select the appropriate kernel to invoke
    int pureHydro = (nrhs == 6);

    double gam; double **srcs;

    if(pureHydro == 1) {
      gam = *mxGetPr(prhs[5]);
      srcs = getGPUSourcePointers(prhs, &amd, 0, 4);
      } else {
      gam = *mxGetPr(prhs[8]);
      srcs = getGPUSourcePointers(prhs, &amd, 0, 7);
      }

    gridsize.x = BLOCKDIM;
    gridsize.y = gridsize.z =1;
    double **destPtr = makeGPUDestinationArrays((int64_t *)mxGetData(prhs[1]), plhs, 1);
    double gg1 = gam*(gam-1);
    
    if(pureHydro == 1) {
      cukern_Soundspeed_hd<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], destPtr[0], gg1, amd.numel);
      } else {
      cukern_Soundspeed_mhd<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], srcs[5], srcs[6], srcs[7], destPtr[0], gg1, amd.numel);
      }

    hipError_t epicFail = hipGetLastError();
    if(epicFail != hipSuccess) cudaLaunchError(epicFail, blocksize, gridsize, &amd, nrhs, "cuda sound speed");


    free(destPtr);


}

// THIS KERNEL CALCULATES SOUNDSPEED IN THE MHD CASE, TAKEN AS THE FAST MA SPEED
__global__ void cukern_Soundspeed_mhd(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *dout, double gg1, int n)
{

int x = threadIdx.x + blockIdx.x * BLOCKDIM;
int dx = blockDim.x * gridDim.x;
double csq;
double invrho = 1.0 / rho[x];

while(x < n) {
//  csq = ( (gg1*(E[x] - .5*(px[x]*px[x] + py[x]*py[x] + pz[x]*pz[x])/rho[x]) + (2.0 -.5*gg1)*(bx[x]*bx[x] + by[x]*by[x] + bz[x]*bz[x]))/rho[x] );
    csq = (gg1*(E[x] - .5*(px[x]*px[x] + py[x]*py[x] + pz[x]*pz[x])*invrho ) + (4 - .5*gg1)*(bx[x]*bx[x] + by[x]*by[x] + bz[x]*bz[x])) * invrho ;
    if(csq < 0.0) csq = 0.0;
    dout[x] = sqrt(csq);
    x += dx;
    }

}

// THIS KERNEL CALCULATES SOUNDSPEED IN THE HYDRODYNAMIC CASE
__global__ void cukern_Soundspeed_hd(double *rho, double *E, double *px, double *py, double *pz, double *dout, double gg1, int n)
{
int x = threadIdx.x + blockIdx.x * BLOCKDIM;
int dx = blockDim.x * gridDim.x;
double csq;

while(x < n) {
    csq = gg1*(E[x] - .5*(px[x]*px[x] + py[x]*py[x] + pz[x]*pz[x]))/rho[x];
    // Imogen's energy flux is unfortunately not positivity preserving
    if(csq < 0.0) csq = 0.0;
    dout[x] = sqrt(csq);
    x += dx;
    }

}


