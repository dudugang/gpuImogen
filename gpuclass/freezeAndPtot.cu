#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"

__global__ void cukern_FreezeSpeed_mhd(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double gam, double *freeze, double *ptot, int nx);
__global__ void cukern_FreezeSpeed_hydro(double *rho, double *E, double *px, double *py, double *pz, double gam, double *freeze, double *ptot, int nx);

#define BLOCKDIM 64
#define MAXPOW   5

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {

  // At least 2 arguments expected
  // Input and result
  if ( (nrhs!=10) && (nrhs!=2))
     mexErrMsgTxt("Wrong number of arguments. Call using [ptot freeze] = FreezeAndPtot(mass, ener, momx, momy, momz, bz, by, bz, gamma, 1)");

  // Get GPU array pointers
  int direction = (int)*mxGetPr(prhs[9]);

  ArrayMetadata amd;
  double **args = getGPUSourcePointers(prhs, &amd, 0, 7);

  dim3 arraySize;
  arraySize.x = amd.dim[0];
  arraySize.y = amd.dim[1];
  arraySize.z = amd.dim[2];
  dim3 blocksize, gridsize;

  blocksize.x = BLOCKDIM; blocksize.y = blocksize.z = 1;
  gridsize.x = arraySize.y;
  gridsize.y = arraySize.z;

  double **ptot = makeGPUDestinationArrays((int64_t *)mxGetData(prhs[0]), plhs, 1); // ptotal array

  int64_t *oldref = (int64_t *)mxGetData(prhs[0]); 
  int64_t fref[5];
  fref[0] = 0;
  fref[1] = oldref[1] - 1;
  fref[2] = arraySize.y;
  fref[3] = arraySize.z;
  fref[4] = 1;

  double **freezea = makeGPUDestinationArrays(&fref[0], &plhs[1], 1); // freeze array

  int ispurehydro = (int)*mxGetPr(prhs[9]);

  if(ispurehydro) {
    cukern_FreezeSpeed_hydro<<<gridsize, blocksize>>>(args[0], args[1], args[2], args[3], args[4],  *mxGetPr(prhs[8]), freezea[0], ptot[0], arraySize.x);
//                                                   (*rho,    *E,      *px,     *py,     *pz,      gam,              *freeze,  *ptot,  nx)
  } else {
    cukern_FreezeSpeed_mhd<<<gridsize, blocksize>>>(args[0], args[1], args[2], args[3], args[4], args[5], args[6], args[7], *mxGetPr(prhs[8]), freezea[0], ptot[0], arraySize.x);
//                                                 (*rho,    *E,      *px,     *py,     *pz,     *bx,     *by,     *bz,     gam,              *freeze,  *ptot,  nx)
  }
  free(ptot);
  free(args);
  free(freezea);

}

__global__ void cukern_FreezeSpeed_mhd(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double gam, double *freeze, double *ptot, int nx)
{
/* gridDim = [ny nz], nx = nx */
int x = threadIdx.x + nx*(blockIdx.x + gridDim.x*blockIdx.y);
int addrMax = nx + nx*(blockIdx.x + gridDim.x*blockIdx.y);

double Cs, CsMax;
double psqhf, bsqhf;
double gg1 = gam*(gam-1.0);

__shared__ double locBloc[BLOCKDIM];

CsMax = 0.0;
locBloc[threadIdx.x] = 0.0;

if(x >= addrMax) return; // If we get a very low resolution

while(x < addrMax) {
  psqhf = .5*(px[x]*px[x]+py[x]*py[x]+pz[x]*pz[x]);
  bsqhf = .5*(bx[x]*bx[x]+by[x]*by[x]+bz[x]*bz[x]);
  // we calculate pressure.
  Cs = (gam-1.0)*(E[x] - psqhf/rho[x]) + (2.0-gam)*bsqhf;
  if(Cs > 0.0) { ptot[x] = Cs; } else { ptot[x] = 0.0; } // Enforce positive semi-definiteness

  Cs    = sqrt(abs( (gg1*(E[x] - psqhf/rho[x] - bsqhf) + 4*bsqhf)/rho[x] )) + abs(px[x]/rho[x]);
  if(Cs > CsMax) CsMax = Cs;

  x += BLOCKDIM;
  }

locBloc[threadIdx.x] = CsMax;

__syncthreads();

if (threadIdx.x % 8 > 0) return; // keep one in 8 threads

// Each searches the max of the nearest 8 points
for(x = 1; x < 8; x++) {
  if(locBloc[threadIdx.x+x] > locBloc[threadIdx.x]) locBloc[threadIdx.x] = locBloc[threadIdx.x+x];
  }

__syncthreads();

// The last thread takes the max of these maxes
if(threadIdx.x > 0) return;
for(x = 8; x < BLOCKDIM; x+= 8) {
  if(locBloc[threadIdx.x+x] > locBloc[0]) locBloc[0] = locBloc[threadIdx.x+x];
  }

freeze[blockIdx.x + gridDim.x*blockIdx.y] = locBloc[0];

}

__global__ void cukern_FreezeSpeed_hydro(double *rho, double *E, double *px, double *py, double *pz, double gam, double *freeze, double *ptot, int nx)
{
int x = threadIdx.x + nx*(blockIdx.x + gridDim.x*blockIdx.y);
int addrMax = nx + nx*(blockIdx.x + gridDim.x*blockIdx.y);

double Cs, CsMax;
double psqhf;
double gg1 = gam*(gam-1.0);

__shared__ double locBloc[BLOCKDIM];

CsMax = 0.0;
locBloc[threadIdx.x] = 0.0;

if(x >= addrMax) return; // If we get a very low resolution

while(x < addrMax) {
  psqhf = .5*(px[x]*px[x]+py[x]*py[x]+pz[x]*pz[x]);

  Cs = (gam-1.0)*(E[x] - psqhf/rho[x]);
  if(Cs > 0.0) { ptot[x] = Cs; } else { ptot[x] = 0.0; }

  Cs    = sqrt(abs( (gg1*(E[x] - psqhf/rho[x]) )/rho[x] )) + abs(px[x]/rho[x]);

  if(Cs > CsMax) CsMax = Cs;

  x += BLOCKDIM;
  }

locBloc[threadIdx.x] = CsMax;

__syncthreads();

if (threadIdx.x % 8 > 0) return; // keep one in 8 threads

// Each searches the max of the nearest 8 points
for(x = 1; x < 8; x++) {
  if(locBloc[threadIdx.x+x] > locBloc[threadIdx.x]) locBloc[threadIdx.x] = locBloc[threadIdx.x+x];
  }

// The last thread takes the max of these maxes
if(threadIdx.x > 0) return;
for(x = 8; x < BLOCKDIM; x+= 8) {
  if(locBloc[threadIdx.x+x] > locBloc[0]) locBloc[0] = locBloc[threadIdx.x+x];
  }

// NOTE: This is the dead-stupid backup if all else fails.
//if(threadIdx.x > 0) return;
//for(x = 1; x < GLOBAL_BLOCKDIM; x++)  if(locBloc[x] > locBloc[0]) locBloc[0] = locBloc[x];

freeze[blockIdx.x + gridDim.x*blockIdx.y] = locBloc[0];


}
