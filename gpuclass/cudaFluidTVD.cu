#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA^M
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cudaCommon.h"

/* THIS FUNCTION 

This is the Cuda Fluid TVD function; It takes a single forward-time step, CFD or MHD, of the
conserved-transport part of the fluid equations using a total variation diminishing scheme to
perform a non-oscillatory update.

Requires predicted half-step values from a 1st order upwind scheme.

*/

#define BLOCKLEN 60
#define BLOCKLENP2 62
#define BLOCKLENP4 64

/*__global__ void cukern_TVDStep_mhd_uniform(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *P, double *Cfreeze, double *rhoW, double *enerW, double *pxW, double *pyW, double *pzW, double lambda, int nx);*/
__global__ void cukern_TVDStep_mhd_uniform(double *P, double *Cfreeze, double halflambda, int nx);
/*__global__ void cukern_TVDStep_hydro_uniform(double *rho, double *E, double *px, double *py, double *pz, double *P, double *Cfreeze, double *rhoW, double *enerW, double *pxW, double *pyW, double *pzW, double lambdahf, int nx);*/
__global__ void cukern_TVDStep_hydro_uniform(double *P, double *Cfreeze, double halfLambda, int nx);

__device__ void cukern_FluxLimiter_VanLeer(double deriv[2][BLOCKLENP4], double flux[2][BLOCKLENP4], int who);
__device__ __inline__ double fluxLimiter_Vanleer(double derivL, double derivR);

__constant__ __device__ double *inputPointers[8];
__constant__ __device__ double *outputPointers[5];

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  // At least 2 arguments expected
  // Input and result
  if ((nrhs!=17) || (nlhs != 0)) mexErrMsgTxt("Wrong number of arguments: call cudaTVDStep(rho, E, px, py, pz, bx, by, bz, P, rho_out, E_out, px_out, py_out, pz_out, C_freeze, lambda, purehydro?)\n");

  // Get source array info and create destination arrays
  ArrayMetadata amd;

  // Get the source gpu arrays as enumerated in the error message
  double **srcs   = getGPUSourcePointers(prhs, &amd, 0, 13);

  // Get the GPU freeze speed. Differs in that it is not the same size
  ArrayMetadata fmd;
  double **gpu_cf = getGPUSourcePointers(prhs, &fmd, 14, 14); 

  // Get flux factor (dt / d) amd determine if we are doing the hydro case or the MHD case
  double lambda   = *mxGetPr(prhs[15]);
  int isPureHydro = (int)*mxGetPr(prhs[16]);

  // Do the usual rigamarole determining array sizes and GPU launch dimensions
  dim3 arraySize;
  arraySize.x = amd.dim[0];
  arraySize.y = amd.dim[1];
  arraySize.z = amd.dim[2];

  dim3 blocksize, gridsize;

  blocksize.x = BLOCKLEN+4;
  blocksize.y = blocksize.z = 1;

  gridsize.x = arraySize.y;
  gridsize.y = arraySize.z;

  // Invoke the kernel
  if(arraySize.x > 1) {
    if(isPureHydro) {
    //cukern_TVDStep_hydro_uniform                         (*rho,    *E,     *px,      *py,     *pz,     *P,      *Cfreeze, *rhoW,  *enerW,     *pxW,     *pyW,     *pzW,     lambda, nx);
      hipMemcpyToSymbol(HIP_SYMBOL(inputPointers),  srcs,     5*sizeof(double *), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(outputPointers), &srcs[9], 5*sizeof(double *), 0, hipMemcpyHostToDevice);
      cukern_TVDStep_hydro_uniform<<<gridsize, blocksize>>>(srcs[8], *gpu_cf, .5*lambda, arraySize.x);
    } else {
    //cukern_TVDStep_mhd_uniform                         (*rho,    *E,      *px,     *py,     *pz,     *bx,     *by,     *bz,     *P,           *Cfreeze, *rhoW,  *enerW,   *pxW,     *pyW,     *pzW, double lambda, int nx);
      hipMemcpyToSymbol(HIP_SYMBOL(inputPointers),  srcs,     8*sizeof(double *), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(outputPointers), &srcs[9], 5*sizeof(double *), 0, hipMemcpyHostToDevice);      
      cukern_TVDStep_mhd_uniform  <<<gridsize, blocksize>>>(srcs[8], *gpu_cf, .5*lambda, arraySize.x);
    }
  }

hipError_t epicFail = hipGetLastError();
if(epicFail != hipSuccess) cudaLaunchError(epicFail, blocksize, gridsize, &amd, isPureHydro, "fluid TVD");

}

/* blockidx.{xy} is our index in {yz}, and gridDim.{xy} gives the {yz} size */
/* Expect invocation with n+4 threads */
__global__ void cukern_TVDStep_mhd_uniform(double *P, double *Cfreeze, double halfLambda, int nx)
{
double c_f, velocity;
double q_i[5];
double b_i[3];
double w_i;
__shared__ double fluxLR[2][BLOCKLENP4];
__shared__ double derivLR[2][BLOCKLENP4];

/* Step 0 - obligatory annoying setup stuff (ASS) */
int I0 = nx*(blockIdx.x + gridDim.x * blockIdx.y);
int Xindex = (threadIdx.x-2);
int Xtrack = Xindex;
Xindex += nx*(threadIdx.x < 2);

int x; /* = Xindex % nx; */
int i;
bool doIflux = (threadIdx.x > 1) && (threadIdx.x < BLOCKLEN+2);

/* Step 1 - calculate W values */
c_f = Cfreeze[blockIdx.x + gridDim.x * blockIdx.y];

while(Xtrack < nx+2) {
    x = I0 + (Xindex % nx);

    q_i[0] = inputPointers[0][x];
    q_i[1] = inputPointers[1][x];       /* So we avoid multiple loops */
    q_i[2] = inputPointers[2][x];      /* over them inside the flux loop */
    q_i[3] = inputPointers[3][x];
    q_i[4] = inputPointers[4][x];
    b_i[0] = inputPointers[5][x];
    b_i[1] = inputPointers[6][x];
    b_i[2] = inputPointers[7][x];
    velocity = q_i[2]/q_i[0];

    /* rho, E, px, py, pz going down */
    /* Iterate over variables to flux */
    for(i = 0; i < 5; i++) {
        /* Step 1 - Calculate raw fluxes */
        switch(i) {
            case 0: w_i = q_i[2]; break;
            case 1: w_i = (velocity * (q_i[1] + P[x]) - b_i[0]*(q_i[2]*b_i[0]+q_i[3]*b_i[1]+q_i[4]*b_i[2])/q_i[0] ); break;
            case 2: w_i = (velocity*q_i[2] + P[x] - b_i[0]*b_i[0]); break;
            case 3: w_i = (velocity*q_i[3]        - b_i[0]*b_i[1]); break;
            case 4: w_i = (velocity*q_i[4]        - b_i[0]*b_i[2]); break;
            }

        /* Step 2 - Decouple to L/R flux */
        fluxLR[0][threadIdx.x] = (q_i[i]*c_f - w_i); /* Left  going flux */
        fluxLR[1][threadIdx.x] = (q_i[i]*c_f + w_i); /* Right going flux */
        __syncthreads();

        /* Step 3 - Differentiate fluxes & call limiter */
            /* left flux */
        derivLR[0][threadIdx.x] = fluxLR[0][(threadIdx.x-1)%BLOCKLENP4] - fluxLR[0][threadIdx.x]; /* left derivative */
        derivLR[1][threadIdx.x] = fluxLR[0][threadIdx.x] - fluxLR[0][(threadIdx.x+1)%BLOCKLENP4]; /* right derivative */
        __syncthreads();
        fluxLR[0][threadIdx.x] += fluxLimiter_Vanleer(derivLR[0][threadIdx.x], derivLR[1][threadIdx.x]);
//        cukern_FluxLimiter_VanLeer(derivLR, fluxLR, 0);
        __syncthreads();

            /* Right flux */
        derivLR[0][threadIdx.x] = fluxLR[1][threadIdx.x] - fluxLR[1][(threadIdx.x-1)%BLOCKLENP4]; /* left derivative */
        derivLR[1][threadIdx.x] = fluxLR[1][(threadIdx.x+1)%BLOCKLENP4] - fluxLR[1][threadIdx.x]; /* right derivative */
        __syncthreads();
        fluxLR[1][threadIdx.x] += fluxLimiter_Vanleer(derivLR[0][threadIdx.x], derivLR[1][threadIdx.x]);
  //      cukern_FluxLimiter_VanLeer(derivLR, fluxLR, 1); 
        __syncthreads();

        /* Step 4 - Perform flux and write to output array */
       if( doIflux && (Xindex < nx) ) {
            outputPointers[i][x] -= halfLambda * ( fluxLR[0][threadIdx.x] - fluxLR[0][threadIdx.x+1] + \
                                                   fluxLR[1][threadIdx.x] - fluxLR[1][threadIdx.x-1]  ); 
          }

        __syncthreads();
        }

    Xindex += BLOCKLEN;
    Xtrack += BLOCKLEN;
    }

}

__global__ void cukern_TVDStep_hydro_uniform(double *P, double *Cfreeze, double halfLambda, int nx)
{
double C_f, velocity;
double q_i[5];
double w_i;
__shared__ double fluxLR[2][BLOCKLENP4];
__shared__ double derivLR[2][BLOCKLENP4];

/* Step 0 - obligatory annoying setup stuff (ASS) */
int I0 = nx*(blockIdx.x + gridDim.x * blockIdx.y);
int Xindex = (threadIdx.x-2);
int Xtrack = Xindex;
Xindex += nx*(threadIdx.x < 2);

int x; /* = Xindex % nx; */
int i;
bool doIflux = (threadIdx.x > 1) && (threadIdx.x < BLOCKLEN+2);

/* Step 1 - calculate W values */
C_f = Cfreeze[blockIdx.x + gridDim.x * blockIdx.y];

while(Xtrack < nx+2) {
    x = I0 + (Xindex % nx);

    q_i[0] = inputPointers[0][x]; /* Preload these out here */
    q_i[1] = inputPointers[1][x]; /* So we avoid multiple loops */
    q_i[2] = inputPointers[2][x]; /* over them inside the flux loop */
    q_i[3] = inputPointers[3][x];
    q_i[4] = inputPointers[4][x];
    velocity = q_i[2] / q_i[0];

    /* rho, E, px, py, pz going down */
    /* Iterate over variables to flux */
    for(i = 0; i < 5; i++) {
        /* Step 1 - Calculate raw fluxes */
        switch(i) {
            case 0: w_i = q_i[2]; break;
            case 1: w_i = (velocity * (q_i[1] + P[x]) ) ; break;
            case 2: w_i = (velocity * q_i[2] + P[x]); break;
            case 3: w_i = (velocity * q_i[3]); break;
            case 4: w_i = (velocity * q_i[4]); break;
            }

        /* Step 2 - Decouple to L/R flux */
/* NOTE there is a missing .5 here, accounted for in the h(al)f of lambdahf */
        fluxLR[0][threadIdx.x] = (C_f*q_i[i] - w_i); /* Left  going flux */
        fluxLR[1][threadIdx.x] = (C_f*q_i[i] + w_i); /* Right going flux */
        __syncthreads();

        /* Step 3 - Differentiate fluxes & call limiter */
            /* left flux */
        derivLR[0][threadIdx.x] = fluxLR[0][(threadIdx.x-1)%BLOCKLENP4] - fluxLR[0][threadIdx.x]; /* left derivative */
        derivLR[1][threadIdx.x] = fluxLR[0][threadIdx.x] - fluxLR[0][(threadIdx.x+1)%BLOCKLENP4]; /* right derivative */
        __syncthreads();
        fluxLR[0][threadIdx.x] += fluxLimiter_Vanleer(derivLR[0][threadIdx.x], derivLR[1][threadIdx.x]);
/*        cukern_FluxLimiter_VanLeer(derivLR, fluxLR, 0);*/
        __syncthreads();

            /* Right flux */
        derivLR[0][threadIdx.x] = fluxLR[1][threadIdx.x] - fluxLR[1][(threadIdx.x-1)%BLOCKLENP4]; /* left derivative */
        derivLR[1][threadIdx.x] = fluxLR[1][(threadIdx.x+1)%BLOCKLENP4] - fluxLR[1][threadIdx.x]; /* right derivative */
        __syncthreads();
        fluxLR[1][threadIdx.x] += fluxLimiter_Vanleer(derivLR[0][threadIdx.x], derivLR[1][threadIdx.x]);
/*        cukern_FluxLimiter_VanLeer(derivLR, fluxLR, 1);*/
        __syncthreads();

        /* Step 4 - Perform flux and write to output array */
       if( doIflux && (Xindex < nx) ) {
            outputPointers[i][x] -= halfLambda * ( fluxLR[0][threadIdx.x] - fluxLR[0][threadIdx.x+1] + \
                                                   fluxLR[1][threadIdx.x] - fluxLR[1][threadIdx.x-1]  );
            }

        __syncthreads();
        }

    Xindex += BLOCKLEN;
    Xtrack += BLOCKLEN;
    }

}


__device__ void cukern_FluxLimiter_VanLeer(double deriv[2][BLOCKLENP4], double flux[2][BLOCKLENP4], int who)
{

double r;

r = deriv[0][threadIdx.x] * deriv[1][threadIdx.x];
if(r < 0.0) r = 0.0;

r = r / ( deriv[0][threadIdx.x] + deriv[1][threadIdx.x]);
if (isnan(r)) { r = 0.0; }

flux[who][threadIdx.x] += r;

}

__device__ double fluxLimiter_Vanleer(double derivL, double derivR)
{
double r;

r = derivL * derivR;
if(r < 0.0) r = 0.0;

r = r / ( derivL + derivR);
if (isnan(r)) { r = 0.0; }

return r;
}


