#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA^M
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cudaCommon.h"

/* THIS FUNCTION 

This is the Cuda Fluid TVD function; It takes a single forward-time step, CFD or MHD, of the
conserved-transport part of the fluid equations using a total variation diminishing scheme to
perform a non-oscillatory update.

Requires predicted half-step values from a 1st order upwind scheme.

*/

#define BLOCKLEN 92
#define BLOCKLENP2 (BLOCKLEN+2)
#define BLOCKLENP4 (BLOCKLEN+4)

/*__global__ void cukern_TVDStep_mhd_uniform(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *P, double *Cfreeze, double *rhoW, double *enerW, double *pxW, double *pyW, double *pzW, double lambda, int nx);*/
__global__ void cukern_TVDStep_mhd_uniform(double *P, double *Cfreeze, double halflambda, int nx);
/*__global__ void cukern_TVDStep_hydro_uniform(double *rho, double *E, double *px, double *py, double *pz, double *P, double *Cfreeze, double *rhoW, double *enerW, double *pxW, double *pyW, double *pzW, double lambdahf, int nx);*/
__global__ void cukern_TVDStep_hydro_uniform(double *P, double *Cfreeze, double halfLambda, int nx);

__device__ void cukern_FluxLimiter_VanLeer(double deriv[2][BLOCKLENP4], double flux[2][BLOCKLENP4], int who);
__device__ __inline__ double fluxLimiter_Vanleer(double derivL, double derivR);

__constant__ __device__ double *inputPointers[8];
__constant__ __device__ double *outputPointers[5];
__constant__ __device__ double fluidParams[2];

#define RHOMIN fluidParams[0]
#define MIN_ETHERM fluidParams[1]

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  // At least 2 arguments expected
  // Input and result
  if ((nrhs!=18) || (nlhs != 0)) mexErrMsgTxt("Wrong number of arguments: call cudaTVDStep(rho, E, px, py, pz, bx, by, bz, P, rho_out, E_out, px_out, py_out, pz_out, C_freeze, lambda, purehydro?)\n");

  cudaCheckError("entering FluidTVD");

  // Get source array info and create destination arrays
  ArrayMetadata amd;

  // Get the source gpu arrays as enumerated in the error message
  double **srcs   = getGPUSourcePointers(prhs, &amd, 0, 13);

  // Get the GPU freeze speed. Differs in that it is not the same size
  ArrayMetadata fmd;
  double **gpu_cf = getGPUSourcePointers(prhs, &fmd, 14, 14); 

  // Get flux factor (dt / dx) amd determine if we are doing the hydro case or the MHD case
  double lambda   = *mxGetPr(prhs[15]);
  int isPureHydro = (int)*mxGetPr(prhs[16]);

  // Do the usual rigamarole determining array sizes and GPU launch dimensions
  dim3 arraySize;
  arraySize.x = amd.dim[0];
  arraySize.y = amd.dim[1];
  arraySize.z = amd.dim[2];

  dim3 blocksize, gridsize;

  blocksize.x = BLOCKLEN+4;
  blocksize.y = blocksize.z = 1;

  gridsize.x = arraySize.y;
  gridsize.y = arraySize.z;

  double *mins = mxGetPr(prhs[17]);
  double rhomin = mins[0];
  double gamma = mins[1];
  double gamHost[2];

  gamHost[0] = rhomin;
// assert     cs > cs_min
//     g P / rho > g rho_min^(g-1)
// (g-1) e / rho > rho_min^(g-1)
//             e > rho rho_min^(g-1)/(g-1)
  gamHost[1] = powl(rhomin, gamma-1.0)/(gamma-1.0);
  hipMemcpyToSymbol(HIP_SYMBOL(fluidParams), &gamHost[0], 2*sizeof(double), 0, hipMemcpyHostToDevice);

  // Invoke the kernel
  if(arraySize.x > 1) {
    if(isPureHydro) {
    //cukern_TVDStep_hydro_uniform                         (*rho,    *E,     *px,      *py,     *pz,     *P,      *Cfreeze, *rhoW,  *enerW,     *pxW,     *pyW,     *pzW,     lambda, nx);
      hipMemcpyToSymbol(HIP_SYMBOL(inputPointers),  srcs,     5*sizeof(double *), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(outputPointers), &srcs[9], 5*sizeof(double *), 0, hipMemcpyHostToDevice);
      cukern_TVDStep_hydro_uniform<<<gridsize, blocksize>>>(srcs[8], *gpu_cf, .5*lambda, arraySize.x);
    } else {
    //cukern_TVDStep_mhd_uniform                         (*rho,    *E,      *px,     *py,     *pz,     *bx,     *by,     *bz,     *P,           *Cfreeze, *rhoW,  *enerW,   *pxW,     *pyW,     *pzW, double lambda, int nx);
      hipMemcpyToSymbol(HIP_SYMBOL(inputPointers),  srcs,     8*sizeof(double *), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(outputPointers), &srcs[9], 5*sizeof(double *), 0, hipMemcpyHostToDevice);      
      cukern_TVDStep_mhd_uniform  <<<gridsize, blocksize>>>(srcs[8], *gpu_cf, .5*lambda, arraySize.x);
    }
  }

hipError_t epicFail = hipGetLastError();
if(epicFail != hipSuccess) cudaLaunchError(epicFail, blocksize, gridsize, &amd, isPureHydro, "fluid TVD");

}

/* blockidx.{xy} is our index in {yz}, and gridDim.{xy} gives the {yz} size */
/* Expect invocation with n+4 threads */
__global__ void cukern_TVDStep_mhd_uniform(double *P, double *Cfreeze, double halfLambda, int nx)
{
double c_f, velocity;
double q_i[5];
double b_i[3];
double w_i;
__shared__ double fluxLR[2][BLOCKLENP4];
__shared__ double fluxDerivA[BLOCKLENP4+1];
__shared__ double fluxDerivB[BLOCKLENP4+1];

/* Step 0 - obligatory annoying setup stuff (ASS) */
int I0 = nx*(blockIdx.x + gridDim.x * blockIdx.y);
int Xindex = (threadIdx.x-2);
int Xtrack = Xindex;
Xindex += nx*(threadIdx.x < 2);

int x; /* = Xindex % nx; */
int i;
bool doIflux = (threadIdx.x > 1) && (threadIdx.x < BLOCKLEN+2);

unsigned int threadIndexL = (threadIdx.x-1)%BLOCKLENP4;

/* Step 1 - calculate W values */
c_f = Cfreeze[blockIdx.x + gridDim.x * blockIdx.y];

double prop_i[5];

while(Xtrack < nx+2) {
    x = I0 + (Xindex % nx);

    q_i[0] = inputPointers[0][x];
    q_i[1] = inputPointers[1][x];       /* So we avoid multiple loops */
    q_i[2] = inputPointers[2][x];      /* over them inside the flux loop */
    q_i[3] = inputPointers[3][x];
    q_i[4] = inputPointers[4][x];
    b_i[0] = inputPointers[5][x];
    b_i[1] = inputPointers[6][x];
    b_i[2] = inputPointers[7][x];
    velocity = q_i[2]/q_i[0];

    /* rho, E, px, py, pz going down */
    /* Iterate over variables to flux */
    for(i = 0; i < 5; i++) {
        /* Step 1 - Calculate raw fluxes */
        switch(i) {
            case 0: w_i = q_i[2]; break;
            case 1: w_i = (velocity * (q_i[1] + P[x]) - b_i[0]*(q_i[2]*b_i[0]+q_i[3]*b_i[1]+q_i[4]*b_i[2])/q_i[0] ); break;
            case 2: w_i = (velocity*q_i[2] + P[x] - b_i[0]*b_i[0]); break;
            case 3: w_i = (velocity*q_i[3]        - b_i[0]*b_i[1]); break;
            case 4: w_i = (velocity*q_i[4]        - b_i[0]*b_i[2]); break;
            }

        /* Step 2 - Decouple to L/R flux */
        fluxLR[0][threadIdx.x] = (q_i[i]*c_f - w_i); /* Left  going flux */
        fluxLR[1][threadIdx.x] = (q_i[i]*c_f + w_i); /* Right going flux */
        __syncthreads();

        /* Step 3 - Differentiate fluxes & call limiter */
            /* left flux */
        fluxDerivA[threadIdx.x] = fluxLR[0][threadIndexL] - fluxLR[0][threadIdx.x]; 
        fluxDerivB[threadIdx.x] = fluxLR[1][threadIdx.x] - fluxLR[1][threadIndexL];
        __syncthreads();

            /* right flux */
        fluxLR[0][threadIdx.x] += fluxLimiter_Vanleer(fluxDerivA[threadIdx.x], fluxDerivA[threadIdx.x+1]);
        fluxLR[1][threadIdx.x] += fluxLimiter_Vanleer(fluxDerivB[threadIdx.x+1], fluxDerivB[threadIdx.x]);
        __syncthreads();

        /* Step 4 - Perform flux and write to output array */
       if( doIflux && (Xindex < nx) ) {
            prop_i[i] = outputPointers[i][x] - halfLambda * ( fluxLR[0][threadIdx.x] - fluxLR[0][threadIdx.x+1] + \
                                                   fluxLR[1][threadIdx.x] - fluxLR[1][threadIndexL]  ); 
          }

        __syncthreads();
        }

    if( doIflux && (Xindex < nx) ) {
      prop_i[0] = (prop_i[0] < RHOMIN) ? RHOMIN : prop_i[0]; // enforce min density

      w_i = .5*(prop_i[2]*prop_i[2] + prop_i[3]*prop_i[3] + prop_i[4]*prop_i[4])/prop_i[0] + .5*(b_i[0]*b_i[0] + b_i[1]*b_i[1] + b_i[2]*b_i[2]);

      if((prop_i[1] - w_i) < prop_i[0]*MIN_ETHERM) {
        prop_i[1] = prop_i[0]*MIN_ETHERM + w_i;
        }

      outputPointers[0][x] = prop_i[0];
      outputPointers[1][x] = prop_i[1];
      outputPointers[2][x] = prop_i[2];
      outputPointers[3][x] = prop_i[3];
      outputPointers[4][x] = prop_i[4];
      }

    Xindex += BLOCKLEN;
    Xtrack += BLOCKLEN;
    }

}

__global__ void cukern_TVDStep_hydro_uniform(double *P, double *Cfreeze, double halfLambda, int nx)
{
double C_f, velocity;
double q_i[5];
double w_i;
__shared__ double fluxLR[2][BLOCKLENP4];
__shared__ double fluxDerivA[BLOCKLENP4+1];
__shared__ double fluxDerivB[BLOCKLENP4+1];

/* Step 0 - obligatory annoying setup stuff (ASS) */
int I0 = nx*(blockIdx.x + gridDim.x * blockIdx.y);
int Xindex = (threadIdx.x-2);
int Xtrack = Xindex;
Xindex += nx*(threadIdx.x < 2);

int x; /* = Xindex % nx; */
int i;
bool doIflux = (threadIdx.x > 1) && (threadIdx.x < BLOCKLEN+2);
double prop_i[5];

unsigned int threadIndexL = (threadIdx.x-1)%BLOCKLENP4;

/* Step 1 - calculate W values */
C_f = Cfreeze[blockIdx.x + gridDim.x * blockIdx.y];

while(Xtrack < nx+2) {
    x = I0 + (Xindex % nx);

    q_i[0] = inputPointers[0][x]; /* Preload these out here */
    q_i[1] = inputPointers[1][x]; /* So we avoid multiple loops */
    q_i[2] = inputPointers[2][x]; /* over them inside the flux loop */
    q_i[3] = inputPointers[3][x];
    q_i[4] = inputPointers[4][x];
    velocity = q_i[2] / q_i[0];

    /* rho, E, px, py, pz going down */
    /* Iterate over variables to flux */
    for(i = 0; i < 5; i++) {
        /* Step 1 - Calculate raw fluxes */
        switch(i) {
            case 0: w_i = q_i[2]; break;
            case 1: w_i = (velocity * (q_i[1] + P[x]) ) ; break;
            case 2: w_i = (velocity * q_i[2] + P[x]); break;
            case 3: w_i = (velocity * q_i[3]); break;
            case 4: w_i = (velocity * q_i[4]); break;
            }

        /* Step 2 - Decouple to L/R flux */
/* NOTE there is a missing .5 here, accounted for in the h(al)f of lambdahf */
        fluxLR[0][threadIdx.x] = (C_f*q_i[i] - w_i); /* Left  going flux */
        fluxLR[1][threadIdx.x] = (C_f*q_i[i] + w_i); /* Right going flux */
        __syncthreads();

        /* Step 3 - Differentiate fluxes & call limiter */
            /* left flux */
        fluxDerivA[threadIdx.x] = fluxLR[0][threadIndexL] - fluxLR[0][threadIdx.x];
        fluxDerivB[threadIdx.x] = fluxLR[1][threadIdx.x] - fluxLR[1][threadIndexL];
        __syncthreads();
        
            /* right flux */
        fluxLR[0][threadIdx.x] += fluxLimiter_Vanleer(fluxDerivA[threadIdx.x], fluxDerivA[threadIdx.x+1]);
        fluxLR[1][threadIdx.x] += fluxLimiter_Vanleer(fluxDerivB[threadIdx.x+1], fluxDerivB[threadIdx.x]);
        __syncthreads();

        /* Step 4 - Perform flux and write to output array */
       if( doIflux && (Xindex < nx) ) {
            prop_i[i] = outputPointers[i][x] - halfLambda * ( fluxLR[0][threadIdx.x] - fluxLR[0][threadIdx.x+1] + \
                                                   fluxLR[1][threadIdx.x] - fluxLR[1][threadIndexL]  );
            }

        __syncthreads();
        }

    if( doIflux && (Xindex < nx) ) {
        prop_i[0] = (prop_i[0] < RHOMIN) ? RHOMIN : prop_i[0];
        w_i = .5*(prop_i[2]*prop_i[2] + prop_i[3]*prop_i[3] + prop_i[4]*prop_i[4])/prop_i[0];

        if((prop_i[1] - w_i) < prop_i[0]*MIN_ETHERM) {
            prop_i[1] = w_i + prop_i[0]*MIN_ETHERM;
            }

        outputPointers[0][x] = prop_i[0];
        outputPointers[1][x] = prop_i[1];
        outputPointers[2][x] = prop_i[2];
        outputPointers[3][x] = prop_i[3];
        outputPointers[4][x] = prop_i[4];
        }

    Xindex += BLOCKLEN;
    Xtrack += BLOCKLEN;
    }

}


__device__ double fluxLimiter_Vanleer(double derivL, double derivR)
{
double r;

r = derivL * derivR;
if(r < 0.0) { r = 0.0; }

r = r / ( derivL + derivR);
if (isnan(r)) { r = 0.0; }

return r;
}


