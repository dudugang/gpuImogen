#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"

#include "cudaCommon.h"
#include "cudaSourceScalarPotential.h"

#define GRADBLOCKX 18
#define GRADBLOCKY 18

#define SRCBLOCKX 16
#define SRCBLOCKY 16

int sourcefunction_Composite(MGArray *fluid, MGArray *phi, MGArray *XYVectors, GeometryParams geom, double rhoNoG, double rhoFullGravity, double omega, double dt, int spaceOrder, int temporalOrder);


__global__ void writeScalarToVector(double *x, long numel, double f);

// compute grad(phi) in XYZ or R-Theta-Z 
template <geometryType_t coords>
__global__ void  cukern_computeScalarGradient3D_h2(double *rho, double *phi, double *f_x, double *f_y, double *f_z, int3 arraysize);
template <geometryType_t coords>
__global__ void  cukern_computeScalarGradient3D_h4_partone(double *rho, double *phi, double *fx, double *fy, int3 arraysize);
__global__ void  cukern_computeScalarGradient3D_h4_parttwo(double *rho, double *phi, double *fz, int3 arraysize);

// compute grad(phi) in X-Y or R-Theta
template <geometryType_t coords>
__global__ void  cukern_computeScalarGradient2D_h2(double *rho, double *phi, double *fx, double *fy, int3 arraysize);
template <geometryType_t coords>
__global__ void  cukern_computeScalarGradient2D_h4(double *rho, double *phi, double *fx, double *fy, int3 arraysize);

// Compute grad(phi) in X-Z or R-Z
__global__ void  cukern_computeScalarGradientRZ_h2(double *rho, double *phi, double *fx, double *fz, int3 arraysize);
__global__ void  cukern_computeScalarGradientRZ_h4(double *rho, double *phi, double *fx, double *fz, int3 arraysize);

__global__ void cukern_FetchPartitionSubset1D(double *in, int nodeN, double *out, int partX0, int partNX);

template <geometryType_t coords>
__global__ void  cukern_sourceComposite_IMP(double *fluidIn, double *Rvector, double *gravgrad, long pitch);

template <geometryType_t coords>
__global__ void  cukern_sourceComposite_RK4(double *fluidIn, double *Rvector, double *gravgrad, long pitch);

// This will probably be slow as balls but should provide a golden standard of accuracy
template <geometryType_t coords>
__global__ void  cukern_sourceComposite_GL4(double *fluidIn, double *Rvector, double *gravgrad, long pitch);

template <geometryType_t coords>
__global__ void  cukern_sourceComposite_GL6(double *fluidIn, double *Rvector, double *gravgrad, long pitch);

__constant__ __device__ double devLambda[12];

#define LAMX devLambda[0]
#define LAMY devLambda[1]
#define LAMZ devLambda[2]

// Define: F = -beta * rho * grad(phi)
// rho_g = density for full effect of gravity 
// rho_c = minimum density to feel gravity at all
// beta = { rho_g < rho         : 1 (NORMAL GRAVITY)                }
//        { rho_c < rho < rho_g : [(rho-rho_c)/(rho_g-rho_c)]^2 }
//        {         rho < rho_c : 0                                 }

// This provides a continuous (though not differentiable at rho = rho_g) way to surpress gravitation of the background fluid
// The original process of cutting gravity off below a critical density a few times the minimum
// density is believed to cause "blowups" at the inner edge of circular flow profiles due to being
// discontinuous. If even smoothness is insufficient and smooth differentiability is required,
// a more-times-continuous profile can be constructed, but let's not go there unless forced.

// Density below which we force gravity effects to zero
#define RHO_FULLG devLambda[3]
#define RHO_NOG devLambda[4]

// 1 / (rho_fullg - rho_nog)
#define G1 devLambda[5]

// rho_nog / (rho_fullg - rho_nog)
#define G2 devLambda[6]
#define RINNER devLambda[7]
#define DELTAR devLambda[8]

// __constant__ parameters for the rotating frame terms
#define OMEGA devLambda[9]
#define DT devLambda[10]
#define TWO_OMEGA_T devLambda[11]

__constant__ __device__ int devIntParams[3];



void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	// At least 2 arguments expected
	// Input and result
	if ((nrhs!=5) || (nlhs != 0)) mexErrMsgTxt("Wrong number of arguments: need cudaTestSourceComposite(FluidManager, phi, GeometryManager, [rhomin, rho_fullg, omega, dt, spaceorder], [xvector yvector])\n");

	CHECK_CUDA_ERROR("entering cudaSourceRotatingFrame");

	// Get source array info and create destination arrays
	MGArray fluid[5], gravPot, xyvec;

	/* FIXME: accept this as a matlab array instead
	 * FIXME: Transfer appropriate segments to __constant__ memory
	 * FIXME: that seems the only reasonable way to avoid partitioning hell
	 */
	double *scalars = mxGetPr(prhs[3]);
	if(mxGetNumberOfElements(prhs[3]) != 6) {
		PRINT_FAULT_HEADER;
		printf("The 4th argument must be a five element vector: [rho_nog, rho_fullg, omega, dt, space order, temporal order]. It contains %i elements.\n", mxGetNumberOfElements(prhs[3]));
		PRINT_FAULT_FOOTER;
		DROP_MEX_ERROR("Invalid arguments, brah!");
	}
	double omega = scalars[2];
	double dt    = scalars[3];
	double rhonog= scalars[0];
	double rhofg = scalars[1];
	int spaceOrder    = (int)scalars[4];
	int timeOrder     = (int)scalars[5];
	GeometryParams geom = accessMatlabGeometryClass(prhs[2]);

	int status;

	status = MGA_accessMatlabArrays(prhs, 4, 4, &xyvec);
	if(CHECK_IMOGEN_ERROR(status) != SUCCESSFUL) { DROP_MEX_ERROR("Failed to access X-Y vector."); }

	status = MGA_accessMatlabArrays(prhs, 1, 1, &gravPot);
	if(CHECK_IMOGEN_ERROR(status) != SUCCESSFUL) { DROP_MEX_ERROR("Failed to access gravity potential array."); }

	dim3 gridsize, blocksize;

	int numFluids = mxGetNumberOfElements(prhs[0]);
	int fluidct;

	for(fluidct = 0; fluidct < numFluids; fluidct++) {
		status = MGA_accessFluidCanister(prhs[0], fluidct, &fluid[0]);
		if(CHECK_IMOGEN_ERROR(status) != SUCCESSFUL) break;

		status = sourcefunction_Composite(&fluid[0], &gravPot, &xyvec, geom, rhonog, rhofg, omega, dt, spaceOrder, timeOrder);
		if(CHECK_IMOGEN_ERROR(status) != SUCCESSFUL) { DROP_MEX_ERROR("Failed to apply rotating frame source terms."); }
	}

}


int sourcefunction_Composite(MGArray *fluid, MGArray *phi, MGArray *XYVectors, GeometryParams geom, double rhoNoG, double rhoFullGravity, double omega, double dt, int spaceOrder, int timeOrder)
{
	dim3 gridsize, blocksize;
	int3 arraysize;

	double lambda[11];

	int i;
	int worked;

	double *devXYset[fluid->nGPUs];
	int sub[6];

	double *dx = &geom.h[0];
	if(spaceOrder == 4) {
		lambda[0] = dt/(12.0*dx[0]);
		lambda[1] = dt/(12.0*dx[1]);
		lambda[2] = dt/(12.0*dx[2]);
	} else {
		lambda[0] = dt/(2.0*dx[0]);
		lambda[1] = dt/(2.0*dx[1]);
		lambda[2] = dt/(2.0*dx[2]);
	}

	lambda[3] = rhoFullGravity;
	lambda[4] = rhoNoG;

    lambda[5] = 1.0/(rhoFullGravity - rhoNoG);
    lambda[6] = rhoNoG/(rhoFullGravity - rhoNoG);

    lambda[7] = geom.Rinner; // This is actually overwritten per partition below
    lambda[8] = dx[1];

	lambda[9] = omega;
	lambda[10]= dt;

	int isThreeD = (fluid->dim[2] > 1);
	int isRZ = (fluid->dim[2] > 1) & (fluid->dim[1] == 1);

	double *gradMem[fluid->nGPUs];

    for(i = 0; i < fluid->nGPUs; i++) {
    	hipSetDevice(fluid->deviceID[i]);
    	calcPartitionExtent(fluid, i, &sub[0]);

    	lambda[7] = geom.Rinner + dx[0] * sub[0]; // Innermost cell coord may change per-partition

    	hipMemcpyToSymbol((const void *)devLambda, lambda, 11*sizeof(double), 0, hipMemcpyHostToDevice);
    	worked = CHECK_CUDA_ERROR("hipMemcpyToSymbol");
    	if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) break;

    	hipMemcpyToSymbol((const void *)devIntParams, &sub[3], 3*sizeof(int), 0, hipMemcpyHostToDevice);
    	worked = CHECK_CUDA_ERROR("memcpy to symbol");
    	if(worked != SUCCESSFUL) break;

    	hipMalloc((void **)&gradMem[i], 3*sub[3]*sub[4]*sub[5]*sizeof(double));
    }

    if(worked != SUCCESSFUL) return worked;

    double *fpi, *ppi;

    // Iterate over all partitions, and here we GO!
    for(i = 0; i < fluid->nGPUs; i++) {
		hipSetDevice(fluid->deviceID[i]);
		worked = CHECK_CUDA_ERROR("hipSetDevice");
		if(worked != SUCCESSFUL) break;

        calcPartitionExtent(fluid, i, sub);

        arraysize.x = sub[3]; arraysize.y = sub[4]; arraysize.z = sub[5];

        blocksize = makeDim3(GRADBLOCKX, GRADBLOCKY, 1);
        gridsize.x = arraysize.x / (blocksize.x - spaceOrder); gridsize.x += ((blocksize.x-spaceOrder) * gridsize.x < arraysize.x);
        if(isRZ) {
        	gridsize.y = arraysize.z / (blocksize.y - spaceOrder); gridsize.y += ((blocksize.y-spaceOrder) * gridsize.y < arraysize.z);
        } else {
        	gridsize.y = arraysize.y / (blocksize.y - spaceOrder); gridsize.y += ((blocksize.y-spaceOrder) * gridsize.y < arraysize.y);
        }
        gridsize.z = 1;

        fpi = fluid->devicePtr[i]; // save some readability below...
        ppi = phi->devicePtr[i];

        switch(spaceOrder) {
        case 2:
        	if(isThreeD) {
        		if(isRZ) {
        			cukern_computeScalarGradientRZ_h2<<<gridsize, blocksize>>>(fpi, ppi, gradMem[i], gradMem[i] + 2*fluid->partNumel[i],  arraysize);
        			writeScalarToVector<<<32, 256>>>(gradMem[i]+fluid->partNumel[i], fluid->partNumel[i], 0.0);
        		} else {
        			if(geom.shape == SQUARE) {
        				cukern_computeScalarGradient3D_h2<SQUARE><<<gridsize, blocksize>>>(fpi, ppi, gradMem[i], gradMem[i]+fluid->partNumel[i], gradMem[i]+fluid->partNumel[i]*2, arraysize); }
        			if(geom.shape == CYLINDRICAL) {
        				cukern_computeScalarGradient3D_h2<CYLINDRICAL><<<gridsize, blocksize>>>(fpi, ppi, gradMem[i], gradMem[i]+fluid->partNumel[i], gradMem[i]+fluid->partNumel[i]*2, arraysize); }
        		}
        	} else {
        		if(geom.shape == SQUARE) {
        			cukern_computeScalarGradient2D_h2<SQUARE><<<gridsize, blocksize>>>(fpi, ppi, gradMem[i], gradMem[i]+fluid->partNumel[i], arraysize); }
        		if(geom.shape == CYLINDRICAL) {
        			cukern_computeScalarGradient2D_h2<CYLINDRICAL><<<gridsize, blocksize>>>(fpi, ppi, gradMem[i], gradMem[i]+fluid->partNumel[i], arraysize); }

        		writeScalarToVector<<<32, 256>>>(gradMem[i]+2*fluid->partNumel[i], fluid->partNumel[i], 0.0);
        	}
        	break;
        case 4:
        	if(isThreeD) {
        		if(isRZ) {
        			cukern_computeScalarGradientRZ_h4<<<gridsize, blocksize>>>(fpi, ppi, gradMem[i], gradMem[i] + 2*fluid->partNumel[i],  arraysize);
        			writeScalarToVector<<<32, 256>>>(gradMem[i]+fluid->partNumel[i], fluid->partNumel[i], 0.0);
        		} else {
        			if(geom.shape == SQUARE) {
        				cukern_computeScalarGradient3D_h4_partone<SQUARE><<<gridsize, blocksize>>>(fpi, ppi, gradMem[i], gradMem[i]+fluid->partNumel[i], arraysize);
        				cukern_computeScalarGradient3D_h4_parttwo<<<gridsize, blocksize>>>(fpi, ppi, gradMem[i]+fluid->partNumel[i]*2, arraysize);
        			}
        			if(geom.shape == CYLINDRICAL) {
        				cukern_computeScalarGradient3D_h4_partone<CYLINDRICAL><<<gridsize, blocksize>>>(fpi, ppi, gradMem[i], gradMem[i]+fluid->partNumel[i], arraysize);
        				cukern_computeScalarGradient3D_h4_parttwo<<<gridsize, blocksize>>>(fpi, ppi, gradMem[i]+fluid->partNumel[i]*2, arraysize);
        			}
        		}
        	} else {
        		if(geom.shape == SQUARE) {
        			cukern_computeScalarGradient2D_h4<SQUARE><<<gridsize, blocksize>>>(fpi, ppi, gradMem[i], gradMem[i]+fluid->partNumel[i], arraysize); }
        		if(geom.shape == CYLINDRICAL) {
        			cukern_computeScalarGradient2D_h4<CYLINDRICAL><<<gridsize, blocksize>>>(fpi, ppi, gradMem[i], gradMem[i]+fluid->partNumel[i], arraysize); }

        		writeScalarToVector<<<32, 256>>>(gradMem[i]+2*fluid->partNumel[i], fluid->partNumel[i], 0.0);

        	}

        	break;
        default:
        	PRINT_FAULT_HEADER;
        	printf("Was passed spatial order parameter of %i, must be passed 2 (2nd order) or 4 (4th order)\n", spaceOrder);
        	PRINT_FAULT_FOOTER;
        	hipFree(gradMem[i]);
        	return ERROR_INVALID_ARGS;
        }

        worked = CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, fluid, i, "cukern_computeScalarGradient");

        // This section extracts the portions of the supplied partition-cloned [X;Y] vector relevant to the current partition
        hipMalloc((void **)&devXYset[i], (sub[3]+sub[4])*sizeof(double));
        worked = CHECK_CUDA_ERROR("hipMalloc");
        if(worked != SUCCESSFUL) break;

        blocksize = makeDim3(128, 1, 1);
        gridsize.x = ROUNDUPTO(sub[3], 128) / 128;
        gridsize.y = gridsize.z = 1;
        cukern_FetchPartitionSubset1D<<<gridsize, blocksize>>>(XYVectors->devicePtr[i], fluid->dim[0], devXYset[i], sub[0], sub[3]);
        worked = CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, XYVectors, i, "cukern_FetchPartitionSubset1D, X");
        if(worked != SUCCESSFUL) break;

        gridsize.x = ROUNDUPTO(sub[4], 128) / 128;
        cukern_FetchPartitionSubset1D<<<gridsize, blocksize>>>(XYVectors->devicePtr[i] + fluid->dim[0], fluid->dim[1], devXYset[i]+sub[3], sub[1], sub[4]);
        worked = CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, XYVectors, i, "cukern_FetchPartitionSubset1D, Y");
        if(worked != SUCCESSFUL) break;

        // Prepare to launch the solver itself!
        arraysize.x = sub[3]; arraysize.y = sub[4]; arraysize.z = sub[5];

        blocksize = makeDim3(SRCBLOCKX, SRCBLOCKY, 1);
        gridsize.x = ROUNDUPTO(arraysize.x, blocksize.x) / blocksize.x;
        gridsize.y = (isRZ) ? 1 : arraysize.z;
        gridsize.z = 1;

        switch(timeOrder) {
        case 2:
        	if(isRZ) {
        		if(geom.shape == SQUARE) {
        			cukern_sourceComposite_IMP<RZSQUARE><<<gridsize, blocksize>>>(fpi, devXYset[i], gradMem[i], fluid->slabPitch[i]/8);
        		} else {
        			cukern_sourceComposite_IMP<RZCYLINDRICAL><<<gridsize, blocksize>>>(fpi, devXYset[i], gradMem[i], fluid->slabPitch[i]/8);
        		}
        	} else {
        		if(geom.shape == SQUARE) {
        			cukern_sourceComposite_IMP<SQUARE><<<gridsize, blocksize>>>(fpi, devXYset[i], gradMem[i], fluid->slabPitch[i]/8);
        		} else {
        			cukern_sourceComposite_IMP<CYLINDRICAL><<<gridsize, blocksize>>>(fpi, devXYset[i], gradMem[i], fluid->slabPitch[i]/8);
        		}
        	}
        	break;
        case 4:
        	if(isRZ) {
        		if(geom.shape == SQUARE) {
        			cukern_sourceComposite_GL4<RZSQUARE><<<gridsize, blocksize>>>(fpi, devXYset[i], gradMem[i], fluid->slabPitch[i]/8);
        		} else {
        			cukern_sourceComposite_GL4<RZCYLINDRICAL><<<gridsize, blocksize>>>(fpi, devXYset[i], gradMem[i], fluid->slabPitch[i]/8);
        		}
        	} else {
        		if(geom.shape == SQUARE) {
        			cukern_sourceComposite_GL4<SQUARE><<<gridsize, blocksize>>>(fpi, devXYset[i], gradMem[i], fluid->slabPitch[i]/8);
        		} else {
        			cukern_sourceComposite_GL4<CYLINDRICAL><<<gridsize, blocksize>>>(fpi, devXYset[i], gradMem[i], fluid->slabPitch[i]/8);
        		}
        	}
        	break;
        case 6:
        	if(isRZ) {
        		if(geom.shape == SQUARE) {
        			cukern_sourceComposite_GL6<RZSQUARE><<<gridsize, blocksize>>>(fpi, devXYset[i], gradMem[i], fluid->slabPitch[i]/8);
        		} else {
        			cukern_sourceComposite_GL6<RZCYLINDRICAL><<<gridsize, blocksize>>>(fpi, devXYset[i], gradMem[i], fluid->slabPitch[i]/8);
        		}
        	} else {
        		if(geom.shape == SQUARE) {
        			cukern_sourceComposite_GL6<SQUARE><<<gridsize, blocksize>>>(fpi, devXYset[i], gradMem[i], fluid->slabPitch[i]/8);
        		} else {
        			cukern_sourceComposite_GL6<CYLINDRICAL><<<gridsize, blocksize>>>(fpi, devXYset[i], gradMem[i], fluid->slabPitch[i]/8);
        		}
        	}
        	break;
        default:
        	PRINT_FAULT_HEADER;
        	printf("Source function requires a temporal order of 2 (implicit midpt), 4 (Gauss-Legendre 4th order) or 6 (GL-6th): Received %i\n", timeOrder);
        	PRINT_FAULT_FOOTER;
        	break;
        }

        worked = CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, fluid, i, "cukernSourceComposite");
        if(worked != SUCCESSFUL) break;
    }

    worked = MGA_exchangeLocalHalos(fluid, 5);
    if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) return worked;

    int j; // This will halt at the stage failed upon if CUDA barfed above
    for(j = 0; j < i; j++) {
    	hipFree((void *)gradMem[j]);
    	hipFree((void *)devXYset[j]);
    }

    // Don't bother checking hipFree if we already have an error caused above, it was just trying to clean up the barf
    if(worked == SUCCESSFUL)
    	worked = CHECK_CUDA_ERROR("hipFree");

    return CHECK_IMOGEN_ERROR(worked);

}

/* Given a density, reads G1 and G2 (devLambda[5, 6]) and returns a factor to
 * rescale density by as follows:
 * RHOCRIT < rho          : 1
 * RHO_FULLG  < rho < RHOCRIT: rho*g1 - g2
 *           rho < RHO_FULLG : 0
 * This piecewise linear continuous function ramps gravity's
 * strength from 1 at/above RHO_FULLG down to 0 at/below RHO_NOG
 */
__device__ double cukern_computeMondFactor(double rho)
{
double x = 1;
return x;/*
if(rho < RHO_FULLG) {
	if(rho < RHO_NOG) {
		x = 0;
	} else {
		x = rho*G1 - G2;
	}
}

return x;
*/
}

/* Second order methods compute dU/dx using the 2-point central derivative,
 *     dU/dx = [ -f(x-h) + f(x+h) ] / 2h + O(h^2)
 * Fourth order methods compute dU/dx using the 4-point central derivative,
 *     dU/dx = [ f(x-2h) - 8 f(x-h) + 8 f(x+h) - f(x+2h) ] / 12h + O(h^4)
 * applied independently to the directions of interest.
 * Phi-direction derivatives in cylindrical geometry acquire an additional factor of 1/r
 * because lambda computes dU/dtheta in this case, not (grad U).(theta-hat).
 */

/* Computes the gradient of 3d array phi using the 2-point centered derivative,
 * and stores phi_x in fx, phi_y in fy, phi_z in fz.
 * All arrays (rho, phi, fx, fy, fz) must be of size arraysize.
 * In cylindrical geometry, f_x -> f_r,
 *                          f_y -> f_phi
 */
template <geometryType_t coords>
__global__ void  cukern_computeScalarGradient3D_h2(double *rho, double *phi, double *fx, double *fy, double *fz, int3 arraysize)
{
int myLocAddr = threadIdx.x + GRADBLOCKX*threadIdx.y;

int myX = threadIdx.x + (GRADBLOCKX-2)*blockIdx.x - 1;
int myY = threadIdx.y + (GRADBLOCKY-2)*blockIdx.y - 1;

if((myX > arraysize.x) || (myY > arraysize.y)) return;

bool IWrite = (threadIdx.x > 0) && (threadIdx.x < (GRADBLOCKX-1)) && (threadIdx.y > 0) && (threadIdx.y < (GRADBLOCKY-1));
IWrite = IWrite && (myX < arraysize.x) && (myY < arraysize.y);

myX = (myX + arraysize.x) % arraysize.x;
myY = (myY + arraysize.y) % arraysize.y;

int globAddr = myX + arraysize.x*myY;

double deltaphi; // Store derivative of phi in one direction

__shared__ double phiA[GRADBLOCKX*GRADBLOCKY];
__shared__ double phiB[GRADBLOCKX*GRADBLOCKY];
__shared__ double phiC[GRADBLOCKX*GRADBLOCKY];

double *U; double *V; double *W;
double *temp;
double mFactor;

U = phiA; V = phiB; W = phiC;

// Preload lower and middle planes
U[myLocAddr] = phi[globAddr + arraysize.x*arraysize.y*(arraysize.z-1)];
V[myLocAddr] = phi[globAddr];

__syncthreads();

int z;
int deltaz = arraysize.x*arraysize.y;
for(z = 0; z < arraysize.z; z++) {
  mFactor = cukern_computeMondFactor(rho[globAddr]);

  if(z >= arraysize.z - 1) deltaz = - arraysize.x*arraysize.y*(arraysize.z-1);

  if(IWrite) {
    deltaphi         = LAMX*(V[myLocAddr+1]-V[myLocAddr-1]);
    fx[globAddr]     = mFactor*deltaphi; // store px <- px - dt * rho dphi/dx;
  }

  if(IWrite) {
  if(coords == SQUARE) {
    deltaphi         = LAMY*(V[myLocAddr+GRADBLOCKX]-V[myLocAddr-GRADBLOCKX]);
    }
    if(coords == CYLINDRICAL) {
    // In cylindrical coords, use dt/dphi * (delta-phi) / r to get d/dy
    deltaphi         = LAMY*(V[myLocAddr+GRADBLOCKX]-V[myLocAddr-GRADBLOCKX]) / (RINNER + DELTAR*myX);
    }
    fy[globAddr]     = mFactor*deltaphi;
  }

  W[myLocAddr]       = phi[globAddr + deltaz]; // load phi(z+1) -> phiC
  __syncthreads();
  deltaphi           = LAMZ*(W[myLocAddr] - U[myLocAddr]);

  if(IWrite) {
    fz[globAddr]     = mFactor*deltaphi;
  }

  temp = U; U = V; V = W; W = temp; // cyclically shift them back
  globAddr += arraysize.x * arraysize.y;

}

}

/* Computes the gradient of 3d array phi using the 4-point centered derivative and
 * stores phi_x in fx, phi_y in fy, phi_z in fz.
 * All arrays (rho, phi, fx, fy, fz) must be of size arraysize.
 * In cylindrical geometry, f_x -> f_r,
 *                          f_y -> f_phi
 * This call must be invoked in two parts:
 * cukern_computeScalarGradient3D_h4_partone computes the X and Y (or r/theta) derivatives,
 * cukern_computeScalarGradient3D_h4_parttwo computes the Z derivative.
 */
template <geometryType_t coords>
__global__ void  cukern_computeScalarGradient3D_h4_partone(double *rho, double *phi, double *fx, double *fy, int3 arraysize)
{
	int myLocAddr = threadIdx.x + GRADBLOCKX*threadIdx.y;

	int myX = threadIdx.x + (GRADBLOCKX-4)*blockIdx.x - 2;
	int myY = threadIdx.y + (GRADBLOCKY-4)*blockIdx.y - 2;

	if((myX > (arraysize.x+1)) || (myY > (arraysize.y+1))) return;

	bool IWrite = (threadIdx.x > 1) && (threadIdx.x < (GRADBLOCKX-2)) && (threadIdx.y > 1) && (threadIdx.y < (GRADBLOCKY-2));
	IWrite = IWrite && (myX < arraysize.x) && (myY < arraysize.y);

	myX = (myX + arraysize.x) % arraysize.x;
	myY = (myY + arraysize.y) % arraysize.y;

	int globAddr = myX + arraysize.x*myY;

	double deltaphi; // Store derivative of phi in one direction

	__shared__ double phishm[GRADBLOCKX*GRADBLOCKY];
	double mFactor;

	__syncthreads();

	int z;
	int deltaz = arraysize.x*arraysize.y;
	for(z = 0; z < arraysize.z; z++) {
		mFactor = cukern_computeMondFactor(rho[globAddr]);

		phishm[myLocAddr] = phi[globAddr];

		__syncthreads();

		if(IWrite) {
			deltaphi         = LAMX*(-phishm[myLocAddr+2]+8.0*phishm[myLocAddr+1]-8.0*phishm[myLocAddr-1]+phishm[myLocAddr-2]);
			fx[globAddr]     = mFactor*deltaphi; // store px <- px - dt * rho dphi/dx;

			if(coords == SQUARE) {
				deltaphi         = LAMY*(-phishm[myLocAddr+2*GRADBLOCKX]+8*phishm[myLocAddr+GRADBLOCKX]-8*phishm[myLocAddr-GRADBLOCKX]+phishm[myLocAddr-2*GRADBLOCKX]);
			}
			if(coords == CYLINDRICAL) {
				// In cylindrical coords, use dt/dphi * (delta-phi) / r to get d/dy
				deltaphi         = LAMY*(-phishm[myLocAddr+2*GRADBLOCKX]+8*phishm[myLocAddr+GRADBLOCKX]-8*phishm[myLocAddr-GRADBLOCKX]+phishm[myLocAddr-2*GRADBLOCKX]) / (RINNER + DELTAR*myX);
			}
			fy[globAddr]     = mFactor*deltaphi;
		}

		globAddr += deltaz;
	}
}

__global__ void  cukern_computeScalarGradient3D_h4_parttwo(double *rho, double *phi, double *fz, int3 arraysize)
{
	int myLocAddr = threadIdx.x + GRADBLOCKX*threadIdx.y;

	int myX = threadIdx.x + (GRADBLOCKX-4)*blockIdx.x - 2;
	int myZ = threadIdx.y + (GRADBLOCKY-4)*blockIdx.y - 2;

	if((myX > (arraysize.x+1)) || (myZ > (arraysize.z+1))) return;

	bool IWrite = (threadIdx.x > 1) && (threadIdx.x < (GRADBLOCKX-2)) && (threadIdx.y > 1) && (threadIdx.y < (GRADBLOCKY-2));
	IWrite = IWrite && (myX < arraysize.x) && (myZ < arraysize.z);

	myX = (myX + arraysize.x) % arraysize.x;
	myZ = (myZ + arraysize.z) % arraysize.z;

	int delta = arraysize.x*arraysize.y;

	int globAddr = myX + delta*myZ;

	double deltaphi; // Store derivative of phi in one direction

	__shared__ double phishm[GRADBLOCKX*GRADBLOCKY];
	double mFactor;

	__syncthreads();

	int y;
	for(y = 0; y < arraysize.y; y++) {
		mFactor = cukern_computeMondFactor(rho[globAddr]);

		phishm[myLocAddr] = phi[globAddr];

		if(IWrite) {
			deltaphi         = LAMZ*(-phishm[myLocAddr+2*GRADBLOCKX]+8*phishm[myLocAddr+GRADBLOCKX]-8*phishm[myLocAddr-GRADBLOCKX]+phishm[myLocAddr-2*GRADBLOCKX]);
			fz[globAddr]     = mFactor*deltaphi;
		}
		globAddr += arraysize.x;
	}
}

/* Compute the gradient of 2d array phi with 2nd order accuracy; store the results in f_x, f_y
 *    In cylindrical geometry, f_x -> f_r,
 *                             f_y -> f_phi
 */
template <geometryType_t coords>
__global__ void  cukern_computeScalarGradient2D_h2(double *rho, double *phi, double *fx, double *fy, int3 arraysize)
{
	int myLocAddr = threadIdx.x + GRADBLOCKX*threadIdx.y;

	int myX = threadIdx.x + (GRADBLOCKX-2)*blockIdx.x - 1;
	int myY = threadIdx.y + (GRADBLOCKY-2)*blockIdx.y - 1;

	if((myX > arraysize.x) || (myY > arraysize.y)) return;

	bool IWrite = (threadIdx.x > 0) && (threadIdx.x < (GRADBLOCKX-1)) && (threadIdx.y > 0) && (threadIdx.y < (GRADBLOCKY-1));
	IWrite = IWrite && (myX < arraysize.x) && (myY < arraysize.y);

	myX = (myX + arraysize.x) % arraysize.x;
	myY = (myY + arraysize.y) % arraysize.y;

	int globAddr = myX + arraysize.x*myY;

	double deltaphi; // Store derivative of phi in one direction
	double mFactor; 
	__shared__ double phiLoc[GRADBLOCKX*GRADBLOCKY];

	phiLoc[myLocAddr] = phi[globAddr];

	__syncthreads(); // Make sure loaded phi is visible

	// coupling is exactly zero if rho <= rhomin
	if(IWrite) {
  		mFactor = cukern_computeMondFactor(rho[globAddr]);
		// compute dt * (dphi/dx)
		deltaphi         = LAMX*(phiLoc[myLocAddr+1]-phiLoc[myLocAddr-1]);
		fx[globAddr] = mFactor*deltaphi;

		// Calculate dt*(dphi/dy)
		if(coords == SQUARE) {
		deltaphi         = LAMY*(phiLoc[myLocAddr+GRADBLOCKX]-phiLoc[myLocAddr-GRADBLOCKX]);
		}
		if(coords == CYLINDRICAL) {
		// Converts d/dphi into physical distance based on R
		deltaphi         = LAMY*(phiLoc[myLocAddr+GRADBLOCKX]-phiLoc[myLocAddr-GRADBLOCKX]) / (RINNER + myX*DELTAR);
		}
		fy[globAddr]     = mFactor*deltaphi;
	}

}

/* Compute the gradient of 2d array phi with 4th order accuracy; store the results in f_x, f_y
 *    In cylindrical geometry, f_x -> f_r,
 *                             f_y -> f_phi
 */
template <geometryType_t coords>
__global__ void  cukern_computeScalarGradient2D_h4(double *rho, double *phi, double *fx, double *fy, int3 arraysize)
{
	int myLocAddr = threadIdx.x + GRADBLOCKX*threadIdx.y;

	int myX = threadIdx.x + (GRADBLOCKX-4)*blockIdx.x - 2;
	int myY = threadIdx.y + (GRADBLOCKY-4)*blockIdx.y - 2;

	if((myX > arraysize.x) || (myY > arraysize.y)) return;

	bool IWrite = (threadIdx.x > 1) && (threadIdx.x < (GRADBLOCKX-2)) && (threadIdx.y > 1) && (threadIdx.y < (GRADBLOCKY-2));
	IWrite = IWrite && (myX < arraysize.x) && (myY < arraysize.y);

	myX = (myX + arraysize.x) % arraysize.x;
	myY = (myY + arraysize.y) % arraysize.y;

	int globAddr = myX + arraysize.x*myY;

	double deltaphi; // Store derivative of phi in one direction
	double mFactor;
	__shared__ double phiLoc[GRADBLOCKX*GRADBLOCKY];

	phiLoc[myLocAddr] = phi[globAddr];

	__syncthreads(); // Make sure loaded phi is visible

	// coupling is exactly zero if rho <= rhomin
	if(IWrite) {	
  		mFactor = cukern_computeMondFactor(rho[globAddr]);
		// compute dt * (dphi/dx)
		deltaphi         = LAMX*(-phiLoc[myLocAddr+2] + 8*phiLoc[myLocAddr+1] - 8*phiLoc[myLocAddr-1] + phiLoc[myLocAddr-2]);
		fx[globAddr] = mFactor*deltaphi;

		// Calculate dt*(dphi/dy)
		if(coords == SQUARE) {
		deltaphi         = LAMY*(-phiLoc[myLocAddr+2*GRADBLOCKX] + 8*phiLoc[myLocAddr+1*GRADBLOCKX] - 8*phiLoc[myLocAddr-1*GRADBLOCKX] + phiLoc[myLocAddr-2*GRADBLOCKX]);
		}
		if(coords == CYLINDRICAL) {
		// Converts d/dphi into physical distance based on R
		deltaphi         = LAMY*(-phiLoc[myLocAddr+2*GRADBLOCKX] + 8*phiLoc[myLocAddr+1*GRADBLOCKX] - 8*phiLoc[myLocAddr-1*GRADBLOCKX] + phiLoc[myLocAddr-2*GRADBLOCKX])/(RINNER + myX*DELTAR);
		}
		fy[globAddr]     = mFactor*deltaphi;
	}

}

/* Compute the gradient of R-Z array phi with 2nd order accuracy; store the results in f_x, f_z
 *    In cylindrical geometry, f_x -> f_r
 */
__global__ void  cukern_computeScalarGradientRZ_h2(double *rho, double *phi, double *fx, double *fz, int3 arraysize)
{
	int myLocAddr = threadIdx.x + GRADBLOCKX*threadIdx.y;

	int myX = threadIdx.x + (GRADBLOCKX-2)*blockIdx.x - 1;
	int myY = threadIdx.y + (GRADBLOCKY-2)*blockIdx.y - 1;

	if((myX > arraysize.x) || (myY > arraysize.z)) return;

	bool IWrite = (threadIdx.x > 0) && (threadIdx.x < (GRADBLOCKX-1)) && (threadIdx.y > 0) && (threadIdx.y < (GRADBLOCKY-1));
	IWrite = IWrite && (myX < arraysize.x) && (myY < arraysize.z);

	myX = (myX + arraysize.x) % arraysize.x;
	myY = (myY + arraysize.z) % arraysize.z;

	int globAddr = myX + arraysize.x*myY;

	double deltaphi; // Store derivative of phi in one direction
	double mFactor;
	__shared__ double phiLoc[GRADBLOCKX*GRADBLOCKY];

	phiLoc[myLocAddr] = phi[globAddr];

	__syncthreads(); // Make sure loaded phi is visible

	// coupling is exactly zero if rho <= rhomin
	if(IWrite) {
  		mFactor = cukern_computeMondFactor(rho[globAddr]);
		// compute dt * (dphi/dx)
		deltaphi         = LAMX*(phiLoc[myLocAddr+1]-phiLoc[myLocAddr-1]);
		fx[globAddr]     = mFactor*deltaphi;

		// Calculate dt*(dphi/dz)
		deltaphi         = LAMZ*(phiLoc[myLocAddr+GRADBLOCKX]-phiLoc[myLocAddr-GRADBLOCKX]);
		fz[globAddr]     = mFactor*deltaphi;
	}

}

/* Compute the gradient of RZ array phi with 4th order accuracy; store the results in f_x, f_y
 *    In cylindrical geometry, f_x -> f_r,
 */
__global__ void  cukern_computeScalarGradientRZ_h4(double *rho, double *phi, double *fx, double *fz, int3 arraysize)
{
	int myLocAddr = threadIdx.x + GRADBLOCKX*threadIdx.y;

	int myX = threadIdx.x + (GRADBLOCKX-4)*blockIdx.x - 2;
	int myY = threadIdx.y + (GRADBLOCKY-4)*blockIdx.y - 2;

	if((myX > arraysize.x) || (myY > arraysize.z)) return;

	bool IWrite = (threadIdx.x > 1) && (threadIdx.x < (GRADBLOCKX-2)) && (threadIdx.y > 1) && (threadIdx.y < (GRADBLOCKY-2));
	IWrite = IWrite && (myX < arraysize.x) && (myY < arraysize.z);

	myX = (myX + arraysize.x) % arraysize.x;
	myY = (myY + arraysize.z) % arraysize.z;

	int globAddr = myX + arraysize.x*myY;

	double deltaphi; // Store derivative of phi in one direction
	double mFactor;
	__shared__ double phiLoc[GRADBLOCKX*GRADBLOCKY];

	phiLoc[myLocAddr] = phi[globAddr];

	__syncthreads(); // Make sure loaded phi is visible

	// coupling is exactly zero if rho <= rhomin
	if(IWrite) {
  		mFactor = cukern_computeMondFactor(rho[globAddr]);
		// compute dt * (dphi/dx)
		deltaphi         = LAMX*(-phiLoc[myLocAddr+2] + 8*phiLoc[myLocAddr+1] - 8*phiLoc[myLocAddr-1] + phiLoc[myLocAddr-2]);
		fx[globAddr]     = mFactor*deltaphi;

		// Calculate dt*(dphi/dz)
		deltaphi         = LAMZ*(-phiLoc[myLocAddr+2*GRADBLOCKX] + 8*phiLoc[myLocAddr+1*GRADBLOCKX] - 8*phiLoc[myLocAddr-1*GRADBLOCKX] + phiLoc[myLocAddr-2*GRADBLOCKX]);
		fz[globAddr]     = mFactor*deltaphi;
	}

}

/* The equations of motion for a rotating frame:
 *
 * a  = -[2 w X v + w X (w X r) ]
 * dv = -[2 w X v + w X (w X r) ] dt
 * dp = -rho dv = -rho [[2 w X v + w X (w X r) ] dt
 * dp = -[2 w X p + rho w X (w X r) ] dt
 *
 * w X p = |I  J  K | = <-w py, w px, 0> = u
 *         |0  0  w |
 *         |px py pz|
 *
 * w X r = <-w y, w x, 0> = s;
 * w X s = |I   J  K| = <-w^2 x, -w^2 y, 0> = -w^2<x,y,0> = b
 *         |0   0  w|
 *         |-wy wx 0|
 * dp = -[2 u + rho b] dt
 *    = -[2 w<-py, px, 0> - rho w^2 <x, y, 0>] dt
 *    = w dt [2<py, -px> + rho w <x, y>] in going to static frame
 *
 * dE = -v dot dp
 */
/* rho, E, Px, Py, Pz: arraysize-sized arrays
   omega: scalar
   Rx: [nx 1 1] sized array
   Ry: [ny 1 1] sized array */

#define JACOBI_ITER_MAX 4
#define NTH (SRCBLOCKX*SRCBLOCKY)

/* Solves the combined equations of a rotating frame and gravity,
 *
 * d/dt[ px ] = - rho (2 w X v + w X (w X r)).xhat - rho dphi/dx
 *     [ py ] = - rho (2 w X v + w X (w X r)).yhat - rho dphi/dy
 *     [ pz ] = - rho (2 w X v + w X (w X r)).zhat - rho dphi/dz
 *     [ E ]  = p.dp/2
 *
 * in either SQUARE or CYLINDRICAL coordinates using the implicit midpoint method,
 *
 *     y_half = y_0 + .5 dt f(y_half);
 *     y_1    = y_0 + dt    f(y_half);
 *
 * The implicit equations are iterated using JACOBI_ITER_MAX Jacobi steps updating vx then vy.
 * Frame rotation is always in the z-hat direction so no nonlinearity appears in the z direction.
 */
template <geometryType_t coords>
__global__ void  cukern_sourceComposite_IMP(double *fluidIn, double *Rvector, double *gravgrad, long pitch)
{
	__shared__ double shar[4*SRCBLOCKX*SRCBLOCKY];
	//__shared__ double px0[SRCBLOCKX*SRCBLOCKY], py0[SRCBLOCKX*SRCBLOCKY];

	/* strategy: XY files, fill in X direction, step in Y direction; griddim.y = Nz */
	int myx = threadIdx.x + SRCBLOCKX*blockIdx.x;
	int myy = threadIdx.y;
	int myz = blockIdx.y;
	int nx = devIntParams[0];
	int ny;
	if((coords == SQUARE) || (coords == CYLINDRICAL)) { // Not RZ coords
		ny = devIntParams[1];
	} else {
		ny = devIntParams[2];
	}

	if(myx >= devIntParams[0]) return; // return if x >= nx

	// Compute global index at the start
	int tileaddr = myx + nx*(myy + ny*myz);
	fluidIn += tileaddr;
	gravgrad += tileaddr;

	tileaddr = threadIdx.x + SRCBLOCKX*threadIdx.y;

	double locX = Rvector[myx];
	Rvector += nx; // Advances this to the Y array for below

	double locY;
	if((coords == CYLINDRICAL) || (coords == RZCYLINDRICAL)) locY = 0.0;

	double locRho, deltaphi;
	double vdel, dener;

	double vx0, vy0, vz0, vphi_combined;

	int jacobiIters;

	for(; myy < ny; myy += SRCBLOCKY) {
		// Only in square XY or XYZ coordinates must we account for a centripetal term in the the 2-direction
		if((coords == SQUARE) || (coords == RZSQUARE)) {
			locY = Rvector[myy];
		}

		locRho = *fluidIn;
		vx0 = fluidIn[2*pitch] / locRho; // convert to vr
		vy0 = fluidIn[3*pitch] / locRho; // convert to vy/vphi
		vz0 = fluidIn[4*pitch] / locRho;
		shar[tileaddr] = vx0;
		shar[tileaddr+NTH] = vy0;

		// Repeatedly perform fixed point iterations to solve the combined time differential operators
		// This yields the implicit Euler value for the midpoint (t = 0.5) if successful
		for(jacobiIters = 0; jacobiIters < JACOBI_ITER_MAX; jacobiIters++) {
			if((coords == SQUARE) || (coords == RZSQUARE)) {
				// Rotating frame contribution, vx
				vdel          = DT*OMEGA*(OMEGA*locX + 2.0*shar[tileaddr+NTH]); // delta-vx
				// Gravity gradient contribution, vx
				deltaphi      = gravgrad[0];
				vdel         -= deltaphi;
				// store predicted value for vx
				shar[tileaddr+2*NTH] = vx0 + .5*vdel;

				// rotating frame contribution, vy
				vdel          = -DT*OMEGA*(OMEGA*locY - 2*shar[tileaddr]);
				// gravity gradient contribution, vy
				deltaphi = gravgrad[pitch];
				vdel         -= deltaphi;
				// store predicted delta for vy
				shar[tileaddr+3*NTH] = vy0 + .5*vdel;
			} else {
				// Rotating frame contribution + cylindrical contribution, pr
				vphi_combined = OMEGA*locX + shar[tileaddr+NTH];
				vdel          = DT*vphi_combined*vphi_combined / locX; // a = (vphi + r*W)^2 / r
				// Gravity gradient contribution, pr
				deltaphi      = gravgrad[0];
				vdel         -= deltaphi;
				// store predicted value for pr
				shar[tileaddr+2*NTH] = vx0 + .5*vdel;

				// rotating frame contribution, ptheta
				vphi_combined = shar[tileaddr+NTH] + 2*locX*OMEGA; // a = -vr vphi - 2 vr w
				vdel          = -DT*shar[tileaddr]*vphi_combined / locX;
				// gravity gradient contribution, ptheta
				deltaphi = gravgrad[pitch];
				vdel         -= deltaphi;
				// store predicted delta for ptheta
				shar[tileaddr+3*NTH] = vy0 + .5*vdel;
			}

			__syncthreads();
			shar[tileaddr]     = shar[tileaddr+2*NTH];
			shar[tileaddr+NTH] = shar[tileaddr+3*NTH];
			__syncthreads();

		}

		// Compute minus the original XY/R-theta kinetic energy density
		dener = -(vx0*vx0+vy0*vy0+vz0*vz0);

		if((coords == SQUARE) || (coords == RZSQUARE)) {
			// Rotating frame contribution, vx
			vdel          = DT*OMEGA*(OMEGA*locX + 2.0*shar[tileaddr+2*NTH]); // delta-vx
			// Gravity gradient contribution, vx
			deltaphi      = gravgrad[0];
			vdel         -= deltaphi;
			// store value for vx
			vx0 += vdel;

			// rotating frame contribution, vy
			vdel          = -DT*OMEGA*(OMEGA*locY - 2*shar[tileaddr+NTH]);
			// gravity gradient contribution, vy
			deltaphi = gravgrad[pitch];
			vdel         -= deltaphi;
			// store delta for vy
			vy0 += vdel;
		} else {
			// Rotating frame contribution + cylindrical contribution, pr
			vphi_combined = OMEGA*locX + shar[tileaddr+NTH];
			vdel          = DT*vphi_combined*vphi_combined/locX;
			// Gravity gradient contribution, pr
			deltaphi      = gravgrad[0];
			vdel         -= deltaphi;
			// store predicted value for pr
			vx0 += vdel;

			// rotating frame contribution, ptheta
			vphi_combined = shar[tileaddr+NTH] + 2*locX*OMEGA;
			vdel          = -DT*shar[tileaddr]*vphi_combined/locX;
			// gravity gradient contribution, ptheta
			deltaphi = gravgrad[pitch];
			vdel         -= deltaphi;
			// store predicted delta for ptheta
			vy0 += vdel;
		}
		
		// Only a linear force in the Z direction: No need to iterate: Exact solution available
		deltaphi = gravgrad[2*pitch];
		vz0 -= deltaphi;

		// Add the new XY/R-theta kinetic energy density
		dener += vx0*vx0+vy0*vy0+vz0*vz0;

		fluidIn[2*pitch] = vx0 * locRho;
		fluidIn[3*pitch] = vy0 * locRho;
		fluidIn[4*pitch] = vz0 * locRho;
		// Change in total energy is exactly the work done by forces
		fluidIn[pitch] += .5*locRho*dener;

		// Hop pointers forward
		fluidIn += nx*SRCBLOCKY;
		gravgrad+= nx*SRCBLOCKY;
	}
}

#define GL4_C1 0.2113248654051871344705659794271924
#define GL4_C2 0.7886751345948128655294340205728076
#define GL4_A11 .25
#define GL4_A12 -0.03867513459481286552943402057280764
#define GL4_A21 0.5386751345948128655294340205728076
#define GL4_A22 .25

/* Solves the combined equations of a rotating frame and gravity
 * in either SQUARE or CYLINDRICAL coordinates using 4th order
 * Gauss-Legendre quadrature: This requires simultaneous self-consistent
 * solution of 2N equations at 2 intermediate points, for N=2 (vx and vy)
 * followed by evaluation of the output sum.
 *
 * The implicit solve makes a forward Euler starter prediction before
 * applying Jacobi iterations to update in the order
 *     vx1, vy1, vx2, vy2
 * for up to JACOBI_MAX_ITER times.
 */
template <geometryType_t coords>
__global__ void  cukern_sourceComposite_GL4(double *fluidIn, double *Rvector, double *gravgrad, long pitch)
{
	__shared__ double shar[4*SRCBLOCKX*SRCBLOCKY];
	//__shared__ double px0[SRCBLOCKX*SRCBLOCKY], py0[SRCBLOCKX*SRCBLOCKY];

	/* strategy: XY files, fill in X direction, step in Y direction; griddim.y = Nz */
	int myx = threadIdx.x + SRCBLOCKX*blockIdx.x;
	int myy = threadIdx.y;
	int myz = blockIdx.y;
	int nx = devIntParams[0];
	int ny;
	if((coords == SQUARE) || (coords == CYLINDRICAL)) { // Not RZ coords
		ny = devIntParams[1];
	} else {
		ny = devIntParams[2];
	}

	if(myx >= devIntParams[0]) return; // return if x >= nx

	// Compute global index at the start
	int tileaddr = myx + nx*(myy + ny*myz);
	fluidIn += tileaddr;
	gravgrad += tileaddr;

	tileaddr = threadIdx.x + SRCBLOCKX*threadIdx.y;

	double locX = Rvector[myx];
	Rvector += nx; // Advances this to the Y array for below

	double locY;
	if((coords == CYLINDRICAL) || (coords == RZCYLINDRICAL)) locY = 0.0;

	double locRho, deltaphi;
	double vdel, dener;

	double vxA, vxB, vyA, vyB;

	double q1, q2; // temp vars?

	int jacobiIters;

	for(; myy < ny; myy += SRCBLOCKY) {
		// Only in square XY or XYZ coordinates must we account for a centripetal term in the the 2-direction
		if((coords == SQUARE) || (coords == RZSQUARE)) {
			locY = Rvector[myy];
		}

		locRho = *fluidIn;
		vxA = fluidIn[2*pitch] / locRho; // convert to vr
		vyA = fluidIn[3*pitch] / locRho; // convert to vy/vphi
		shar[tileaddr] = vxA;
		shar[tileaddr+NTH] = vyA;

		// Generate a 1st order prediction for what the values will be using fwd euler
		// This is worth roughly 1 iteration but as can be seen will take way less time
		if((coords == SQUARE) || (coords == RZSQUARE)) {
		/////
		/////
		/////
		} else {
			q1 = OMEGA*locX + vyA;
			q2 = -vxA*(vyA + 2*OMEGA*locX);

			deltaphi      = gravgrad[0];
			vxB  = vxA + GL4_C2*(DT*q1*q1/locX - deltaphi);
			vxA += GL4_C1*(DT*q1*q1/locX - deltaphi);

			deltaphi = gravgrad[pitch];
			vyB  = vyA + GL4_C2*(DT*q2/locX - deltaphi);
			vyA += GL4_C1*(DT*q2/locX - deltaphi);
		}

		// Repeatedly perform fixed point iterations to solve the combined differential operators
		for(jacobiIters = 0; jacobiIters < JACOBI_ITER_MAX; jacobiIters++) {
			if((coords == SQUARE) || (coords == RZSQUARE)) {
			/////////////////
			/////////////// ruh-roh
			///////////////
			} else {
				// Rotating frame contribution + cylindrical contribution, vr, step A
				q1 = OMEGA*locX + vyA;
				q2 = OMEGA*locX + vyB;
				// Gravity gradient contribution, vr, step A
				deltaphi      = gravgrad[0];
				// Improve estimates for radial velocity
				vdel         = -GL4_C1*deltaphi + DT*(q1*q1*GL4_A11 + q2*q2*GL4_A12)/locX;
				vxA = shar[tileaddr] + vdel;
				vdel         = -GL4_C2*deltaphi + DT*(q1*q1*GL4_A21 + q2*q2*GL4_A22)/locX;
				vxB = shar[tileaddr] + vdel;

				// Load azimuthal gravity gradient
				deltaphi = gravgrad[pitch];

				q1 = GL4_A11*vxA*(vyA+2*locX*OMEGA);
				q2 = vxB*(vyB+2*locX*OMEGA); // Note we leave the GL quadrature coefficient off and can reuse q2
				vdel          = -DT*(q1+GL4_A12*q2)/locX - GL4_C1 * deltaphi;
				vyA = shar[tileaddr + NTH] + vdel;

				q1 = GL4_A21*vxA*(vyA+2*locX*OMEGA);
				vdel          = -DT*(q1+GL4_A22*q2)/locX - GL4_C2 * deltaphi;
				vyB = shar[tileaddr+NTH] + vdel;
			}

		}

		// Compute minus the original kinetic energy density
		q1 = shar[tileaddr];
		q2 = shar[tileaddr+NTH];
		dener = -(q1*q1+q2*q2);
		q1 = fluidIn[4*pitch] / locRho;
		dener -= q1*q1;

		if((coords == SQUARE) || (coords == RZSQUARE)) {
		///////////////
		//////////// ruh-roh
		/////////////
		} else {
			// evaluate final Vr
			q1 = OMEGA*locX + vyA;
			q2 = OMEGA*locX + vyB;
			deltaphi = gravgrad[0];
			shar[tileaddr] = shar[tileaddr] - deltaphi + .5*DT*(q1*q1+q2*q2)/locX;

			// evalute final Vphi
			deltaphi = gravgrad[pitch];
			shar[tileaddr+NTH] = shar[tileaddr+NTH] - deltaphi - .5*DT*(vxA*(vyA+2*OMEGA*locX)+vxB*(vyB+2*OMEGA*locX))/locX;
		}
		vxA = shar[tileaddr];
		vyA = shar[tileaddr+NTH];

		// Only a linear force in the Z direction: No need to iterate: Exact solution available
		deltaphi = gravgrad[2*pitch];
		q1 = fluidIn[4*pitch] / locRho - deltaphi;

		// Add the new XY/R-theta kinetic energy density
		dener += (vxA*vxA + vyA*vyA + q1*q1);

		fluidIn[2*pitch] = vxA * locRho;
		fluidIn[3*pitch] = vyA * locRho;
		fluidIn[4*pitch] = q1  * locRho;
		// Change in total energy is exactly the work done by forces
		fluidIn[pitch] += .5*locRho*dener;

		// Hop pointers forward
		fluidIn += nx*SRCBLOCKY;
		gravgrad+= nx*SRCBLOCKY;
	}
}

#define GL6_C1 0.28918148932210804453
#define GL6_C2 .5
#define GL6_C3 0.71081851067789195547
#define GL6_A11 0.13888888888888888889
#define GL6_A21 0.30026319498086459244
#define GL6_A31 0.26798833376246945173
#define GL6_A12 -0.035976667524938903456
#define GL6_A22 0.22222222222222222222
#define GL6_A32 0.4804211119693833479
#define GL6_A13 0.0097894440153083260496
#define GL6_A23 -0.02248541720308681466
#define GL6_A33 0.13888888888888888889
#define GL6_B1 0.27777777777777777778
#define GL6_B2 0.44444444444444444444
#define GL6_B3 0.27777777777777777778
/* Solves the combined equations of a rotating frame and gravity
 * in either SQUARE or CYLINDRICAL coordinates using 6th order
 * Gauss-Legendre quadrature: This requires simultaneous self-consistent
 * solution of 3N equations at 3 intermediate points, for N=2 (vx and vy)
 * followed by evaluation of the output sum.
 *
 * The implicit solve makes a forward Euler starter prediction before
 * applying Jacobi iterations to update in the order
 *     vx1, vx2, vx3, vy1, vy2, vy3
 * for up to JACOBI_MAX_ITER times.
 */
template <geometryType_t coords>
__global__ void  cukern_sourceComposite_GL6(double *fluidIn, double *Rvector, double *gravgrad, long pitch)
{
	__shared__ double shar[6*SRCBLOCKX*SRCBLOCKY];
	//__shared__ double px0[SRCBLOCKX*SRCBLOCKY], py0[SRCBLOCKX*SRCBLOCKY];

	// strategy: XY files, fill in X direction, step in Y direction; griddim.y = Nz
	int myx = threadIdx.x + SRCBLOCKX*blockIdx.x;
	int myy = threadIdx.y;
	int myz = blockIdx.y;
	int nx = devIntParams[0];
	int ny;
	if((coords == SQUARE) || (coords == CYLINDRICAL)) { // Not RZ coords
		ny = devIntParams[1];
	} else {
		ny = devIntParams[2];
	}

	if(myx >= devIntParams[0]) return; // return if x >= nx

	// Compute global index at the start
	int tileaddr = myx + nx*(myy + ny*myz);
	fluidIn += tileaddr;
	gravgrad += tileaddr;

	tileaddr = threadIdx.x + SRCBLOCKX*threadIdx.y;

	double locX = Rvector[myx];
	Rvector += nx; // Advances this to the Y array for below

	double locY;
	if((coords == CYLINDRICAL) || (coords == RZCYLINDRICAL)) locY = 0.0;

	double locRho, deltaphi;
	double vdel, dener;

	double vxA, vxB, vxC, vyA, vyB, vyC;

	double q1, q2, q3; // temp vars?

	int jacobiIters;

	for(; myy < ny; myy += SRCBLOCKY) {
		// Only in square XY or XYZ coordinates must we account for a centripetal term in the the 2-direction
		if((coords == SQUARE) || (coords == RZSQUARE)) {
			locY = Rvector[myy];
		}

		locRho = *fluidIn;
		vxA = fluidIn[2*pitch] / locRho; // convert to vr
		vyA = fluidIn[3*pitch] / locRho; // convert to vy/vphi
		shar[tileaddr] = vxA;
		shar[tileaddr+NTH] = vyA;

		// Generate a 1st order prediction for what the values will be using fwd euler
		// This is worth roughly 1 iteration but as can be seen will take way less time
		if((coords == SQUARE) || (coords == RZSQUARE)) {
		/////
		/////
		/////
		} else {
			q1 = OMEGA*locX + vyA;
			q2 = -vxA*(vyA + 2*OMEGA*locX);

			deltaphi      = gravgrad[0];
			vxC  = vxA + GL6_C3*(DT*q1*q1/locX - deltaphi);
			vxB  = vxA + GL6_C2*(DT*q1*q1/locX - deltaphi);
			vxA +=       GL6_C1*(DT*q1*q1/locX - deltaphi);

			deltaphi = gravgrad[pitch];
			vyC  = vyA + GL6_C3*(DT*q2/locX - deltaphi);
			vyB  = vyA + GL6_C2*(DT*q2/locX - deltaphi);
			vyA +=       GL6_C1*(DT*q2/locX - deltaphi);
		}

		// Repeatedly perform fixed point iterations to solve the combined time differential operators
		// This yields the implicit Euler value for the midpoint (t = 0.5) if successful
		for(jacobiIters = 0; jacobiIters < JACOBI_ITER_MAX; jacobiIters++) {
			if((coords == SQUARE) || (coords == RZSQUARE)) {
			///////////////
			/////////////// ruh-roh
			///////////////
			} else {
				// Rotating frame contribution + cylindrical contribution, Vr:
				// Depends only on Vtheta... improve all estimates for Vr now:
				q1 = OMEGA*locX + vyA;
				q2 = OMEGA*locX + vyB;
				q3 = OMEGA*locX + vyC;
				// Gravity gradient contribution, vr
				deltaphi      = gravgrad[0];

				vdel         = -GL6_C1*deltaphi + DT*(q1*q1*GL6_A11 + q2*q2*GL6_A12+q3*q3*GL6_A13)/locX;
				vxA = shar[tileaddr] + vdel;
				vdel         = -GL6_C2*deltaphi + DT*(q1*q1*GL6_A21 + q2*q2*GL6_A22+q3*q3*GL6_A23)/locX;
				vxB = shar[tileaddr] + vdel;
				vdel         = -GL6_C3*deltaphi + DT*(q1*q1*GL6_A31 + q2*q2*GL6_A32+q3*q3*GL6_A33)/locX;
				vxC = shar[tileaddr] + vdel;

				// gravity gradient contribution, vtheta
				deltaphi = gravgrad[pitch];
				// rotating frame contribution, vtheta
				q1 = vxA*(vyA+2*locX*OMEGA);
				q2 = vxB*(vyB+2*locX*OMEGA);
				q3 = vxC*(vyC+2*locX*OMEGA);
				vdel          = -DT*(GL6_A11*q1 + GL6_A12*q2+GL6_A13*q3)/locX - GL6_C1 * deltaphi;
				vyA = shar[tileaddr+NTH] + vdel;

				// update q1 & improve vyB
				q1 = vxA*(vyA+2*locX*OMEGA);
				vdel          = -DT*(GL6_A21*q1 + GL6_A22*q2+GL6_A23*q3)/locX - GL6_C2 * deltaphi;
				vyB = shar[tileaddr+NTH] + vdel;

				// update q2 & improve vyC
				q2 = vxB*(vyB+2*locX*OMEGA);
				vdel          = -DT*(GL6_A31*q1 + GL6_A32*q2+GL6_A33*q3)/locX - GL6_C3 * deltaphi;
				vyC = shar[tileaddr+NTH] + vdel;
			}

		}

		// Compute minus the original kinetic energy density
		q1 = shar[tileaddr];
		q2 = shar[tileaddr+NTH];
		dener = -(q1*q1+q2*q2);
		q1 = fluidIn[4*pitch] / locRho;
		dener -= q1*q1;

		if((coords == SQUARE) || (coords == RZSQUARE)) {
		///////////////
		//////////// ruh-roh
		/////////////
		} else {
			// evaluate final Vr
			q1 = OMEGA*locX + vyA;
			q2 = OMEGA*locX + vyB;
			q3 = OMEGA*locX + vyC;
			deltaphi = gravgrad[0];
			shar[tileaddr] = shar[tileaddr] - deltaphi + DT*(GL6_B1*q1*q1 + GL6_B2*q2*q2 + GL6_B3*q3*q3)/locX;

			// evalute final Vphi
			q1 = vxA*(vyA+2*OMEGA*locX);
			q2 = vxB*(vyB+2*OMEGA*locX);
			q3 = vxC*(vyC+2*OMEGA*locX);
			deltaphi = gravgrad[pitch];
			shar[tileaddr+NTH] = shar[tileaddr+NTH] - deltaphi - DT*(GL6_B1*q1 + GL6_B2*q2 + GL6_B3*q3)/locX;
		}
		vxA = shar[tileaddr];
		vyA = shar[tileaddr+NTH];

		// Only a linear force in the Z direction: No need to iterate: Exact solution available
		deltaphi = gravgrad[2*pitch];
		q1 = fluidIn[4*pitch] / locRho - deltaphi;

		// Add the new XY/R-theta kinetic energy density
		dener += (vxA*vxA + vyA*vyA + q1*q1);

		fluidIn[2*pitch] = vxA * locRho;
		fluidIn[3*pitch] = vyA * locRho;
		fluidIn[4*pitch] = q1  * locRho;
		// Change in total energy is exactly the work done by forces
		fluidIn[pitch] += .5*locRho*dener;

		// Hop pointers forward
		fluidIn += nx*SRCBLOCKY;
		gravgrad+= nx*SRCBLOCKY;
	}
}

/* Solves the combined equations of a rotating frame and gravity
 * in either SQUARE or CYLINDRICAL coordinates using the well-known
 * 4th order explicit multistage method of Runge & Kutta.
 */
template <geometryType_t coords>
__global__ void  cukern_sourceComposite_RK4(double *fluidIn, double *Rvector, double *gravgrad, long pitch)
{
	__shared__ double shar[4*SRCBLOCKX*SRCBLOCKY];
	//__shared__ double px0[SRCBLOCKX*SRCBLOCKY], py0[SRCBLOCKX*SRCBLOCKY];

	/* strategy: XY files, fill in X direction, step in Y direction; griddim.y = Nz */
	int myx = threadIdx.x + SRCBLOCKX*blockIdx.x;
	int myy = threadIdx.y;
	int myz = blockIdx.y;
	int nx = devIntParams[0];
	int ny;
	if((coords == SQUARE) || (coords == CYLINDRICAL)) { // Not RZ coords
		ny = devIntParams[1];
	} else {
		ny = devIntParams[2];
	}

	if(myx >= devIntParams[0]) return; // return if x >= nx

	// Compute global index at the start
	int tileaddr = myx + nx*(myy + ny*myz);
	fluidIn += tileaddr;
	gravgrad += tileaddr;

	tileaddr = threadIdx.x + SRCBLOCKX*threadIdx.y;

	double locX = Rvector[myx];
	Rvector += nx; // Advances this to the Y array for below

	double locY;
	if((coords == CYLINDRICAL) || (coords == RZCYLINDRICAL)) locY = 0.0;

	double locRho, deltaphi;
	double vdel, dener;

	double vx0, vy0, vxS, vyS, vphi_combined;

	int stageCount; double alpha, beta;
	alpha = 1.0/6.0;
	beta = 0.5; 

	for(; myy < ny; myy += SRCBLOCKY) {
		// Only in square XY or XYZ coordinates must we account for a centripetal term in the the 2-direction
		if((coords == SQUARE) || (coords == RZSQUARE)) {
			locY = Rvector[myy];
		}

		locRho = *fluidIn;
		vx0 = fluidIn[2*pitch] / locRho; // convert to vr
		vy0 = fluidIn[3*pitch] / locRho; // convert to vy/vphi

		shar[tileaddr] = vxS = vx0;
		shar[tileaddr+NTH] = vyS = vy0;

		for(stageCount = 0; stageCount < 4; stageCount++) {

		if((coords == SQUARE) || (coords == RZSQUARE)) {
			// Rotating frame contribution, vx
			vdel          = DT*OMEGA*(OMEGA*locX + 2.0*vyS); // delta-vx
			// Gravity gradient contribution, vx
			deltaphi      = gravgrad[0];
			vdel         -= deltaphi;
			// store predicted value for vx
			shar[tileaddr+2*NTH] = vx0 + beta*vdel;
			// Accumulate delta
			shar[tileaddr]      += alpha*vdel;

			// rotating frame contribution, vy
			vdel          = -DT*OMEGA*(OMEGA*locY - 2*vxS);
			// gravity gradient contribution, vy
			deltaphi = gravgrad[pitch];
			vdel         -= deltaphi;
			// store predicted delta for vy
			shar[tileaddr+3*NTH] = vy0 + beta*vdel;
			// Accumulate delta
			shar[tileaddr]      += alpha*vdel;
		} else {
			// Rotating frame contribution + cylindrical contribution, pr
			vphi_combined = OMEGA*locX + shar[tileaddr+NTH];
			vdel          = DT*vphi_combined*vphi_combined / locX;
			// Gravity gradient contribution, pr
			deltaphi      = gravgrad[0];
			vdel         -= deltaphi;
			// store predicted value for vr
			shar[tileaddr+2*NTH] = vx0 + beta*vdel;
			// Accumulate delta
			shar[tileaddr]      += alpha*vdel;

			// rotating frame contribution, ptheta
			vphi_combined = shar[tileaddr+NTH] + 2*locX*OMEGA;
			vdel          = -DT*shar[tileaddr]*vphi_combined / locX;
			// gravity gradient contribution, ptheta
			deltaphi = gravgrad[pitch];
			vdel         -= deltaphi;
			// store predicted delta for vtheta
			shar[tileaddr+3*NTH] = vy0 + beta*vdel;
			// Accumulate delta
			shar[tileaddr]      += alpha*vdel;
		}

		__syncthreads();
		vxS = shar[tileaddr + 2*NTH];
		vyS = shar[tileaddr + 3*NTH];
		__syncthreads();

		switch(stageCount) {
			case 0: alpha = 1.0/3.0; break;
			case 1: beta = 1.0; break;
			case 2: alpha = 1.0/6.0; break;
		}

		}

		vphi_combined = fluidIn[4*pitch] / locRho; // vz...

		dener = -(vx0*vx0+vy0*vy0+vphi_combined*vphi_combined);

		deltaphi = gravgrad[2*pitch];
		vphi_combined -= deltaphi;

		// Download the final values from shmem
		vxS = shar[tileaddr];
		vyS = shar[tileaddr + NTH];

		// Add the new XY/R-theta kinetic energy density
		dener += vxS*vxS+vyS*vyS+vphi_combined*vphi_combined;

		fluidIn[2*pitch] = vxS * locRho;
		fluidIn[3*pitch] = vyS * locRho;
		fluidIn[4*pitch] = vphi_combined * locRho;

		// Change in total energy is exactly the work done by forces
		fluidIn[pitch] += .5*locRho*dener;

		// Hop pointers forward
		fluidIn += nx*SRCBLOCKY;
		gravgrad+= nx*SRCBLOCKY;
	}
}



/* Simple kernel:
 * Given in[0 ... (nodeN-1)], copies the segment in[partX0 ... (partX0 + partNX -1)] to out[0 ... (partNX-1)]
 * and helpfully wraps addresses circularly
 * invoke with gridDim.x * blockDim.x >= partNX
 */
__global__ void cukern_FetchPartitionSubset1D(double *in, int nodeN, double *out, int partX0, int partNX)
{
// calculate output address
int addrOut = threadIdx.x + blockDim.x * blockIdx.x;
if(addrOut >= partNX) return;

// Affine map back to input address
int addrIn = addrOut + partX0;
if(addrIn < 0) {
	double delta = in[1]-in[0];
	out[addrOut] = in[0]+delta*addrIn;
} else {
	out[addrOut] = in[addrIn];
}

}

/* Converts the fluid slab array from conservative
 * 		[rho, Etotal, px, py, pz]
 * variables to
 * 		[rho, Einternal, vx, vy, vz]
 * primitive variables which may be more suited for some computations. */
__global__ void cukern_cvtToPrimitiveVars(double *fluid, long partNumel, long pitch)
{
	unsigned int globAddr = threadIdx.x + blockDim.x*blockIdx.x;

	if(globAddr >= partNumel) return;

	double rhoinv, p[3], Etot;

	fluid += globAddr;

	for(; globAddr < partNumel; globAddr += blockDim.x*gridDim.x) {
		rhoinv = 1.0/fluid[0];
		Etot = fluid[pitch];
		p[0] = fluid[2*pitch];
		p[1] = fluid[3*pitch];
		p[2] = fluid[4*pitch];

		fluid[2*pitch] = p[0]*rhoinv;
		fluid[3*pitch] = p[1]*rhoinv;
		fluid[4*pitch] = p[2]*rhoinv;

		Etot -= .5*(p[0]*p[0]+p[1]*p[1]+p[2]*p[2])*rhoinv;
		fluid[pitch] = Etot;

		fluid += blockDim.x*gridDim.x;
	}
}


/* Converts the fluid slab array from primitive
 * 		[rho, Einternal, vx, vy, vz]
 * variables to conservative
 * 		[rho, Etotal, px, py, pz]
 * variables which are mandatory for conservative flux differencing */
__global__ void cukern_cvtToConservativeVars(double *fluid, long partNumel, long pitch)
{
	unsigned int globAddr = threadIdx.x + blockDim.x*blockIdx.x;

	if(globAddr >= partNumel) return;

	double rho, v[3], Eint;

	fluid += globAddr;

	for(; globAddr < partNumel; globAddr += blockDim.x*gridDim.x) {
		rho = fluid[0];
		Eint = fluid[pitch];
		v[0] = fluid[2*pitch];
		v[1] = fluid[3*pitch];
		v[2] = fluid[4*pitch];

		fluid[2*pitch] = v[0]*rho;
		fluid[3*pitch] = v[1]*rho;
		fluid[4*pitch] = v[2]*rho;

		Eint += .5*(v[0]*v[0]+v[1]*v[1]+v[2]*v[2])*rho;
		fluid[pitch] = Eint;

		fluid += blockDim.x*gridDim.x;
	}

}

// FIXME implement cvtGasdustToBarydelta

// FIXME implement cvtBarydeltaToGasdust

// Needed with the gradient calculators in 2D because they leave the empty directions uninitialized
// Vomits the value f into array x, from x[0] to x[numel-1]
__global__ void writeScalarToVector(double *x, long numel, double f)
{
	long a = threadIdx.x + blockDim.x*blockIdx.x;

	for(; a < numel; a+= blockDim.x*gridDim.x) {
		x[a] = f;

	}

}
