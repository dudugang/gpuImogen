#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  // wrapper for hipFree().
  if((nlhs != 0) || (nrhs == 0)) mexErrMsgTxt("GPU_free: syntax is GPU_free(arbitrarily many GPU_Types, gpu tags, or ImogenArrays)");

  CHECK_CUDA_ERROR("Entering GPU_free()");
  MGArray t[nrhs];

  int worked = accessMGArrays(prhs, 0, nrhs-1, &t[0]);

  int i, j;

  for(i = 0; i < nrhs; i++) {
	  if(t[i].numSlabs < 1) continue; // This is a slab reference and was never actually allocated. Ignore it.
    for(j = 0; j < t[i].nGPUs; j++) {
      hipSetDevice(t[i].deviceID[j]);
      CHECK_CUDA_ERROR("hipSetDevice()");
      hipError_t result = hipFree(t[i].devicePtr[j]);
      CHECK_CUDA_ERROR("After GPU_free()");
    }
  }

return;
}
