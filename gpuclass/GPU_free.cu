#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

// static paramaters

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  // wrapper for hipFree().
  if((nlhs != 0) || (nrhs != 1)) mexErrMsgTxt("GPU_free: syntax is GPU_free(GPU_MemPtr object)");

  if(mxGetClassID(prhs[0]) != mxINT64_CLASS) mexErrMsgTxt("GPU_free: pass a not-gpupointer");

  int64_t *t = (int64_t *)mxGetData(prhs[0]);

  double *d = (double *)t[0];

  hipError_t result = hipFree(d);

  if(result == hipErrorInvalidDevicePointer) mexErrMsgTxt("GPU_free: cuda claims invalid device pointer.");

  return;
}
