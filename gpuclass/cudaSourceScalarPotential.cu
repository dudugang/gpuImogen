#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"

#include "cudaCommon.h"
#include "cudaSourceScalarPotential.h"

#define BLOCKDIMX 18
#define BLOCKDIMY 18

template <geometryType_t coords>
__global__ void  cukern_applyScalarPotential(double *rho, double *E, double *px, double *py, double *pz, double *phi, int3 arraysize);
/*mass.gputag, mom(1).gputag, mom(2).gputag, mom(3).gputag, ener.gputag, run.potentialField.gputag, 2*run.time.dTime);*/

template <geometryType_t coords>
__global__ void  cukern_applyScalarPotential_2D(double *rho, double *E, double *px, double *py, double *phi, int3 arraysize);

__constant__ __device__ double devLambda[9];

#define LAMX devLambda[0]
#define LAMY devLambda[1]
#define LAMZ devLambda[2]

// Define: F = -beta * rho * grad(phi)
// rho_g = density for full effect of gravity 
// rho_c = minimum density to feel gravity at all
// beta = { rho_g < rho         : 1                                 }
//        { rho_c < rho < rho_g : [(rho-rho_c)/(rho_rho_g-rho_c)]^2 }
//        {         rho < rho_c : 0                                 }

// This provides a continuous (though not differentiable at rho = rho_g) way to surpress gravitation of the background fluid
// The original process of cutting gravity off below a critical density a few times the minimum
// density is believed to cause "blowups" at the inner edge of circular flow profiles due to being
// discontinuous. If even smoothness is insufficient and smooth differentiability is required,
// a more-times-continuous profile can be constructed, but let's not go there unless forced.

// Density below which we force gravity effects to zero
#define RHOMIN devLambda[3]
#define RHOGRAV devLambda[4]
// 1 / (rho_g - rho_c)
#define G1 devLambda[5]
// rho_c / (rho_g - rho_c)
#define G2 devLambda[6]
#define RINNER devLambda[7]
#define DELTAR devLambda[8]


void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{

    if ((nrhs!=6) || (nlhs != 0)) mexErrMsgTxt("Wrong number of arguments: need cudaApplyScalarPotential(FluidManager, phi, dt, GeometryManager, rhomin, rho_fullg)\n");

    if(CHECK_CUDA_ERROR("entering cudaSourceScalarPotential") != SUCCESSFUL) { DROP_MEX_ERROR("Failed upon entry to cudaSourceScalarPotential."); }
    
    // Get source array info and create destination arrays
    MGArray fluid[5];
    MGArray phi;
    int worked = MGA_accessMatlabArrays(prhs, 1, 1, &phi);
    if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) { DROP_MEX_ERROR("Failed to access input arrays."); }

    // Each partition uses the same common parameters
    double dt = *mxGetPr(prhs[2]);
    GeometryParams geom = accessMatlabGeometryClass(prhs[3]);
    double rhoMinimum = *mxGetPr(prhs[4]); /* minimum rho, rho_c */
    double rhoFull    = *mxGetPr(prhs[5]); /* rho_g */

    int numFluids = mxGetNumberOfElements(prhs[0]);
    int fluidct;
// FIXME require separate rhomin/rho_fullg per fluid becuase they will generally have distinct characteristic scales of density.
    for(fluidct = 0; fluidct < numFluids; fluidct++) {
    	worked = MGA_accessFluidCanister(prhs[0], fluidct, &fluid[0]);
#if 0
    	mxArray *flprop = mxGetProperty(prhs[0], fluidct, "MINMASS");
    	if(flprop != NULL) {
    		rhoMinimum = *((double *)mxGetPr(flprop));
    	} else {
    		worked = ERROR_NULL_POINTER;
    	}
#endif
    	if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) break;
    	worked = sourcefunction_ScalarPotential(&fluid[0], &phi, dt, geom, rhoMinimum, rhoFull);
    	if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) break;
    }

}

int sourcefunction_ScalarPotential(MGArray *fluid, MGArray *phi, double dt, GeometryParams geom, double minRho, double rhoFullGravity)
{
    double *dx = &geom.h[0];

    dim3 gridsize, blocksize;
    int3 arraysize;
    int i, sub[6];
    int worked;

    double lambda[9];
    lambda[0] = dt/(2.0*dx[0]);
    lambda[1] = dt/(2.0*dx[1]);
    lambda[2] = dt/(2.0*dx[2]);
    lambda[3] = minRho; /* minimum rho, rho_c */
    lambda[4] = rhoFullGravity; /* rho_g */

    lambda[5] = 1.0/(lambda[4] - lambda[3]); /* 1/(rho_g - rho_c) */
    lambda[6] = lambda[3]*lambda[5];

    lambda[7] = geom.Rinner;
    lambda[8] = dx[1];

    for(i = 0; i < fluid->nGPUs; i++) {
    	hipSetDevice(fluid->deviceID[i]);
    	hipMemcpyToSymbol(HIP_SYMBOL(devLambda), lambda, 9*sizeof(double), 0, hipMemcpyHostToDevice);
    	worked = CHECK_CUDA_ERROR("hipMemcpyToSymbol");
    	if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) break;
    }

    if(worked != SUCCESSFUL) return worked;

    int isThreeD = (fluid->dim[2] > 1);

    // Iterate over all partitions, and here we GO!
    for(i = 0; i < fluid->nGPUs; i++) {
    	hipSetDevice(fluid->deviceID[i]);
        calcPartitionExtent(fluid, i, sub);


        arraysize.x = sub[3]; arraysize.y = sub[4]; arraysize.z = sub[5];

        blocksize = makeDim3(BLOCKDIMX, BLOCKDIMY, 1);
        gridsize.x = arraysize.x / (blocksize.x - 2); gridsize.x += ((blocksize.x-2) * gridsize.x < arraysize.x);
        gridsize.y = arraysize.y / (blocksize.y - 2); gridsize.y += ((blocksize.y-2) * gridsize.y < arraysize.y);
        gridsize.z = 1;

        if(isThreeD) {
	if(geom.shape == SQUARE) { 
        cukern_applyScalarPotential<SQUARE><<<gridsize, blocksize>>>(
            fluid[0].devicePtr[i],
            fluid[1].devicePtr[i],
            fluid[2].devicePtr[i],
            fluid[3].devicePtr[i],
            fluid[4].devicePtr[i],
            phi->devicePtr[i], arraysize);
	    }
	    if(geom.shape == CYLINDRICAL) {
        cukern_applyScalarPotential<CYLINDRICAL><<<gridsize, blocksize>>>(
            fluid[0].devicePtr[i],
            fluid[1].devicePtr[i],
            fluid[2].devicePtr[i],
            fluid[3].devicePtr[i],
            fluid[4].devicePtr[i],
            phi->devicePtr[i], arraysize);
}
        } else {
	if(geom.shape == SQUARE) {
        	cukern_applyScalarPotential_2D<SQUARE><<<gridsize, blocksize>>>(
        	            fluid[0].devicePtr[i],
        	            fluid[1].devicePtr[i],
        	            fluid[2].devicePtr[i],
        	            fluid[3].devicePtr[i],
        	            phi->devicePtr[i], arraysize);
			    }
			    if(geom.shape == CYLINDRICAL) {
        	cukern_applyScalarPotential_2D<CYLINDRICAL><<<gridsize, blocksize>>>(
        	            fluid[0].devicePtr[i],
        	            fluid[1].devicePtr[i],
        	            fluid[2].devicePtr[i],
        	            fluid[3].devicePtr[i],
        	            phi->devicePtr[i], arraysize);

			    }

        }
        worked = CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, fluid, i, "scalar potential kernel");
        if(worked != SUCCESSFUL) break;
    }

    return CHECK_IMOGEN_ERROR(worked);

}

/*
 * dP = -rho grad(phi) dt
 * dE = -rho v \cdot grad(phi) dt
 */
template <geometryType_t coords>
__global__ void  cukern_applyScalarPotential(double *rho, double *E, double *px, double *py, double *pz, double *phi, int3 arraysize)
{

int myLocAddr = threadIdx.x + BLOCKDIMX*threadIdx.y;

int myX = threadIdx.x + (BLOCKDIMX-2)*blockIdx.x - 1;
int myY = threadIdx.y + (BLOCKDIMY-2)*blockIdx.y - 1;

if((myX > arraysize.x) || (myY > arraysize.y)) return;

bool IWrite = (threadIdx.x > 0) && (threadIdx.x < (BLOCKDIMX-1)) && (threadIdx.y > 0) && (threadIdx.y < (BLOCKDIMY-1));
IWrite = IWrite && (myX < arraysize.x) && (myY < arraysize.y);

myX = (myX + arraysize.x) % arraysize.x;
myY = (myY + arraysize.y) % arraysize.y;

int globAddr = myX + arraysize.x*myY;

double deltaphi; // Store derivative of phi in one direction
double rhomin = devLambda[3];

__shared__ double phiA[BLOCKDIMX*BLOCKDIMY];
__shared__ double phiB[BLOCKDIMX*BLOCKDIMY];
__shared__ double phiC[BLOCKDIMX*BLOCKDIMY];

__shared__ double locrho[BLOCKDIMX*BLOCKDIMY];
__shared__ double ener[BLOCKDIMX*BLOCKDIMY];

double *U; double *V; double *W;
double *temp;

U = phiA; V = phiB; W = phiC;

// Preload lower and middle planes
U[myLocAddr] = phi[globAddr + arraysize.x*arraysize.y*(arraysize.z-1)];
V[myLocAddr] = phi[globAddr];

__syncthreads();

int z;
int deltaz = arraysize.x*arraysize.y;
for(z = 0; z < arraysize.z; z++) {
  if(z >= arraysize.z - 1) deltaz = - arraysize.x*arraysize.y*(arraysize.z-1);

  ener[myLocAddr]   = 0;
  locrho[myLocAddr] = rho[globAddr]; // rho(z) -> rho
  W[myLocAddr]      = px[globAddr]; // load px(z) -> phiC
  __syncthreads();

  if(IWrite && (locrho[myLocAddr] > rhomin)) {
    deltaphi         = LAMX*(V[myLocAddr+1]-V[myLocAddr-1]);
    if(locrho[myLocAddr] < RHOGRAV) { deltaphi *= (locrho[myLocAddr]*G1 - G2); } // reduce G for low density
    ener[myLocAddr] -= deltaphi*W[myLocAddr]; // ener -= dt * px * dphi/dx
    px[globAddr]     = W[myLocAddr] - deltaphi*locrho[myLocAddr]; // store px <- px - dt * rho dphi/dx;
  }

  W[myLocAddr] = py[globAddr]; // load py(z) -> phiC
  __syncthreads();
  if(IWrite && (locrho[myLocAddr] > rhomin)) {
  if(coords == SQUARE) {
    deltaphi         = LAMY*(V[myLocAddr+BLOCKDIMX]-V[myLocAddr-BLOCKDIMX]);
    }
    if(coords == CYLINDRICAL) {
    // In cylindrical coords, use dt/dphi * (delta-phi) / r to get d/dy
    deltaphi         = LAMY*(V[myLocAddr+BLOCKDIMX]-V[myLocAddr-BLOCKDIMX]) / (RINNER + DELTAR*myX);
    }
   if(locrho[myLocAddr] < RHOGRAV) { deltaphi *= (locrho[myLocAddr]*G1 - G2); } // reduce G for low density
    ener[myLocAddr] -= deltaphi*W[myLocAddr]; // ener -= dt * py * dphi/dy
    py[globAddr]     = W[myLocAddr] - deltaphi*locrho[myLocAddr]; // store py <- py - rho dphi/dy;
  }

  W[myLocAddr]       = phi[globAddr + deltaz]; // load phi(z+1) -> phiC
  __syncthreads();
  deltaphi           = LAMZ*(W[myLocAddr] - U[myLocAddr]);
  if(locrho[myLocAddr] < RHOGRAV) { deltaphi *= (locrho[myLocAddr]*G1 - G2); } // reduce G for low density
  __syncthreads();

  U[myLocAddr]       = pz[globAddr]; // load pz(z) -> phiA
  __syncthreads();
  if(IWrite && (locrho[myLocAddr] > rhomin)) {
    E[globAddr]     += ener[myLocAddr] - deltaphi*U[myLocAddr]; // Store E[x] <- ener - dt *pz * dphi/dz
    pz[globAddr]     = U[myLocAddr] - deltaphi*locrho[myLocAddr]; // store pz <- pz - rho dphi/dz;
  }

  temp = U; U = V; V = W; W = temp; // cyclically shift them back
  globAddr += arraysize.x * arraysize.y;

}

}


/*
 * dP = -rho grad(phi) dt
 * dE = -rho v \cdot grad(phi) dt
 * 
 * Exact integrals at fixed position:
 * P2 = P1 - rho grad(phi) t
 * E2 = E1 - P1 \cdot grad(phi) t + .5 rho grad(phi) \cdot grad(phi) t^2
 */
template <geometryType_t coords>
__global__ void  cukern_applyScalarPotential_2D(double *rho, double *E, double *px, double *py, double *phi, int3 arraysize)
{
	int myLocAddr = threadIdx.x + BLOCKDIMX*threadIdx.y;

	int myX = threadIdx.x + (BLOCKDIMX-2)*blockIdx.x - 1;
	int myY = threadIdx.y + (BLOCKDIMY-2)*blockIdx.y - 1;

	if((myX > arraysize.x) || (myY > arraysize.y)) return;

	bool IWrite = (threadIdx.x > 0) && (threadIdx.x < (BLOCKDIMX-1)) && (threadIdx.y > 0) && (threadIdx.y < (BLOCKDIMY-1));
	IWrite = IWrite && (myX < arraysize.x) && (myY < arraysize.y);

	myX = (myX + arraysize.x) % arraysize.x;
	myY = (myY + arraysize.y) % arraysize.y;

	int globAddr = myX + arraysize.x*myY;

	double deltaphi; // Store derivative of phi in one direction
	double rhomin = devLambda[3];
	double tmpMom;

	__shared__ double phiLoc[BLOCKDIMX*BLOCKDIMY];
	__shared__ double rhoLoc[BLOCKDIMX*BLOCKDIMY];
	double enerLoc = 0.0;

	rhoLoc[myLocAddr] = rho[globAddr]; // rho(z) -> rho
	phiLoc[myLocAddr] = phi[globAddr];

	__syncthreads(); // Make sure loaded phi is visible

	// coupling is exactly zero if rho <= rhomin
	if(IWrite && (rhoLoc[myLocAddr] > rhomin)) {
		// compute dt * (dphi/dx)
		deltaphi         = LAMX*(phiLoc[myLocAddr+1]-phiLoc[myLocAddr-1]);
		// reduce coupling for low densities
		if(rhoLoc[myLocAddr] < RHOGRAV) { deltaphi *= (rhoLoc[myLocAddr]*G1 - G2); }
		// Load px
		tmpMom = px[globAddr];
		// Store delta-E due to change in x momentum: ener -= (dt * dphi/dx) * (px = rho vx) -= rho delta-phi
		enerLoc -= deltaphi*(tmpMom - .5*rhoLoc[myLocAddr]*deltaphi);
		// Update X momentum
		px[globAddr]     = tmpMom - deltaphi*rhoLoc[myLocAddr]; // store px <- px - dt * rho dphi/dx;
		// Calculate dt*(dphi/dy)
		if(coords == SQUARE) {
		deltaphi         = LAMY*(phiLoc[myLocAddr+BLOCKDIMX]-phiLoc[myLocAddr-BLOCKDIMX]);
		}
		if(coords == CYLINDRICAL) {
		// Converts d/dphi into physical distance based on R
		deltaphi         = LAMY*(phiLoc[myLocAddr+BLOCKDIMX]-phiLoc[myLocAddr-BLOCKDIMX]) / (RINNER + myX*DELTAR);
		}
		
		// reduce G for low density
		if(rhoLoc[myLocAddr] < RHOGRAV) { deltaphi *= (rhoLoc[myLocAddr]*G1 - G2); }
		// Load py
		tmpMom = py[globAddr];
		// Update global energy array with this & previous delta-E values
		E[globAddr] += enerLoc - deltaphi*(tmpMom - .5*rhoLoc[myLocAddr]*deltaphi); // ener -= dt * py * dphi/dy
		// Update Y momentum array
		py[globAddr]     = tmpMom - deltaphi*rhoLoc[myLocAddr]; // store py <- py - rho dphi/dy;
	}



}

