#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"

/* THIS FUNCTION
   Given F(xi, yi, zi) and direction dir = { X = 1, Y = 2, Z = 3), this function calculates

   F(xi, yi, zi) <- (F(xi, yi, zi) + F(xi + 1*(dir == 1), yi + 1*(dir == 2), zi + 1*(dir == 3) )/2

   using circular boundary conditions on all 3 directions */

__global__ void cukern_ForwardAverageX(double *in, double *out, int nx);
__global__ void cukern_ForwardAverageY(double *in, double *out, int nx, int ny);
__global__ void cukern_ForwardAverageZ(double *in, double *out, int nx, int nz);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
    if ((nrhs != 2) || (nlhs != 1)) {
        mexErrMsgTxt("Arguments must be result = cudaFwdAverage(array, direction)\n");
        }

    CHECK_CUDA_ERROR("entering cudaFwdAverage");

    MGArray in;
    int worked = MGA_accessMatlabArrays(prhs, 0, 0, &in);
    MGArray *out = MGA_createReturnedArrays(plhs, 1, &in);

    // Establish launch dimensions & a few other parameters
    int direction = (int)*mxGetPr(prhs[1]);

    dim3 arraySize = makeDim3(&in.dim[0]);
    dim3 blocksize, gridsize;
    blocksize.z = 1;
    gridsize.z = 1;

    hipSetDevice(in.deviceID[0]);
    CHECK_CUDA_ERROR("hipSetDevice()");

    // Interpolate the grid-aligned velocity
    switch(direction) {
        case 1:
//            blocksize.x = 18; blocksize.y = 8;
//            gridsize.x = arraySize.x / 14; gridsize.x += (14 * gridsize.x < arraySize.x);
//            gridsize.y = arraySize.y / blocksize.y; gridsize.y += (blocksize.y * gridsize.y < arraySize.y);
//            cukern_fwdAverageX<<<gridsize, blocksize>>>(srcs[0], dest[0], arraySize);
            blocksize = makeDim3(128, 1, 1);
            gridsize.x = arraySize.y; gridsize.y = arraySize.z;
            cukern_ForwardAverageX<<<gridsize, blocksize>>>(in.devicePtr[0], out->devicePtr[0], arraySize.x);
            break;
        case 2:
//            blocksize.x = 8; blocksize.y = 18;
//            gridsize.x = arraySize.x / 8; gridsize.x += (8 * gridsize.x < arraySize.x);
//            gridsize.y = arraySize.y / 14; gridsize.y += (14 * gridsize.x < arraySize.y);
//            cukern_fwdAverageY<<<gridsize, blocksize>>>(srcs[0], dest[0], arraySize);
            blocksize = makeDim3(64, 1, 1);
            gridsize.x = arraySize.x / 64; gridsize.x += (64*gridsize.x < arraySize.x);
            gridsize.y = arraySize.z;
            cukern_ForwardAverageY<<<gridsize, blocksize>>>(in.devicePtr[0], out->devicePtr[0], arraySize.x, arraySize.y);
            break;
        case 3:
//            blocksize.x = 18; blocksize.y = 8;
//            gridsize.x = arraySize.z / 14; gridsize.x += (14 * gridsize.x < arraySize.z);
//            gridsize.y = arraySize.x / blocksize.y; gridsize.y += (blocksize.y * gridsize.y < arraySize.x);
//            cukern_fwdAverageZ<<<gridsize, blocksize>>>(srcs[0], dest[0], arraySize);
              blocksize = makeDim3(64, 1, 1);
              gridsize.x = arraySize.x / 64; gridsize.x += (64*gridsize.x < arraySize.x);
              gridsize.y = arraySize.y;
              cukern_ForwardAverageZ<<<gridsize, blocksize>>>(in.devicePtr[0], out->devicePtr[0], arraySize.x, arraySize.z);
            break;
        }

    free(out);

    CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, &in, direction, "Forward averaging");

}

/*
Invoke with a grid for which blockdim.x = size(array, Y) and blockdim.y = size(array,Z);
with 128 threads in the X direction.
*/
__global__ void cukern_ForwardAverageX(double *in, double *out, int nx)
{
int yAddr = blockIdx.x;
int zAddr = blockIdx.y;
int ny = gridDim.x;

int addrMax = nx*(yAddr + ny*zAddr + 1); // The address which we must not reach or go beyond is the start of the next line
int readBase = threadIdx.x + nx*(yAddr + ny*zAddr);
int writeBase= readBase;
readBase -= nx*(readBase >= addrMax);

__shared__ double lStore[256];

int locAddr = threadIdx.x;

//lStore[locAddr] = in[globBase + readX]; // load the first memory segment
lStore[locAddr] = in[readBase]; // load the first memory segment

do {
    readBase += 128;
    readBase -= nx*(readBase >= addrMax);
    lStore[(locAddr + 128) % 256] = in[readBase];

    __syncthreads(); // We have now read ahead by a segment. Calculate forward average, comrades!

    if(writeBase < addrMax) { out[writeBase] = .5*(lStore[locAddr] + lStore[(locAddr+1)%256]); } // If we are within range, that is.
    writeBase += 128; // Advance write address
    if(writeBase >= addrMax) return; // If write address is beyond nx, we're finished.
    locAddr ^= 128;

    __syncthreads();

    } while(1);

}

/* Invoke with a blockdim of <64, 1, 1> threads
Invoke with a griddim = <ceil[nx / 64], nz, 1> */
__global__ void cukern_ForwardAverageY(double *in, double *out, int nx, int ny)
{
int xaddr = blockDim.x * blockIdx.x + threadIdx.x; // There are however many X threads
if(xaddr >= nx) return; // truncate this right off

__shared__ double tileA[64];
__shared__ double tileB[64];

double *setA = tileA;
double *setB = tileB;
double *swap;

int readBase = xaddr + nx*ny*blockIdx.y; // set Raddr to x + nx ny z
int writeBase = readBase;
int addrMax = readBase + nx*(ny - 1); // Set this to the max address we want to handle in the loop

setB[threadIdx.x] = in[readBase]; // load set B (e.g. row 0)

while(writeBase < addrMax) { // Exit one BEFORE the max address to handle (since the max is a special case)
    swap = setB; // exchange A/B pointers
    setB = setA;
    setA = swap; // swap so that row 0 is set A
    
//    __syncthreads();

    readBase += nx; // move pointer down one row
    setB[threadIdx.x] = in[readBase]; // load row 1 into set B

//    __syncthreads();

    out[writeBase] = .5*(setB[threadIdx.x] + setA[threadIdx.x]); // average written to output

    writeBase += nx;
    }

readBase = xaddr + nx*ny*blockIdx.y; // reset readbase
setA[threadIdx.x] = in[readBase];
out[writeBase] = .5*(setB[threadIdx.x] + setA[threadIdx.x]); // average written to output

}

/* Invoke with a blockdim of <64, 1, 1> threads
Invoke with a griddim = <ceil[nx / 64], ny, 1> */
__global__ void cukern_ForwardAverageZ(double *in, double *out, int nx, int nz)
{
int xaddr = blockDim.x * blockIdx.x + threadIdx.x; // There are however magridDim.y X threads
if(xaddr >= nx) return; // truncate this right off

__shared__ double tileA[64];
__shared__ double tileB[64];

double *setA = tileA;
double *setB = tileB;
double *swap;

int readBase = xaddr + nx*blockIdx.y; // set Raddr to x + nx gridDim.y z
int writeBase = readBase;
int addrMax = readBase + nx*gridDim.y*(nz - 1); // Set this to the max address we want to handle in the loop

setB[threadIdx.x] = in[readBase]; // load set B (e.g. row 0)

while(writeBase < addrMax) { // Exit one BEFORE the max address to handle (since the max is a special case)
    swap = setB; // exchange A/B pointers
    setB = setA;
    setA = swap; // swap so that row 0 is set A

//    __syncthreads();

    readBase += nx*gridDim.y; // move pointer down one row
    setB[threadIdx.x] = in[readBase]; // load row 1 into set B

//    __syncthreads();

    out[writeBase] = .5*(setB[threadIdx.x] + setA[threadIdx.x]); // average written to output

    writeBase += nx*gridDim.y;
    }

readBase = xaddr + nx*blockIdx.y; // reset readbase
setA[threadIdx.x] = in[readBase];
out[writeBase] = .5*(setB[threadIdx.x] + setA[threadIdx.x]); // average written to output

}

