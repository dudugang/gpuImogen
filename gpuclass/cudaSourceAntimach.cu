#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"

#define BLOCKDIMX 16
#define BLOCKDIMY 16

__global__ void  cukern_AntiMach(double *rho, double *E, double *px, double *py, double *pz, int3 arraysize);

__constant__ __device__ double devLambda[2];

/*mass.gputag, ener.gputag, mom(1).gputag, mom(2).gputag, 1, run.time.dTime, xg.GPU_MemPtr, yg.GPU_MemPtr*/

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
    // At least 2 arguments expected
    // Input and result
    if ((nrhs!=7) || (nlhs != 0)) mexErrMsgTxt("Wrong number of arguments: need cudaApplyScalarPotential(rho, E, px, py, pz, gamma, m_max)\n");


    cudaCheckError("entering cudaSourceRotatingFrame");

    // Get source array info and create destination arrays
    ArrayMetadata amd;
    double **srcs = getGPUSourcePointers(prhs, &amd, 0, 4);

    double maxmach = *mxGetPr(prhs[6]); // get dt and dx to obey CFL such that .5 a tau^2 << h
    double polygamma = *mxGetPr(prhs[5]);

    dim3 gridsize, blocksize;
    int3 arraysize; arraysize.x = amd.dim[0]; arraysize.y = amd.dim[1]; arraysize.z = amd.dim[2];

    blocksize.x = BLOCKDIMX; blocksize.y = BLOCKDIMY; blocksize.z = 1;
    gridsize.x = arraysize.x / (blocksize.x); gridsize.x += ((blocksize.x) * gridsize.x < amd.dim[0]);
    gridsize.y = arraysize.z;
    gridsize.z = 1;

    double lambda[2];
    lambda[0] = maxmach;
    lambda[1] = polygamma*(polygamma-1.0);

    hipMemcpyToSymbol(HIP_SYMBOL(devLambda), &lambda[0], 2*sizeof(double), 0, hipMemcpyHostToDevice);
    cukern_AntiMach<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], arraysize);

    hipError_t epicFail = hipGetLastError();
    if(epicFail != hipSuccess) cudaLaunchError(epicFail, blocksize, gridsize, &amd, -1, "applyScalarPotential");

}

/* rho, E, Px, Py, Pz: arraysize-sized arrays */
#define M_0sq devLambda[0]
#define GG1 devLambda[1]

__global__ void  cukern_AntiMach(double *rho, double *E, double *px, double *py, double *pz, int3 arraysize)
{
/* strategy: XY files, fill in X direction, step in Y direction; griddim.y = Nz */
int myx = threadIdx.x + BLOCKDIMX*blockIdx.x;
int myy = threadIdx.y;
int myz = blockIdx.y;
int nx = arraysize.x; int ny = arraysize.y;

if(myx >= arraysize.x) return; 

int globaddr = myx + nx*(myy + ny*myz);

double locRho;
double locMom[3];
double locEner;
/*double inv_rsqr, xy;*/
double momsq;
double mach;

double dmomentum;

#ifdef DEBUGME
int stopme = (myz == 40) && (blockIdx.x == 12);
#endif

for(; myy < ny; myy += BLOCKDIMY) {

  locRho    = rho[globaddr];
  locMom[0] = px[globaddr];
  locMom[1] = py[globaddr];
  locMom[2] = pz[globaddr];
  locEner   = E[globaddr];
 
  // calculate local momentum density
  momsq = locMom[0]*locMom[0]+locMom[1]*locMom[1]+locMom[2]*locMom[2];

  // mach squared = v^2 / c_s^2 = p^2 / [rho (gg1 (E - .5 p^2 / rho) ) ]
  //                            = p^2 / [ gg1 ( rho E - .5 p^2) ]
  mach = momsq/(GG1*(locRho*locEner-.5*momsq));

  if(mach > M_0sq) { 
    mach = sqrt(mach/M_0sq) - 1; // Calculate mach-hat - 1 as our braking parameter
    // calculate rho * accel * dt (= dmomentum) * [ vector(momentum) * (ms/m0-1)^2 / |momentum| ]
    dmomentum = 1.0/(1.0 + mach*mach);

    // Apply braking force
    px[globaddr] = locMom[0] *dmomentum;
    py[globaddr] = locMom[1] *dmomentum;
    pz[globaddr] = locMom[2] *dmomentum;
    E[globaddr]  = locEner + .5*momsq*(dmomentum*dmomentum - 1.0)/locRho;
    }

  globaddr += nx*BLOCKDIMY;
  }
}

