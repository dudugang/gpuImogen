#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"

__global__ void cukern_ArrayTranspose2D(double *src, double *dst, int nx, int ny);
__global__ void cukern_ArrayExchangeY(double *src, double *dst,   int nx, int ny, int nz);
__global__ void cukern_ArrayExchangeZ(double *src, double *dst,   int nx, int ny, int nz);

#define BDIM 16

__global__ void cukern_dumbblit(double *src, double *dst, int nx, int ny, int nz);

__global__ void cukern_dumbblit(double *src, double *dst, int nx, int ny, int nz)
{
	//int myx = threadIdx.x + BDIM*blockIdx.x;
	//int myy = threadIdx.y + BDIM*((blockIdx.y + blockIdx.x) % gridDim.y);
	//int myaddr = myx + nx*myy;

	//if((myx < nx) && (myy < ny)) dst[myaddr] = src[myaddr];
	return;

}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{

	dim3 blocksize; blocksize.x = blocksize.y = BDIM; blocksize.z = 1;
	dim3 gridsize;

        if(nrhs != 2) { mexErrMsgTxt("Input args must be cudaArrayRotate2(GPU array, dir to transpose with X)"); }
        int makeNew = 0;
        switch(nlhs) {
            case 0: makeNew = 0; break;
            case 1: makeNew = 1; break;
            default: mexErrMsgTxt("cudaArrayRotate must return zero (alters input) or one (returns new) arguments."); break;
        }
	CHECK_CUDA_ERROR("entering cudaArrayRotate2");

	MGArray src;
	int worked = accessMGArrays(prhs, 0, 0, &src);

	/* This function will make the partition direction track the transposition of indices
	 * Such that if partitioning direction is X and a Y transpose is done, partition is in Y.
	 * The full matrix
	 * transpose =  XY | XZ | YZ |
	 *            +----+----+----+
	 * Part.    X | Y  | Z  | X  |
	 * initial  Y | X  | Y  | Z  | <- Output array will have partition in this direction
	 * direct   Z | Z  | X  | Y  |
	 */

	MGArray copy = src;
	MGArray *clone;

	int indExchange = (int)*mxGetPr(prhs[1]);
	int i, sub[6];

	switch(src.dim[2] > 1 ? 3 : 2) { /* on # dimensions */
	case 3: // x/y/z array, exchanging x and y
		if(indExchange == 2) {
			gridsize.x = src.dim[0] / BDIM; if(gridsize.x*BDIM < src.dim[0]) gridsize.x++;
			gridsize.y = src.dim[1] / BDIM; if(gridsize.y*BDIM < src.dim[1]) gridsize.y++;

			blocksize.x = blocksize.y = BDIM; blocksize.z = 1;

			// Transpose X and Y sizes
			copy.dim[0] = src.dim[1];
			copy.dim[1] = src.dim[0];
			// Flip the partition direction if appropriate
			if(copy.partitionDir == PARTITION_X) {
				copy.partitionDir = PARTITION_Y;
			} else if(copy.partitionDir == PARTITION_Y) {
				copy.partitionDir = PARTITION_X;
			}

			// Recalculate the partition sizes
			for(i = 0; i < copy.nGPUs; i++) {
				calcPartitionExtent(&copy, i, sub);
				copy.partNumel[i] = sub[3]*sub[4]*sub[5];
			}

			// Setup the new array: Either create new or overwrite original input tag
                        if(makeNew) {
                            clone = createMGArrays(plhs, 1, &copy);
                        } else { 
                            clone = allocMGArrays(1, &copy);
                            serializeMGArrayToTag(clone, (int64_t *)mxGetData(prhs[0]));
                        }

			for(i = 0; i < copy.nGPUs; i++) {
				hipSetDevice(copy.deviceID[i]);
				CHECK_CUDA_ERROR("hipSetDevice()");
                                calcPartitionExtent(&src, i, sub);
				cukern_ArrayExchangeY<<<gridsize, blocksize>>>(src.devicePtr[i], clone->devicePtr[i], sub[3],sub[4],sub[5]);
				CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, &src, i, "array transposition");
                                if(makeNew == false) hipFree(src.devicePtr[i]);
				CHECK_CUDA_ERROR("hipFree");
			}
			free(clone);
		}
		if(indExchange == 3) {
			gridsize.x = src.dim[0] / BDIM; if(gridsize.x*BDIM < src.dim[0]) gridsize.x++;
			gridsize.y = src.dim[2] / BDIM; if(gridsize.y*BDIM < src.dim[2]) gridsize.y++;

			blocksize.x = blocksize.y = BDIM; blocksize.z = 1;

			// Transpose X and Z
			copy.dim[0] = src.dim[2];
			copy.dim[2] = src.dim[0];
                        // Flip the partition direction if appropriate
                        if(copy.partitionDir == PARTITION_X) {
                                copy.partitionDir = PARTITION_Z;
                        } else if(copy.partitionDir == PARTITION_Z) {
                                copy.partitionDir = PARTITION_X;
                        }

			// Recalculate the partition sizes
			for(i = 0; i < copy.nGPUs; i++) {
				calcPartitionExtent(&copy, i, sub);
				copy.partNumel[i] = sub[3]*sub[4]*sub[5];
			}
                        // Setup the new array: Either create new or overwrite original input tag
                        if(makeNew) {
                            clone = createMGArrays(plhs, 1, &copy);
                        } else {
                            clone = allocMGArrays(1, &copy);
                            serializeMGArrayToTag(clone, (int64_t *)mxGetData(prhs[0]));
                        }

			for(i = 0; i < copy.nGPUs; i++) {
				hipSetDevice(copy.deviceID[i]);
				CHECK_CUDA_ERROR("hipSetDevice()");
                                calcPartitionExtent(&src, i, sub);
				cukern_ArrayExchangeZ<<<gridsize, blocksize>>>(src.devicePtr[i], clone->devicePtr[i], sub[3], sub[4], sub[5]);
				CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, &src, i, "array transposition");
				if(makeNew == false) hipFree(src.devicePtr[i]);
                                CHECK_CUDA_ERROR("hipFree()");
			}

			free(clone);
		}
		break;
	case 2:
		gridsize.x = src.dim[0] / BDIM; if(gridsize.x*BDIM < src.dim[0]) gridsize.x++;
		gridsize.y = src.dim[1] / BDIM; if(gridsize.y*BDIM < src.dim[1]) gridsize.y++;

		blocksize.x = blocksize.y = BDIM; blocksize.z = 1;

		// Transpose X and Y
		copy.dim[0] = src.dim[1];
		copy.dim[1] = src.dim[0];
                // Flip the partition direction if appropriate
                if(copy.partitionDir == PARTITION_X) {
                        copy.partitionDir = PARTITION_Y;
                } else if(copy.partitionDir == PARTITION_Y) {
                        copy.partitionDir = PARTITION_X;
                }

		// Recalculate the partition sizes
		for(i = 0; i < copy.nGPUs; i++) {
			calcPartitionExtent(&copy, i, sub);
			copy.partNumel[i] = sub[3]*sub[4]*sub[5];
		}

                // Setup the new array: Either create new or overwrite original input tag
                if(makeNew) {
                    clone = createMGArrays(plhs, 1, &copy);
                } else {
                    clone = allocMGArrays(1, &copy);
                    serializeMGArrayToTag(clone, (int64_t *)mxGetData(prhs[0]));
                }

		for(i = 0; i < copy.nGPUs; i++) {
			hipSetDevice(copy.deviceID[i]);
			CHECK_CUDA_ERROR("hipSetDevice()");
                        calcPartitionExtent(&src, i, sub);
			cukern_ArrayTranspose2D<<<gridsize, blocksize>>>(src.devicePtr[i], clone->devicePtr[i], sub[3], sub[4]);
			CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, &src, i, "array transposition");
			if(makeNew == false) hipFree(src.devicePtr[i]);
                        CHECK_CUDA_ERROR("hipFree()");
		}
		free(clone);

		break;
	}




}

__global__ void cukern_ArrayTranspose2D(double *src, double *dst, int nx, int ny)
{
	__shared__ double tmp[BDIM][BDIM];

	int myx = threadIdx.x + BDIM*blockIdx.x;
	int myy = threadIdx.y + BDIM*((blockIdx.y + blockIdx.x) % gridDim.y);
	int myAddr = myx + nx*myy;

	if((myx < nx) && (myy < ny)) tmp[threadIdx.y][threadIdx.x] = src[myAddr];

	__syncthreads();

	//myx = threadIdx.x + BDIM*((blockIdx.y + blockIdx.x) % gridDim.y);
	myAddr = myy + threadIdx.x - threadIdx.y;
	//myy = threadIdx.y + BDIM*blockIdx.x;
	myy  = myx + threadIdx.y - threadIdx.x;
	myx = myAddr;

	myAddr = myx + ny*myy;

	if((myx < ny) && (myy < nx)) dst[myAddr] = tmp[threadIdx.x][threadIdx.y];

}

__global__ void cukern_ArrayExchangeY(double *src, double *dst, int nx, int ny, int nz)
{

	__shared__ double tmp[BDIM][BDIM];

	int myx = threadIdx.x + BDIM*blockIdx.x;
	int myy = threadIdx.y + BDIM*((blockIdx.y + blockIdx.x) % gridDim.y);
	int mySrcAddr = myx + nx*myy;
	bool doRead = 0;
	bool doWrite = 0;

	if((myx < nx) && (myy < ny)) doRead = 1;

	myx = threadIdx.x + BDIM*((blockIdx.y + blockIdx.x) % gridDim.y);
	myy = threadIdx.y + BDIM*blockIdx.x;
	int myDstAddr = myx + ny*myy;

	if((myx < ny) && (myy < nx)) doWrite = 1;

	for(myx = 0; myx < nz; myx++) {
		if(doRead) tmp[threadIdx.y][threadIdx.x] = src[mySrcAddr];
		mySrcAddr += nx*ny;
		__syncthreads();

		if(doWrite) dst[myDstAddr] = tmp[threadIdx.x][threadIdx.y];
		myDstAddr += nx*ny;
		__syncthreads();
	}

}

__global__ void cukern_ArrayExchangeZ(double*src, double *dst, int nx, int ny, int nz)
{
	__shared__ double tmp[BDIM][BDIM];

	int myx = threadIdx.x + BDIM*blockIdx.x;
	int myz = threadIdx.y + BDIM*((blockIdx.y + blockIdx.x) % gridDim.y);
	int mySrcAddr = myx + nx*ny*myz;
	bool doRead = 0;
	bool doWrite = 0;

	if((myx < nx) && (myz < nz)) doRead = 1;

	myx = threadIdx.x + BDIM*((blockIdx.y + blockIdx.x) % gridDim.y);
	myz = threadIdx.y + BDIM*blockIdx.x;
	int myDstAddr = myx + nz*ny*myz;

	if((myx < nz) && (myz < nx)) doWrite = 1;

	for(myx = 0; myx < ny; myx++) {
		if(doRead) tmp[threadIdx.y][threadIdx.x] = src[mySrcAddr];
		mySrcAddr += nx;
		__syncthreads();

		if(doWrite) dst[myDstAddr] = tmp[threadIdx.x][threadIdx.y];
		myDstAddr += nz;
		__syncthreads();
	}


}

