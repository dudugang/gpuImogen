#include "hip/hip_runtime.h"
/*
 * flux_ML_iface.c
 *
 *  Created on: Nov 25, 2015
 *      Author: erik
 */

#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

#include "hip/hip_runtime.h"
#include "mpi.h"

#include "cudaCommon.h"
#include "cudaFluidStep.h"
#include "flux.h"

// Only uncomment this if you plan to debug this file.
//#define DEBUGMODE


#ifdef DEBUGMODE
    #include "debug_inserts.h"
#endif

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	int wanted_nlhs = 0;
#ifdef DEBUGMODE
	wanted_nlhs = 1;
#endif

	if ((nrhs!= 7) || (nlhs != wanted_nlhs)) mexErrMsgTxt("Wrong number of arguments: need flux_ML_iface(fluid, bx, by, bz, [dt, purehydro?, fluid gamma, min density, order, step #, step method, geom_type, geom_Rinner], topology, {dx,dy,dz})\n");

	MGArray fluid[5];

	/* Access bx/by/bz cell-centered arrays if magnetic!!!! */
	/* ... */

    int idxpost = 4; // 8 for the old way

	double *scalars = mxGetPr(prhs[idxpost]);

	if(mxGetNumberOfElements(prhs[idxpost]) != 8) {
		DROP_MEX_ERROR("Must rx 8 parameters in params vector: [dt, purehydro?, fluid gamma, rhomin, order, step #, step method, geomtype, Rin]");
	}

	double dt     = scalars[0]; /* Access lambda (dt / dx) */
	int ishydro   = scalars[1]; /* determine if purely hydrodynamic */
	double gamma  = scalars[2]; /* Adiabatic index of fluid */

	int sweepDirect = (int)scalars[3]; /* Identify if forwards (sweepDirect = 1) or backwards (-1) */
	int stepNum     = (int)scalars[4]; /* step number (used to pick the permutation of the fluid propagators) */
	int stepMethod  = (int)scalars[5]; /* 1=HLL, 2=HLLC, 3=Xin/Jin */

	int geomType    = (int)scalars[6];
	double geomRin  = scalars[7];

	/* Access topology structure */
	ParallelTopology topo;
	topoStructureToC(prhs[idxpost+1], &topo);

	FluidStepParams fsp;
	fsp.dt = dt;

	// Getting cell spacing data:
	int gotcells = mxIsCell(prhs[idxpost+2]);

	if(gotcells) {
		mxArray *dxi;
		int q;
		for(q = 0; q < 3; q++) {
			dxi = mxGetCell(prhs[idxpost+2], q);
			if(dxi != NULL) {
				fsp.h[q] = *mxGetPr(dxi);
			} else {
				printf("Attempted to get %ith cell element", q+1);
				mexErrMsgTxt("Attempt to get array in {dx,dy,dz} failed!\n");
			}
		}
	} else {
		mexErrMsgTxt("Expected argument 11 to be {dx, dy, dz}, was not a cell array");
	}

	fsp.geomCylindricalRinner = geomRin;
	switch(geomType) {
	default:
	case 1:
		fsp.geometryType = SQUARE;
		break;
	case 2:
		fsp.geometryType = CYLINDRICAL;
	}
	fsp.onlyHydro = ishydro;
	fsp.stepDirection = sweepDirect;
	fsp.stepMethod = stepMethod;
	fsp.thermoGamma = gamma;

	int numFluids = mxGetNumberOfElements(prhs[0]);
	int fluidct;
	CHECK_CUDA_ERROR("entering compiled fluid step");

	int status;

	for(fluidct = 0; fluidct < numFluids; fluidct++) {
		status = MGA_accessFluidCanister(prhs[0], fluidct, &fluid[0]);
		if(CHECK_IMOGEN_ERROR(status) != SUCCESSFUL) break;
		double rhoMin;
		mxArray *flprop = mxGetProperty(prhs[0], fluidct, "MINMASS");
		if(flprop != NULL) {
			rhoMin = *((double *)mxGetPr(flprop));
		} else {
			status = ERROR_NULL_POINTER;
		}
		fsp.minimumRho = rhoMin;
		//performFluidUpdate_3D(&fluid[0], &topo, sweepDirect, stepNum, &lambda[0], gamma, rhoMin, stepMethod);
		performFluidUpdate_3D(&fluid[0], &topo, fsp, stepNum, sweepDirect);
		if(CHECK_IMOGEN_ERROR(status) != SUCCESSFUL) break;
	}
	if(status != SUCCESSFUL) {
		DROP_MEX_ERROR("Fluid update code returned unsuccessfully!");
	}

}
