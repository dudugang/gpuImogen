#include "hip/hip_runtime.h"
/*
 * flux_ML_iface.c
 *
 *  Created on: Nov 25, 2015
 *      Author: erik
 */

#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

#include "hip/hip_runtime.h"
#include "mpi.h"

#include "cudaCommon.h"
#include "flux.h"

// Only uncomment this if you plan to debug this file.
//#define DEBUGMODE


#ifdef DEBUGMODE
    #include "debug_inserts.h"
#endif

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	int wanted_nlhs = 0;
#ifdef DEBUGMODE
	wanted_nlhs = 1;
#endif

	if ((nrhs!= 7) || (nlhs != wanted_nlhs)) mexErrMsgTxt("Wrong number of arguments: need flux_ML_iface(fluid, bx, by, bz, [dt, purehydro?, fluid gamma, min density, order, step #, step method], topology, {dx,dy,dz})\n");

	MGArray fluid[5];

	/* Access bx/by/bz cell-centered arrays if magnetic!!!! */
	/* ... */

    int idxpost = 4; // 8 for the old way

	double *scalars = mxGetPr(prhs[idxpost]);

	if(mxGetNumberOfElements(prhs[idxpost]) != 6) {
		DROP_MEX_ERROR("Must rx 7 parameters in params vector: [dt, purehydro?, fluid gamma, order, step #, step method]");
	}

	double dt     = scalars[0]; /* Access lambda (dt / dx) */
	int ishydro   = scalars[1]; /* determine if purely hydrodynamic */
	double gamma  = scalars[2]; /* Adiabatic index of fluid */

	int sweepDirect = (int)scalars[3]; /* Identify if forwards (sweepDirect = 1) or backwards (-1) */
	int stepNum     = (int)scalars[4]; /* step number (used to pick the permutation of the fluid propagators) */
	int stepMethod  = (int)scalars[5]; /* 1=HLL, 2=HLLC, 3=Xin/Jin */

	/* Access topology structure */
	ParallelTopology topo;
	topoStructureToC(prhs[idxpost+1], &topo);

	double lambda[3];

	// Getting cell spacing data:
	int gotcells = mxIsCell(prhs[idxpost+2]);

	if(gotcells) {
		mxArray *dxi;
		int q;
		for(q = 0; q < 3; q++) {
			dxi = mxGetCell(prhs[idxpost+2], q);
			if(dxi != NULL) {
				lambda[q] = dt / (*mxGetPr(dxi));
			} else {
				printf("Attempted to get %ith cell element", q+1);
				mexErrMsgTxt("Attempt to get array in {dx,dy,dz} failed!\n");
			}
		}
	} else {
		mexErrMsgTxt("Expected argument 11 to be {dx, dy, dz}, was not a cell array");
	}

	int numFluids = mxGetNumberOfElements(prhs[0]);
	int fluidct;
	CHECK_CUDA_ERROR("entering compiled fluid step");

	int status;

	for(fluidct = 0; fluidct < numFluids; fluidct++) {
		status = MGA_accessFluidCanister(prhs[0], fluidct, &fluid[0]);
		if(CHECK_IMOGEN_ERROR(status) != SUCCESSFUL) break;
		double rhoMin;
		mxArray *flprop = mxGetProperty(prhs[0], fluidct, "MINMASS");
		if(flprop != NULL) {
			rhoMin = *((double *)mxGetPr(flprop));
		} else {
			status = ERROR_NULL_POINTER;
		}
		performFluidUpdate_3D(&fluid[0], &topo, sweepDirect, stepNum, &lambda[0], gamma, rhoMin, stepMethod);
		if(CHECK_IMOGEN_ERROR(status) != SUCCESSFUL) break;
	}
	if(status != SUCCESSFUL) {
		DROP_MEX_ERROR("Fluid update code returned unsuccessfully!");
	}

}
