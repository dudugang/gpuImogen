#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"

// host_array = GPU_download(gpu type)

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  // At least 2 arguments expected
  // Input and result
  if((nlhs != 1) || (nrhs != 1)) { mexErrMsgTxt("Form: host_array = GPU_download(GPU array)"); }

  CHECK_CUDA_ERROR("entering GPU_download");

  MGArray m;
  
  MGA_accessMatlabArrays(prhs, 0, 0, &m);

  int nd = 3;
  if(m.dim[2] == 1) {
    nd = 2;
    if(m.dim[1] == 1) {
      nd = 1;
    }
  }
  mwSize odims[3];
  odims[0] = m.dim[0];
  odims[1] = m.dim[1];
  odims[2] = m.dim[2];

  // Create output numeric array
  plhs[0] = mxCreateNumericArray(nd, odims, mxDOUBLE_CLASS, mxREAL);

  double *result = mxGetPr(plhs[0]);

  int itworked = MGA_downloadArrayToCPU(&m, &result, 0);

  return;
}
