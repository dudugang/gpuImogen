#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cudaCommon.h"

/* THIS FUNCTION
The cudaArrayAtomic function is meant to perform operations that operate elementsize
on single arrays. The only such functions yet encountered are in "control" functions where
we require that either density be kept to a minimum value, or that NaNs be replaced by 0s.
*/

__global__ void cukern_ArraySetMin(double *array, double min,    int n);
__global__ void cukern_ArraySetMax(double *array, double max,    int n);
__global__ void cukern_ArrayFixNaN(double *array, double fixval, int n);

#define BLOCKDIM 256

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  // At least 2 arguments expected
  // Input and result
  if (nrhs!=3)
     mexErrMsgTxt("Wrong number of arguments. Expected form: cudaArrayAtomic(gputag, value, [1: set min, 2: set max, 3: NaN->value])");

  // Get GPU array pointers
  double val       = *mxGetPr(prhs[1]);
  int operation = (int)*mxGetPr(prhs[2]);

  ArrayMetadata amd;
  double **atomArray = getGPUSourcePointers(prhs, &amd, 0, 0);

CHECK_CUDA_ERROR("Entering cudaArrayAtomic");

  switch(operation) {
    case 1: cukern_ArraySetMin<<<128, BLOCKDIM>>>(atomArray[0], val, amd.numel); break;
    case 2: cukern_ArraySetMax<<<128, BLOCKDIM>>>(atomArray[0], val, amd.numel); break;
    case 3: cukern_ArrayFixNaN<<<128, BLOCKDIM>>>(atomArray[0], val, amd.numel); break;
  }

CHECK_CUDA_LAUNCH_ERROR(256, 128, &amd, operation, "array min/max/nan sweeping");

}

__global__ void cukern_ArraySetMin(double *array, double min, int n)
{
int x = threadIdx.x + blockDim.x * blockIdx.x;
int dx = blockDim.x * gridDim.x;

while(x < n) {
    if(array[x] < min) array[x] = min;
    x += dx;
    }

}

__global__ void cukern_ArraySetMax(double *array, double max, int n)
{
int x = threadIdx.x + blockDim.x * blockIdx.x;
int dx = blockDim.x * gridDim.x;

while(x < n) {
    if(array[x] > max) array[x] = max;
    x += dx;
    }

}

__global__ void cukern_ArrayFixNaN(double *array, double fixval, int n)
{
int x = threadIdx.x + blockDim.x * blockIdx.x;
int dx = blockDim.x * gridDim.x;

while(x < n) {
    if( isnan(array[x])) array[x] = fixval;
    x += dx;
    }

}

